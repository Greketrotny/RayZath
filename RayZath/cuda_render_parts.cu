#include "hip/hip_runtime.h"
#include "cuda_render_parts.cuh"
#include "rzexception.h"

#include "hiprand.h"

namespace RayZath
{
	namespace CudaEngine
	{
		// ~~~~~~~~ [STRUCT] CudaMaterial ~~~~~~~~
		CudaMaterial& CudaMaterial::operator=(const Material& material)
		{
			this->reflectance = material.GetReflectance();
			this->glossiness = material.GetGlossiness();
			this->transmittance = material.GetTransmittance();
			this->ior = material.GetIndexOfRefraction();
			this->emittance = material.GetEmittance();
			this->scattering = material.GetScattering();
			return *this;
		}
		// ~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~ 



		// ~~~~~~~~ [STRUCT] RandomNumbers ~~~~~~~~
		void RandomNumbers::Reconstruct()
		{
			// generate random numbers
			for (uint32_t i = 0u; i < s_count; ++i)
				m_unsigned_uniform[i] = (rand() % RAND_MAX) / static_cast<float>(RAND_MAX);
		}
		// ~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~

		// ~~~~~~~~ [SRUCT] Seeds ~~~~~~~~
		void Seeds::Reconstruct(hipStream_t& stream)
		{
			// generate random seeds
			for (uint32_t i = 0u; i < s_count; ++i)
				m_seeds[i] = rand() % s_count;
		}
		// ~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~



		// ~~~~~~~~ [STRUCT] CudaConstantKernel ~~~~~~~~
		void CudaConstantKernel::Reconstruct()
		{
			m_random_numbers.Reconstruct();
		}
		// ~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~

		// ~~~~~~~~ [CLASS] CudaGlobalKernel ~~~~~~~~
		CudaGlobalKernel::CudaGlobalKernel()
			: m_render_idx(0u)
		{}
		CudaGlobalKernel::~CudaGlobalKernel()
		{}

		void CudaGlobalKernel::Reconstruct(
			uint32_t render_idx,
			hipStream_t& stream)
		{
			m_render_idx = render_idx;
			m_seeds.Reconstruct(stream);
		}
		// ~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~



		// ~~~~~~~~ [STRUCT] CudaTexture ~~~~~~~~
		hipChannelFormatDesc CudaTexture::chanelDesc = hipCreateChannelDesc<uchar4>();

		CudaTexture::CudaTexture()
			: textureArray(nullptr)
			, textureObject(0)
		{}
		CudaTexture::~CudaTexture()
		{
			if (textureObject) CudaErrorCheck(hipDestroyTextureObject(textureObject));
			if (textureArray)  CudaErrorCheck(hipFreeArray(textureArray));

			this->textureObject = 0;
			this->textureArray = nullptr;
		}

		void CudaTexture::Reconstruct(
			const Texture& host_texture,
			hipStream_t& mirror_stream)
		{
			if (this->textureArray == nullptr)
			{//--> hostMesh has texture but device equivalent doesn't

				// texture array allocation
				CudaErrorCheck(hipMallocArray(
					&this->textureArray,
					&this->chanelDesc,
					host_texture.GetBitmap().GetWidth(), host_texture.GetBitmap().GetHeight()));

				// copy host texture data to device array
				CudaErrorCheck(hipMemcpyToArray(
					this->textureArray,
					0, 0, host_texture.GetBitmap().GetMapAddress(),
					host_texture.GetBitmap().GetWidth() * host_texture.GetBitmap().GetHeight() * sizeof(Graphics::Color),
					hipMemcpyKind::hipMemcpyHostToDevice));

				// specify resource description			
				memset(&this->resDesc, 0, sizeof(hipResourceDesc));
				this->resDesc.resType = hipResourceType::hipResourceTypeArray;
				this->resDesc.res.array.array = this->textureArray;

				// specify texture object parameters
				memset(&this->textureDesc, 0, (sizeof(hipTextureDesc)));
				this->textureDesc.addressMode[0] = hipTextureAddressMode::hipAddressModeWrap;
				this->textureDesc.addressMode[1] = hipTextureAddressMode::hipAddressModeWrap;
				if (host_texture.GetFilterMode() == Texture::FilterMode::Point)
					this->textureDesc.filterMode = hipTextureFilterMode::hipFilterModePoint;
				else
					this->textureDesc.filterMode = hipTextureFilterMode::hipFilterModeLinear;
				this->textureDesc.readMode = hipTextureReadMode::hipReadModeNormalizedFloat;
				this->textureDesc.normalizedCoords = 1;

				// craete texture object
				CudaErrorCheck(hipCreateTextureObject(
					&this->textureObject,
					&this->resDesc,
					&this->textureDesc,
					nullptr));
			}
			else
			{//--> Both hostMesh and deviceMesh have texture

				// get texture array info (width and height)
				hipExtent arrayInfo;
				CudaErrorCheck(hipArrayGetInfo(nullptr, &arrayInfo, nullptr, this->textureArray));

				if (arrayInfo.width * arrayInfo.height != host_texture.GetBitmap().GetWidth() * host_texture.GetBitmap().GetHeight())
				{//--> size of hostMesh texture and CudaMesh texture doesn't match

					// free CudaMesh array
					CudaErrorCheck(hipFreeArray(this->textureArray));

					// array allocation
					CudaErrorCheck(hipMallocArray(
						&this->textureArray,
						&this->chanelDesc,
						host_texture.GetBitmap().GetWidth(), host_texture.GetBitmap().GetHeight()));
					this->resDesc.res.array.array = this->textureArray;

					// copy host texture data to device array
					CudaErrorCheck(hipMemcpyToArray(
						this->textureArray,
						0, 0, host_texture.GetBitmap().GetMapAddress(),
						host_texture.GetBitmap().GetWidth() * host_texture.GetBitmap().GetHeight() * sizeof(Graphics::Color),
						hipMemcpyKind::hipMemcpyHostToDevice));
				}
				else
				{//--> Everything does match so do asynchronous texture update (TODO)

					//// copy host texture data to device array
					//CudaErrorCheck(hipMemcpyToArray(
					//	this->textureArray,
					//	0, 0, host_texture.GetBitmap().GetMapAddress(),
					//	host_texture.GetBitmap().GetWidth() * host_texture.GetBitmap().GetHeight() * sizeof(Graphics::Color),
					//	hipMemcpyKind::hipMemcpyHostToDevice));

					// copy host texture data to device array
					CudaErrorCheck(cudaMemcpyToArrayAsync(
						this->textureArray,
						0, 0, host_texture.GetBitmap().GetMapAddress(),
						host_texture.GetBitmap().GetWidth() * host_texture.GetBitmap().GetHeight() * sizeof(Graphics::Color),
						hipMemcpyKind::hipMemcpyHostToDevice, mirror_stream));
				}
			}
		}
		// ~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~



		// ~~~~~~~~ [STRUCT] CudaTriangle ~~~~~~~~
		CudaTriangle::CudaTriangle(const Triangle& hostTriangle)
			: v1(nullptr), v2(nullptr), v3(nullptr)
			, t1(nullptr), t2(nullptr), t3(nullptr)
			, n1(nullptr), n2(nullptr), n3(nullptr)
		{
			this->normal = hostTriangle.normal;
			this->color = hostTriangle.color;
		}
		CudaTriangle::~CudaTriangle()
		{
		}
		// ~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~
	}
}