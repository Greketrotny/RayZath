#include "hip/hip_runtime.h"
#include "cuda_render_parts.cuh"
#include "rzexception.h"

#include "hiprand.h"

namespace RayZath
{
	// ~~~~~~~~ [STRUCT] CudaMaterial ~~~~~~~~
	CudaMaterial& CudaMaterial::operator=(const Material& material)
	{
		this->reflectance = material.GetReflectance();
		this->glossiness = material.GetGlossiness();
		this->transmitance = material.GetTransmitance();
		this->ior = material.GetIndexOfRefraction();
		this->emitance = material.GetEmitance();
		return *this;
	}
	// ~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~ 



	// ~~~~~~~~ [STRUCT] RandomNumbers ~~~~~~~~
	HostPinnedMemory RandomNumbers::s_hpm(RandomNumbers::s_count * sizeof(*RandomNumbers::m_unsigned_uniform));

	__host__ RandomNumbers::RandomNumbers()
	{
		/*CudaErrorCheck(hipMalloc(
			(void**)&m_unsigned_uniform, 
			RandomNumbers::s_count * sizeof(*m_unsigned_uniform)));

		CudaErrorCheck(hipMalloc(
			(void**)&m_signed_uniform, 
			RandomNumbers::s_count * sizeof(*m_signed_uniform)));*/
	}
	__host__ RandomNumbers::~RandomNumbers()
	{
	/*	if (m_unsigned_uniform) CudaErrorCheck(hipFree(m_unsigned_uniform));
		m_unsigned_uniform = nullptr;

		if (m_signed_uniform) CudaErrorCheck(hipFree(m_signed_uniform));
		m_signed_uniform = nullptr;*/
	}

	__host__ void RandomNumbers::Reconstruct(hipStream_t& mirror_stream)
	{
		float* hRandNumbers = (float*)s_hpm.GetPointerToMemory();

		// [>] Generate unsigned uniform random floats
		for (unsigned int i = 0; i < s_count; ++i)
			hRandNumbers[i] = (rand() % RAND_MAX) / static_cast<float>(RAND_MAX);

		CudaErrorCheck(hipMemcpyAsync(
			m_unsigned_uniform, hRandNumbers, 
			RandomNumbers::s_count * sizeof(*m_unsigned_uniform), 
			hipMemcpyKind::hipMemcpyHostToDevice, mirror_stream));
		CudaErrorCheck(hipStreamSynchronize(mirror_stream));


		// [>] Generate signed uniform random floats
		for (unsigned int i = 0; i < s_count; ++i)
			hRandNumbers[i] = (((rand() % RAND_MAX) / static_cast<float>(RAND_MAX)) * 2.0f) - 1.0f;

		CudaErrorCheck(hipMemcpyAsync(m_signed_uniform, hRandNumbers, 
			RandomNumbers::s_count * sizeof(*m_signed_uniform), 
			hipMemcpyKind::hipMemcpyHostToDevice, mirror_stream));
		CudaErrorCheck(hipStreamSynchronize(mirror_stream));
	}
	// ~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~



	// ~~~~~~~~ [CLASS] CudaRenderingKErnel ~~~~~~~~
	__host__ CudaKernelData::CudaKernelData()
		: renderIndex(0u)
	{}
	__host__ CudaKernelData::~CudaKernelData()
	{}

	__host__ void CudaKernelData::Reconstruct(
		unsigned int renderIndex,
		hipStream_t& mirrorStream)
	{
		this->renderIndex = renderIndex;
		this->randomNumbers.Reconstruct(mirrorStream);
	}
	// ~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~



	// ~~~~~~~~ [STRUCT] CudaTexture ~~~~~~~~
	hipChannelFormatDesc CudaTexture::chanelDesc = hipCreateChannelDesc<uchar4>();

	CudaTexture::CudaTexture()
		: textureArray(nullptr)
		, textureObject(0)
	{}
	CudaTexture::~CudaTexture()
	{
		if (textureObject) CudaErrorCheck(hipDestroyTextureObject(textureObject));
		if (textureArray)  CudaErrorCheck(hipFreeArray(textureArray));

		this->textureObject = 0;
		this->textureArray = nullptr;
	}

	void CudaTexture::Reconstruct(
		const Texture& host_texture, 
		hipStream_t& mirror_stream)
	{
		if (this->textureArray == nullptr)
		{//--> hostMesh has texture but device equivalent doesn't

			// texture array allocation
			CudaErrorCheck(hipMallocArray(
				&this->textureArray,
				&this->chanelDesc,
				host_texture.GetBitmap().GetWidth(), host_texture.GetBitmap().GetHeight()));

			// copy host texture data to device array
			CudaErrorCheck(hipMemcpyToArray(
				this->textureArray,
				0, 0, host_texture.GetBitmap().GetMapAddress(),
				host_texture.GetBitmap().GetWidth() * host_texture.GetBitmap().GetHeight() * sizeof(Graphics::Color),
				hipMemcpyKind::hipMemcpyHostToDevice));

			// specify resource description			
			memset(&this->resDesc, 0, sizeof(hipResourceDesc));
			this->resDesc.resType = hipResourceType::hipResourceTypeArray;
			this->resDesc.res.array.array = this->textureArray;

			// specify texture object parameters
			memset(&this->textureDesc, 0, (sizeof(hipTextureDesc)));
			this->textureDesc.addressMode[0] = hipTextureAddressMode::hipAddressModeWrap;
			this->textureDesc.addressMode[1] = hipTextureAddressMode::hipAddressModeWrap;
			if (host_texture.GetFilterMode() == Texture::FilterMode::Point)	
				this->textureDesc.filterMode = hipTextureFilterMode::hipFilterModePoint;
			else															
				this->textureDesc.filterMode = hipTextureFilterMode::hipFilterModeLinear;
			this->textureDesc.readMode = hipTextureReadMode::hipReadModeNormalizedFloat;
			this->textureDesc.normalizedCoords = 1;

			// craete texture object
			CudaErrorCheck(hipCreateTextureObject(
				&this->textureObject,
				&this->resDesc,
				&this->textureDesc,
				nullptr));
		}
		else
		{//--> Both hostMesh and deviceMesh have texture

			// get texture array info (width and height)
			hipExtent arrayInfo;
			CudaErrorCheck(hipArrayGetInfo(nullptr, &arrayInfo, nullptr, this->textureArray));

			if (arrayInfo.width * arrayInfo.height != host_texture.GetBitmap().GetWidth() * host_texture.GetBitmap().GetHeight())
			{//--> size of hostMesh texture and CudaMesh texture doesn't match

				// free CudaMesh array
				CudaErrorCheck(hipFreeArray(this->textureArray));

				// array allocation
				CudaErrorCheck(hipMallocArray(
					&this->textureArray,
					&this->chanelDesc,
					host_texture.GetBitmap().GetWidth(), host_texture.GetBitmap().GetHeight()));
				this->resDesc.res.array.array = this->textureArray;

				// copy host texture data to device array
				CudaErrorCheck(hipMemcpyToArray(
					this->textureArray,
					0, 0, host_texture.GetBitmap().GetMapAddress(),
					host_texture.GetBitmap().GetWidth() * host_texture.GetBitmap().GetHeight()* sizeof(Graphics::Color),
					hipMemcpyKind::hipMemcpyHostToDevice));
			}
			else
			{//--> Everything does match so do asynchronous texture update (TODO)

				//// copy host texture data to device array
				//CudaErrorCheck(hipMemcpyToArray(
				//	this->textureArray,
				//	0, 0, host_texture.GetBitmap().GetMapAddress(),
				//	host_texture.GetBitmap().GetWidth() * host_texture.GetBitmap().GetHeight() * sizeof(Graphics::Color),
				//	hipMemcpyKind::hipMemcpyHostToDevice));

				// copy host texture data to device array
				CudaErrorCheck(cudaMemcpyToArrayAsync(
					this->textureArray,
					0, 0, host_texture.GetBitmap().GetMapAddress(),
					host_texture.GetBitmap().GetWidth() * host_texture.GetBitmap().GetHeight() * sizeof(Graphics::Color),
					hipMemcpyKind::hipMemcpyHostToDevice, mirror_stream));
			}
		}
	}
	// ~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~



	// ~~~~~~~~ [STRUCT] CudaTriangle ~~~~~~~~
	__host__ CudaTriangle::CudaTriangle(const Triangle& hostTriangle)
	{
		this->normal = hostTriangle.normal;
		this->color = hostTriangle.color;
	}
	__host__ CudaTriangle::~CudaTriangle()
	{
	}
	// ~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~
}