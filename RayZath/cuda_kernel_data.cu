#include "hip/hip_runtime.h"
#include "cuda_kernel_data.cuh"

#include "engine_parts.hpp"
#include "cuda_exception.hpp"

#include <random>
#include <algorithm>

namespace RayZath::Cuda::Kernel
{
	// ~~~~~~~~ [SRUCT] Seeds ~~~~~~~~
	void Seeds::reconstruct()
	{
		std::random_device rd;
		std::mt19937 gen(rd());
		std::uniform_real_distribution<float> dis(-10.0f, 10.0f);

		for (uint32_t i = 0u; i < s_count; ++i)
			m_seeds[i] = dis(gen);
	}
	// ~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~


	// ~~~~~~~~ RenderConfig::LightSampling ~~~~~~~~
	__host__ RenderConfig::LightSampling& RenderConfig::LightSampling::operator=(
		const RayZath::Engine::LightSampling& light_sampling)
	{
		// at least one to avoid division by zero
		m_spot_light = std::max(light_sampling.spotLight(), uint8_t(1));
		m_direct_light = std::max(light_sampling.directLight(), uint8_t(1));

		return *this;
	}
	// ~~~~~~~~ RenderConfig::Tracing ~~~~~~~~
	__host__ RenderConfig::Tracing& RenderConfig::Tracing::operator=(
		const RayZath::Engine::Tracing& tracing)
	{
		m_max_depth = tracing.maxDepth();

		return *this;
	}

	// ~~~~~~~~ RenderConfig ~~~~~~~~
	RenderConfig& RenderConfig::operator=(const RayZath::Engine::RenderConfig& render_config)
	{
		m_light_sampling = render_config.lightSampling();
		m_tracing = render_config.tracing();

		return *this;
	}


	// ~~~~~~~~ [STRUCT] ConstantKernel ~~~~~~~~
	void ConstantKernel::reconstruct(const RayZath::Engine::RenderConfig& render_config)
	{
		m_seeds.reconstruct();
		m_render_config = render_config;
	}
	// ~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~


	// ~~~~~~~~ [CLASS] GlobalKernel ~~~~~~~~
	GlobalKernel::GlobalKernel()
		: m_render_idx(0u)
	{}

	void GlobalKernel::reconstruct(
		uint32_t render_idx,
		hipStream_t& stream)
	{
		m_render_idx = render_idx;
	}
	// ~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~

	__constant__ ConstantKernel const_kernel[2];

	__host__ void copyConstantKernel(
		const ConstantKernel* hCudaConstantKernel,
		const uint32_t& update_idx,
		hipStream_t& stream)
	{
		RZAssertCoreCUDA(hipMemcpyToSymbolAsync(
			(const void*)const_kernel, hCudaConstantKernel,
			sizeof(ConstantKernel), update_idx * sizeof(ConstantKernel),
			hipMemcpyKind::hipMemcpyHostToDevice, stream));
		RZAssertCoreCUDA(hipStreamSynchronize(stream));
	}
}