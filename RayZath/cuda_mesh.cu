#include "cuda_mesh.cuh"

#include "hip/hip_texture_types.h"
#include "texture_indirect_functions.h"

namespace RayZath
{
	// ~~~~~~~~ [CLASS] CudaMesh ~~~~~~~~
	HostPinnedMemory CudaMesh::hostPinnedMemory(0xFFFF);

	__host__ CudaMesh::CudaMesh()
		: vertices()
		, texcrds()
		, triangles()
		, texture(nullptr)
	{
	}
	__host__ CudaMesh::~CudaMesh()
	{
		// destroy all CudaMesh components
		DestroyTextures();
	}

	__host__ void CudaMesh::Reconstruct(
		Mesh& hMesh, 
		hipStream_t& mirror_stream)
	{
		if (!hMesh.GetStateRegister().IsModified()) return;

		this->vertices.Reconstruct(hMesh.GetMeshStructure().GetVertices(), hostPinnedMemory, mirror_stream);
		this->texcrds.Reconstruct(hMesh.GetMeshStructure().GetTexcrds(), hostPinnedMemory, mirror_stream);
		this->triangles.Reconstruct(
			hMesh.GetMeshStructure().GetTriangles(),
			hMesh.GetMeshStructure().GetVertices(),
			hMesh.GetMeshStructure().GetTexcrds(),
			this->vertices,
			this->texcrds,
			hostPinnedMemory, mirror_stream);

		this->position = hMesh.GetPosition();
		this->rotation = hMesh.GetRotation();
		this->center = hMesh.GetCenter();
		this->scale = hMesh.GetScale();
		this->material = hMesh.GetMaterial();
		this->boundingVolume = hMesh.GetBoundingBox();

		CudaMesh::MirrorTextures(hMesh, &mirror_stream);

		hMesh.GetStateRegister().MakeUnmodified();
	}

	__host__ void CudaMesh::MirrorTextures(const Mesh& hostMesh, hipStream_t* mirrorStream)
	{
		if (hostMesh.GetTexture() != nullptr)
		{
			if (this->texture == nullptr)
			{
				// host created texture so device must too
				CudaTexture* hostCudaTexture = (CudaTexture*)malloc(sizeof(CudaTexture));
				new (hostCudaTexture) CudaTexture();

				hostCudaTexture->Reconstruct(*hostMesh.GetTexture(), *mirrorStream);

				CudaErrorCheck(hipMalloc(&this->texture, sizeof(CudaTexture)));
				CudaErrorCheck(hipMemcpy(
					this->texture, hostCudaTexture, 
					sizeof(*this->texture), 
					hipMemcpyKind::hipMemcpyHostToDevice));
				free(hostCudaTexture);
			}
			else
			{
				//if (!hostMesh.UpdateRequests.GetUpdateRequestState(Mesh::MeshUpdateRequestTexture))
				//	return;

				// on both sides is texture - only mirror
				CudaTexture* hostCudaTexture = (CudaTexture*)this->hostPinnedMemory.GetPointerToMemory();
				if (this->hostPinnedMemory.GetSize() < sizeof(CudaTexture)) return;	// TODO: throw an exception (to few host-pinned memory)

				CudaErrorCheck(hipMemcpyAsync(
					hostCudaTexture, this->texture, 
					sizeof(CudaTexture), 
					hipMemcpyKind::hipMemcpyDeviceToHost, *mirrorStream));
				CudaErrorCheck(hipStreamSynchronize(*mirrorStream));

				hostCudaTexture->Reconstruct(*hostMesh.GetTexture(), *mirrorStream);

				CudaErrorCheck(hipMemcpy(
					this->texture, hostCudaTexture, 
					sizeof(CudaTexture), 
					hipMemcpyKind::hipMemcpyHostToDevice));
				CudaErrorCheck(hipStreamSynchronize(*mirrorStream));
			}
		}
		else
		{
			if (this->texture != nullptr)
			{
				// host has unloaded texture so destroy texture on device
				CudaTexture* hostCudaTexture = (CudaTexture*)malloc(sizeof(CudaTexture));
				CudaErrorCheck(hipMemcpy(
					hostCudaTexture, this->texture, 
					sizeof(CudaTexture), 
					hipMemcpyKind::hipMemcpyDeviceToHost));

				hostCudaTexture->~CudaTexture();

				CudaErrorCheck(hipFree(this->texture));
				this->texture = nullptr;
				free(hostCudaTexture);
			}
		}
	}
	__host__ void CudaMesh::DestroyTextures()
	{
		if (this->texture)
		{
			CudaTexture* hostCudaTexture = (CudaTexture*)malloc(sizeof(CudaTexture));
			CudaErrorCheck(hipMemcpy(
				hostCudaTexture, this->texture, 
				sizeof(CudaTexture), 
				hipMemcpyKind::hipMemcpyDeviceToHost));

			hostCudaTexture->~CudaTexture();

			free(hostCudaTexture);

			CudaErrorCheck(hipFree(this->texture));
			this->texture = nullptr;
		}
	}
	// ~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~
}