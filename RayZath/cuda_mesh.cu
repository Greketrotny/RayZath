#include "cuda_mesh.cuh"

#include "cuda_world.cuh"

namespace RayZath::Cuda
{
	// ~~~~~~~~ [CLASS] MeshStructure ~~~~~~~~
	HostPinnedMemory MeshStructure::m_hpm_trs(sizeof(Triangle) * 64u);
	HostPinnedMemory MeshStructure::m_hpm_nodes(
		sizeof(TreeNode)*
		MeshStructure::sm_max_bvh_depth*
		MeshStructure::sm_max_child_count);

	__host__ MeshStructure::MeshStructure()
		: mp_triangles(nullptr)
		, m_triangle_capacity(0u)
		, m_triangle_count(0u)
		, mp_nodes(nullptr)
		, m_node_capacity(0u)
		, m_node_count(0u)
	{}
	__host__ MeshStructure::~MeshStructure()
	{
		if (mp_triangles)
			CudaErrorCheck(hipFree(mp_triangles));
		if (mp_nodes)
			CudaErrorCheck(hipFree(mp_nodes));
	}


	__host__ void MeshStructure::Reconstruct(
		const World& hCudaWorld,
		const RayZath::Engine::Handle<RayZath::Engine::MeshStructure>& hMeshStructure,
		hipStream_t& mirror_stream)
	{
		if (!hMeshStructure->GetStateRegister().IsModified()) return;

		const uint32_t tree_size = hMeshStructure->GetTriangles().GetBVH().GetTreeSize();
		if (tree_size == 0u || hMeshStructure->GetTriangles().GetCount() == 0u)
		{	// tree is empty so release all content

			if (mp_nodes) CudaErrorCheck(hipFree(mp_nodes));
			mp_nodes = nullptr;
			m_node_capacity = 0u;
			m_node_count = 0u;

			if (mp_triangles) CudaErrorCheck(hipFree(mp_triangles));
			mp_triangles = nullptr;
			m_triangle_capacity = 0u;
			m_triangle_count = 0u;

			hMeshStructure->GetStateRegister().MakeUnmodified();
			return;
		}

		// allocate memory for tree nodes and triangles
		if (tree_size != m_node_capacity)
		{
			if (mp_nodes) CudaErrorCheck(hipFree(mp_nodes));
			m_node_capacity = hMeshStructure->GetTriangles().GetBVH().GetTreeSize();
			CudaErrorCheck(hipMalloc((void**)&mp_nodes, sizeof(*mp_nodes) * m_node_capacity));
		}
		const uint32_t h_capacity = hMeshStructure->GetTriangles().GetCapacity();
		if (m_triangle_capacity != h_capacity)
		{
			if (mp_triangles) CudaErrorCheck(hipFree(mp_triangles));
			m_triangle_capacity = h_capacity;
			CudaErrorCheck(hipMalloc((void**)&mp_triangles, sizeof(*mp_triangles) * m_triangle_capacity));
		}

		m_node_count = 0u;
		m_triangle_count = 0u;

		// reserve hpm for triangle chunks
		const uint32_t trs_chunk_size = uint32_t(m_hpm_trs.GetSize() / sizeof(*mp_triangles));
		Triangle* const hCudaTriangles = (Triangle*)(m_hpm_trs.GetPointerToMemory());
		RZAssert(trs_chunk_size > 16u, "Too few hpm for triangle reconstruction");
		uint32_t trs_in_chunk = 0u;


		auto CopyTrsChunk = [&]() -> void
		{
			if (trs_in_chunk == 0u) return;

			RZAssert(m_triangle_count <= m_triangle_capacity, "qwer");

			CudaErrorCheck(hipMemcpyAsync(
				mp_triangles + m_triangle_count - trs_in_chunk,
				hCudaTriangles,
				sizeof(*mp_triangles) * trs_in_chunk,
				hipMemcpyKind::hipMemcpyHostToDevice, mirror_stream));
			CudaErrorCheck(hipStreamSynchronize(mirror_stream));
			trs_in_chunk = 0u;
		};
		auto AddTriangle = [&](const RayZath::Engine::Triangle& hTriangle) -> void
		{
			if (trs_in_chunk >= trs_chunk_size)
				CopyTrsChunk();

			Triangle& hCudaTriangle = *(hCudaTriangles + trs_in_chunk);
			new (&hCudaTriangle) Triangle(hTriangle);

			if (hTriangle.AreVertsValid())
			{
				hCudaTriangle.SetVertices(
					vec3f(hMeshStructure->GetVertices()[hTriangle.vertices[0]]),
					vec3f(hMeshStructure->GetVertices()[hTriangle.vertices[1]]),
					vec3f(hMeshStructure->GetVertices()[hTriangle.vertices[2]]));
			}
			else
			{
				hCudaTriangle.SetVertices(
					vec3f(0.0f, 0.0f, 0.0f), vec3f(1.0f, 0.0f, 0.0f), vec3f(0.0f, 1.0f, 0.0f));
			}
			if (hTriangle.AreTexcrdsValid())
			{
				hCudaTriangle.SetTexcrds(
					vec2f(hMeshStructure->GetTexcrds()[hTriangle.texcrds[0]]),
					vec2f(hMeshStructure->GetTexcrds()[hTriangle.texcrds[1]]),
					vec2f(hMeshStructure->GetTexcrds()[hTriangle.texcrds[2]]));
			}
			else
			{
				hCudaTriangle.SetTexcrds(vec2f(), vec2f(), vec2f());
			}
			if (hTriangle.AreNormalsValid())
			{
				hCudaTriangle.SetNormals(
					vec3f(hMeshStructure->GetNormals()[hTriangle.normals[0]]),
					vec3f(hMeshStructure->GetNormals()[hTriangle.normals[1]]),
					vec3f(hMeshStructure->GetNormals()[hTriangle.normals[2]]));
			}
			else
			{
				hCudaTriangle.SetNormals(
					vec3f(hTriangle.normal),
					vec3f(hTriangle.normal),
					vec3f(hTriangle.normal));
			}

			trs_in_chunk++;
			m_triangle_count++;
		};

		auto BuildNode = [&](
			const auto& BuildNodeFunc,
			TreeNode& hCudaNode,
			TreeNode* const hCudaEndNode,
			const RayZath::Engine::ComponentTreeNode<RayZath::Engine::Triangle>& hNode) -> void
		{
			if (hNode.IsLeaf())
			{
				const uint32_t leaf_size = hNode.GetObjectCount();
				hCudaNode.SetRange(m_triangle_count, m_triangle_count + leaf_size);
				for (uint32_t i = 0u; i < leaf_size; i++)
				{
					AddTriangle(*hNode.GetObject(i));
				}
			}
			else
			{
				const uint32_t child_count = hNode.GetChildCount();
				hCudaNode.SetRange(m_node_count, m_node_count + child_count);
				const uint32_t child_begin_idx = m_node_count;
				m_node_count += child_count;
				for (uint32_t i = 0u, c = 0u; i < 8u; i++)
				{
					const auto* const hChildNode =
						hNode.GetChild(i);

					if (hChildNode)
					{
						TreeNode* const hCudaChildNode = hCudaEndNode + c++;
						new (hCudaChildNode) TreeNode(
							hChildNode->GetBoundingBox(),
							hChildNode->IsLeaf());
						BuildNodeFunc(BuildNodeFunc, *hCudaChildNode, hCudaEndNode + child_count, *hChildNode);
					}
				}

				CudaErrorCheck(hipMemcpyAsync(
					mp_nodes + child_begin_idx,
					hCudaEndNode,
					sizeof(*mp_nodes) * child_count,
					hipMemcpyKind::hipMemcpyHostToDevice,
					mirror_stream));
				CudaErrorCheck(hipStreamSynchronize(mirror_stream));
			}
		};

		// get hpm for root node
		const auto& hRootNode = hMeshStructure->GetTriangles().GetBVH().GetRootNode();
		TreeNode* hCudaRootNode = (TreeNode*)m_hpm_nodes.GetPointerToMemory();
		m_node_count++;

		// build root node
		new (hCudaRootNode) TreeNode(hRootNode.GetBoundingBox(), hRootNode.IsLeaf());
		BuildNode(BuildNode, *hCudaRootNode, hCudaRootNode + 1u, hRootNode);

		// copy root node to device
		CudaErrorCheck(hipMemcpyAsync(
			mp_nodes,
			hCudaRootNode,
			sizeof(*hCudaRootNode),
			hipMemcpyKind::hipMemcpyHostToDevice, mirror_stream));
		CudaErrorCheck(hipStreamSynchronize(mirror_stream));

		// copy last possibly not  full chunk of triangles to device
		CopyTrsChunk();

		hMeshStructure->GetStateRegister().MakeUnmodified();
	}
	// ~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~



	// ~~~~~~~~ [CLASS] Mesh ~~~~~~~~
	__host__ Mesh::Mesh()
		: mesh_structure(nullptr)
		, materials{}
	{}

	__host__ void Mesh::Reconstruct(
		const World& hCudaWorld,
		const Engine::Handle<Engine::Mesh>& hMesh,
		hipStream_t& mirror_stream)
	{
		if (!hMesh->GetStateRegister().IsModified()) return;

		transformation = hMesh->GetTransformation();
		bounding_box = hMesh->GetBoundingBox();

		// mesh structure
		auto& hStructure = hMesh->GetStructure();
		if (hStructure)
		{
			if (hStructure.GetAccessor()->GetIdx() < hCudaWorld.mesh_structures.GetCount())
			{
				this->mesh_structure =
					hCudaWorld.mesh_structures.GetStorageAddress() +
					hStructure.GetAccessor()->GetIdx();
			}
			else this->mesh_structure = nullptr;
		}
		else this->mesh_structure = nullptr;

		// materials
		for (uint32_t i = 0u; i < Engine::Mesh::GetMaterialCapacity(); i++)
		{
			auto& hMaterial = hMesh->GetMaterial(i);
			if (hMaterial)
			{
				if (hMaterial.GetAccessor()->GetIdx() < hCudaWorld.materials.GetCount())
				{
					materials[i] =
						hCudaWorld.materials.GetStorageAddress() +
						hMaterial.GetAccessor()->GetIdx();
				}
				else materials[i] = hCudaWorld.default_material;
			}
			else materials[i] = hCudaWorld.default_material;
		}


		hMesh->GetStateRegister().MakeUnmodified();
	}
}