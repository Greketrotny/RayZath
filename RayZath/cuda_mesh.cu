#include "cuda_mesh.cuh"

#include "cuda_exception.hpp"
#include "cuda_world.cuh"

namespace RayZath::Cuda
{
	// ~~~~~~~~ [CLASS] MeshStructure ~~~~~~~~
	HostPinnedMemory MeshStructure::m_hpm_trs(sizeof(Triangle) * 1024u);
	HostPinnedMemory MeshStructure::m_hpm_nodes(sizeof(TreeNode) * 1024u);

	__host__ MeshStructure::~MeshStructure()
	{
		if (mp_triangles) RZAssertCoreCUDA(hipFree(mp_triangles));
		if (mp_nodes) RZAssertCoreCUDA(hipFree(mp_nodes));
	}

	__host__ void MeshStructure::reconstruct(
		[[maybe_unused]] const World& hCudaWorld,
		const RayZath::Engine::Handle<RayZath::Engine::MeshStructure>& hMeshStructure,
		hipStream_t& mirror_stream)
	{
		if (!hMeshStructure->stateRegister().IsModified()) return;

		const uint32_t tree_size = hMeshStructure->triangles().getBVH().GetRootNode().treeSize();
		if (tree_size == 0u || hMeshStructure->triangles().count() == 0u)
		{	// tree is empty so release all content

			if (mp_nodes) RZAssertCoreCUDA(hipFree(mp_nodes));
			mp_nodes = nullptr;
			m_node_capacity = 0u;
			m_node_count = 0u;

			if (mp_triangles) RZAssertCoreCUDA(hipFree(mp_triangles));
			mp_triangles = nullptr;
			m_triangle_capacity = 0u;
			m_triangle_count = 0u;

			hMeshStructure->stateRegister().MakeUnmodified();
			return;
		}

		// allocate memory for tree nodes and triangles
		if (tree_size != m_node_capacity)
		{
			if (mp_nodes) RZAssertCoreCUDA(hipFree(mp_nodes));
			m_node_capacity = tree_size;
			RZAssertCoreCUDA(hipMalloc((void**)&mp_nodes, sizeof(*mp_nodes) * m_node_capacity));
		}
		const uint32_t h_capacity = hMeshStructure->triangles().capacity();
		if (m_triangle_capacity != h_capacity)
		{
			if (mp_triangles) RZAssertCoreCUDA(hipFree(mp_triangles));
			m_triangle_capacity = h_capacity;
			RZAssertCoreCUDA(hipMalloc((void**)&mp_triangles, sizeof(*mp_triangles) * m_triangle_capacity));
		}

		m_node_count = 0u;
		m_triangle_count = 0u;

		// reserve hpm for triangle chunks
		const uint32_t trs_chunk_size = uint32_t(m_hpm_trs.size() / sizeof(*mp_triangles));
		Triangle* const hCudaTriangles = (Triangle*)(m_hpm_trs.GetPointerToMemory());
		RZAssert(trs_chunk_size > 16u, "Too few hpm for triangle reconstruction");
		uint32_t trs_in_chunk = 0u;

		const uint32_t nodes_chunk_size = uint32_t(m_hpm_nodes.size() / sizeof(*mp_nodes));
		TreeNode* const h_cuda_nodes = (TreeNode*)(m_hpm_nodes.GetPointerToMemory());
		RZAssert(nodes_chunk_size > 16u, "Too few hpm for tree node reconstruction");
		uint32_t nodes_in_chunk = 0u;


		auto CopyTrianglesChunk = [&]() -> void
		{
			if (trs_in_chunk == 0u) return;

			RZAssert(m_triangle_count <= m_triangle_capacity, "triangle count exceeded capacity");

			RZAssertCoreCUDA(hipMemcpyAsync(
				mp_triangles + m_triangle_count - trs_in_chunk,
				hCudaTriangles,
				sizeof(*mp_triangles) * trs_in_chunk,
				hipMemcpyKind::hipMemcpyHostToDevice, mirror_stream));
			RZAssertCoreCUDA(hipStreamSynchronize(mirror_stream));
			trs_in_chunk = 0u;
		};
		auto CopyNodesChunk = [&]() -> void
		{
			if (nodes_in_chunk == 0u) return;
			RZAssert(m_node_count <= m_node_capacity, "node count exceeded capacity");

			RZAssertCoreCUDA(hipMemcpyAsync(
				mp_nodes + m_node_count - nodes_in_chunk,
				h_cuda_nodes,
				sizeof(*mp_nodes) * nodes_in_chunk,
				hipMemcpyKind::hipMemcpyHostToDevice, mirror_stream));
			RZAssertCoreCUDA(hipStreamSynchronize(mirror_stream));
			nodes_in_chunk = 0u;
		};

		auto AddTriangle = [&](const RayZath::Engine::Triangle& hTriangle) -> void
		{
			if (trs_in_chunk >= trs_chunk_size)
				CopyTrianglesChunk();

			Triangle& hCudaTriangle = *(hCudaTriangles + trs_in_chunk);
			new (&hCudaTriangle) Triangle(hTriangle);

			if (hTriangle.areVertsValid())
			{
				hCudaTriangle.setVertices(
					vec3f(hMeshStructure->vertices()[hTriangle.vertices[0]]),
					vec3f(hMeshStructure->vertices()[hTriangle.vertices[1]]),
					vec3f(hMeshStructure->vertices()[hTriangle.vertices[2]]));
			}
			else
			{
				hCudaTriangle.setVertices(
					vec3f(0.0f, 0.0f, 0.0f), vec3f(1.0f, 0.0f, 0.0f), vec3f(0.0f, 1.0f, 0.0f));
			}
			if (hTriangle.areTexcrdsValid())
			{
				hCudaTriangle.setTexcrds(
					vec2f(hMeshStructure->texcrds()[hTriangle.texcrds[0]]),
					vec2f(hMeshStructure->texcrds()[hTriangle.texcrds[1]]),
					vec2f(hMeshStructure->texcrds()[hTriangle.texcrds[2]]));
			}
			else
			{
				hCudaTriangle.setTexcrds(vec2f(), vec2f(), vec2f());
			}
			if (hTriangle.areNormalsValid())
			{
				hCudaTriangle.setNormals(
					vec3f(hMeshStructure->normals()[hTriangle.normals[0]]),
					vec3f(hMeshStructure->normals()[hTriangle.normals[1]]),
					vec3f(hMeshStructure->normals()[hTriangle.normals[2]]));
			}
			else
			{
				hCudaTriangle.setNormals(
					vec3f(hTriangle.normal),
					vec3f(hTriangle.normal),
					vec3f(hTriangle.normal));
			}

			trs_in_chunk++;
			m_triangle_count++;
		};
		auto AddNode = [&](TreeNode&& node) -> void
		{
			if (nodes_in_chunk >= nodes_chunk_size)
				CopyNodesChunk();

			TreeNode& h_cuda_node = *(h_cuda_nodes + nodes_in_chunk);
			h_cuda_node = std::move(node);

			nodes_in_chunk++;
			m_node_count++;
		};

		auto BuildChildren = [&](
			const auto& BuildChildrenFunc,
			const RayZath::Engine::ComponentTreeNode<RayZath::Engine::Triangle>& hNode) -> void
		{
			RZAssert(!hNode.isLeaf(), "node had no children");

			const auto& child1 = hNode.children()->first;
			const auto first_subtree_size = child1.treeSize() - 1;
			if (child1.isLeaf())
			{
				AddNode(TreeNode(
					child1.boundingBox(), 0,
					m_triangle_count, uint32_t(child1.objects().size())));
				for (const auto* object : child1.objects())
					if (object) { AddTriangle(*object); }
			}
			else
			{
				AddNode(TreeNode(
					child1.boundingBox(), uint32_t(child1.children()->type),
					m_node_count + 2, 0));
			}

			const auto& child2 = hNode.children()->second;
			if (child2.isLeaf())
			{
				AddNode(TreeNode(
					child2.boundingBox(), 0,
					m_triangle_count, uint32_t(child2.objects().size())));
				for (const auto* object : child2.objects())
					if (object) { AddTriangle(*object); }
			}
			else
			{
				AddNode(TreeNode(
					child2.boundingBox(), uint32_t(child2.children()->type),
					m_node_count + first_subtree_size + 1, 0));
			}

			if (!child1.isLeaf()) BuildChildrenFunc(BuildChildrenFunc, child1);
			if (!child2.isLeaf()) BuildChildrenFunc(BuildChildrenFunc, child2);
		};

		const auto& hRoot = hMeshStructure->triangles().getBVH().GetRootNode();
		if (hRoot.isLeaf())
		{
			AddNode(TreeNode(
				hRoot.boundingBox(), 0,
				m_triangle_count, uint32_t(hRoot.objects().size())));
			for (const auto* object : hRoot.objects())
				if (object) { AddTriangle(*object); }
		}
		else
		{
			AddNode(TreeNode(
				hRoot.boundingBox(), uint32_t(hRoot.children()->type),
				m_node_count + 1, 0));

			BuildChildren(BuildChildren, hRoot);
		}

		CopyTrianglesChunk();
		CopyNodesChunk();

		hMeshStructure->stateRegister().MakeUnmodified();
	}
	// ~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~



	// ~~~~~~~~ [CLASS] Mesh ~~~~~~~~
	__host__ Instance::Instance()
		: materials{}
		, m_mesh_idx{}
	{}

	__host__ void Instance::reconstruct(
		const World& hCudaWorld,
		const Engine::Handle<Engine::Instance>& hMesh,
		[[maybe_unused]] hipStream_t& mirror_stream)
	{
		if (!hMesh || !hMesh->stateRegister().IsModified()) return;

		transformation = hMesh->transformationInGroup();
		bounding_box = hMesh->boundingBox();

		m_mesh_idx = hMesh.accessor()->idx();

		// mesh structure
		auto& hStructure = hMesh->meshStructure();
		if (hStructure)
		{
			if (hStructure.accessor()->idx() < hCudaWorld.mesh_structures.count())
			{
				this->mesh_structure =
					hCudaWorld.mesh_structures.storageAddress() +
					hStructure.accessor()->idx();
			}
			else this->mesh_structure = nullptr;
		}
		else this->mesh_structure = nullptr;

		// materials
		for (uint32_t i = 0u; i < Engine::Instance::materialCapacity(); i++)
		{
			auto& hMaterial = hMesh->material(i);
			if (hMaterial)
			{
				if (hMaterial.accessor()->idx() < hCudaWorld.materials.count())
				{
					materials[i] =
						hCudaWorld.materials.storageAddress() +
						hMaterial.accessor()->idx();
				}
				else materials[i] = hCudaWorld.default_material;
			}
			else materials[i] = hCudaWorld.default_material;
		}


		hMesh->stateRegister().MakeUnmodified();
	}
}