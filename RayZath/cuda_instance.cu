#include "cuda_instance.cuh"

#include "cuda_exception.hpp"
#include "cuda_world.cuh"

namespace RayZath::Cuda
{
	HostPinnedMemory Mesh::m_hpm_trs(sizeof(Triangle) * 1024u);
	HostPinnedMemory Mesh::m_hpm_nodes(sizeof(TreeNode) * 1024u);

	__host__ Mesh::~Mesh()
	{
		if (mp_triangles) RZAssertCoreCUDA(hipFree(mp_triangles));
		if (mp_nodes) RZAssertCoreCUDA(hipFree(mp_nodes));
	}

	__host__ void Mesh::reconstruct(
		[[maybe_unused]] const World& hCudaWorld,
		const RayZath::Engine::Handle<RayZath::Engine::Mesh>& hMesh,
		hipStream_t& mirror_stream)
	{
		if (!hMesh->stateRegister().IsModified()) return;

		const uint32_t tree_size = hMesh->triangles().getBVH().rootNode().treeSize();
		if (tree_size == 0u || hMesh->triangles().count() == 0u)
		{	// tree is empty so release all content

			if (mp_nodes) RZAssertCoreCUDA(hipFree(mp_nodes));
			mp_nodes = nullptr;
			m_node_capacity = 0u;
			m_node_count = 0u;

			if (mp_triangles) RZAssertCoreCUDA(hipFree(mp_triangles));
			mp_triangles = nullptr;
			m_triangle_capacity = 0u;
			m_triangle_count = 0u;

			hMesh->stateRegister().MakeUnmodified();
			return;
		}

		// allocate memory for tree nodes and triangles
		if (tree_size != m_node_capacity)
		{
			if (mp_nodes) RZAssertCoreCUDA(hipFree(mp_nodes));
			m_node_capacity = tree_size;
			RZAssertCoreCUDA(hipMalloc((void**)&mp_nodes, sizeof(*mp_nodes) * m_node_capacity));
		}
		const uint32_t h_capacity = hMesh->triangles().capacity();
		if (m_triangle_capacity != h_capacity)
		{
			if (mp_triangles) RZAssertCoreCUDA(hipFree(mp_triangles));
			m_triangle_capacity = h_capacity;
			RZAssertCoreCUDA(hipMalloc((void**)&mp_triangles, sizeof(*mp_triangles) * m_triangle_capacity));
		}

		m_node_count = 0u;
		m_triangle_count = 0u;

		// reserve hpm for triangle chunks
		const uint32_t trs_chunk_size = uint32_t(m_hpm_trs.size() / sizeof(*mp_triangles));
		Triangle* const hCudaTriangles = (Triangle*)(m_hpm_trs.GetPointerToMemory());
		RZAssert(trs_chunk_size > 16u, "Too few hpm for triangle reconstruction");
		uint32_t trs_in_chunk = 0u;

		const uint32_t nodes_chunk_size = uint32_t(m_hpm_nodes.size() / sizeof(*mp_nodes));
		TreeNode* const h_cuda_nodes = (TreeNode*)(m_hpm_nodes.GetPointerToMemory());
		RZAssert(nodes_chunk_size > 16u, "Too few hpm for tree node reconstruction");
		uint32_t nodes_in_chunk = 0u;


		auto CopyTrianglesChunk = [&]() -> void
		{
			if (trs_in_chunk == 0u) return;

			RZAssert(m_triangle_count <= m_triangle_capacity, "triangle count exceeded capacity");

			RZAssertCoreCUDA(hipMemcpyAsync(
				mp_triangles + m_triangle_count - trs_in_chunk,
				hCudaTriangles,
				sizeof(*mp_triangles) * trs_in_chunk,
				hipMemcpyKind::hipMemcpyHostToDevice, mirror_stream));
			RZAssertCoreCUDA(hipStreamSynchronize(mirror_stream));
			trs_in_chunk = 0u;
		};
		auto CopyNodesChunk = [&]() -> void
		{
			if (nodes_in_chunk == 0u) return;
			RZAssert(m_node_count <= m_node_capacity, "node count exceeded capacity");

			RZAssertCoreCUDA(hipMemcpyAsync(
				mp_nodes + m_node_count - nodes_in_chunk,
				h_cuda_nodes,
				sizeof(*mp_nodes) * nodes_in_chunk,
				hipMemcpyKind::hipMemcpyHostToDevice, mirror_stream));
			RZAssertCoreCUDA(hipStreamSynchronize(mirror_stream));
			nodes_in_chunk = 0u;
		};

		auto AddTriangle = [&](const RayZath::Engine::Triangle& hTriangle) -> void
		{
			if (trs_in_chunk >= trs_chunk_size)
				CopyTrianglesChunk();

			Triangle& hCudaTriangle = *(hCudaTriangles + trs_in_chunk);
			new (&hCudaTriangle) Triangle(hTriangle);

			if (hTriangle.areVertsValid())
			{
				hCudaTriangle.setVertices(
					vec3f(hMesh->vertices()[hTriangle.vertices[0]]),
					vec3f(hMesh->vertices()[hTriangle.vertices[1]]),
					vec3f(hMesh->vertices()[hTriangle.vertices[2]]));
			}
			else
			{
				hCudaTriangle.setVertices(
					vec3f(0.0f, 0.0f, 0.0f), vec3f(1.0f, 0.0f, 0.0f), vec3f(0.0f, 1.0f, 0.0f));
			}
			if (hTriangle.areTexcrdsValid())
			{
				hCudaTriangle.setTexcrds(
					vec2f(hMesh->texcrds()[hTriangle.texcrds[0]]),
					vec2f(hMesh->texcrds()[hTriangle.texcrds[1]]),
					vec2f(hMesh->texcrds()[hTriangle.texcrds[2]]));
			}
			else
			{
				hCudaTriangle.setTexcrds(vec2f(0.0f, 0.0f), vec2f(0.0f, 1.0f), vec2f(1.0f, 0.0f));
			}
			if (hTriangle.areNormalsValid())
			{
				hCudaTriangle.setNormals(
					vec3f(hMesh->normals()[hTriangle.normals[0]]),
					vec3f(hMesh->normals()[hTriangle.normals[1]]),
					vec3f(hMesh->normals()[hTriangle.normals[2]]));
			}
			else
			{
				hCudaTriangle.setNormals(
					vec3f(hTriangle.normal),
					vec3f(hTriangle.normal),
					vec3f(hTriangle.normal));
			}

			trs_in_chunk++;
			m_triangle_count++;
		};
		auto AddNode = [&](TreeNode&& node) -> void
		{
			if (nodes_in_chunk >= nodes_chunk_size)
				CopyNodesChunk();

			TreeNode& h_cuda_node = *(h_cuda_nodes + nodes_in_chunk);
			h_cuda_node = std::move(node);

			nodes_in_chunk++;
			m_node_count++;
		};

		auto BuildChildren = [&](
			const auto& BuildChildrenFunc,
			const RayZath::Engine::ComponentTreeNode<RayZath::Engine::Triangle>& hNode) -> void
		{
			RZAssert(!hNode.isLeaf(), "node had no children");

			const auto& child1 = hNode.children()->first;
			const auto first_subtree_size = child1.treeSize() - 1;
			if (child1.isLeaf())
			{
				AddNode(TreeNode(
					child1.boundingBox(), 0,
					m_triangle_count, uint32_t(child1.objects().size())));
				for (const auto* object : child1.objects())
					if (object) { AddTriangle(*object); }
			}
			else
			{
				AddNode(TreeNode(
					child1.boundingBox(), uint32_t(child1.children()->type),
					m_node_count + 2, 0));
			}

			const auto& child2 = hNode.children()->second;
			if (child2.isLeaf())
			{
				AddNode(TreeNode(
					child2.boundingBox(), 0,
					m_triangle_count, uint32_t(child2.objects().size())));
				for (const auto* object : child2.objects())
					if (object) { AddTriangle(*object); }
			}
			else
			{
				AddNode(TreeNode(
					child2.boundingBox(), uint32_t(child2.children()->type),
					m_node_count + first_subtree_size + 1, 0));
			}

			if (!child1.isLeaf()) BuildChildrenFunc(BuildChildrenFunc, child1);
			if (!child2.isLeaf()) BuildChildrenFunc(BuildChildrenFunc, child2);
		};

		const auto& hRoot = hMesh->triangles().getBVH().rootNode();
		if (hRoot.isLeaf())
		{
			AddNode(TreeNode(
				hRoot.boundingBox(), 0,
				m_triangle_count, uint32_t(hRoot.objects().size())));
			for (const auto* object : hRoot.objects())
				if (object) { AddTriangle(*object); }
		}
		else
		{
			AddNode(TreeNode(
				hRoot.boundingBox(), uint32_t(hRoot.children()->type),
				m_node_count + 1, 0));

			BuildChildren(BuildChildren, hRoot);
		}

		CopyTrianglesChunk();
		CopyNodesChunk();

		hMesh->stateRegister().MakeUnmodified();
	}
	// ~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~



	// ~~~~~~~~ [CLASS] Mesh ~~~~~~~~
	__host__ Instance::Instance()
		: materials{}
		, m_instance_idx{}
	{}

	__host__ void Instance::reconstruct(
		const World& hCudaWorld,
		const Engine::Handle<Engine::Instance>& hInstance,
		[[maybe_unused]] hipStream_t& mirror_stream)
	{
		if (!hInstance || !hInstance->stateRegister().IsModified()) return;

		transformation = hInstance->transformationInGroup();
		bounding_box = hInstance->boundingBox();

		m_instance_idx = hInstance.accessor()->idx();

		// mesh
		auto& hMesh = hInstance->mesh();
		if (hMesh)
		{
			if (hMesh.accessor()->idx() < hCudaWorld.meshes.count())
			{
				this->mesh =
					hCudaWorld.meshes.storageAddress() +
					hMesh.accessor()->idx();
			}
			else this->mesh = nullptr;
		}
		else this->mesh = nullptr;

		// materials
		for (uint32_t i = 0u; i < Engine::Instance::materialCapacity(); i++)
		{
			auto& hMaterial = hInstance->material(i);
			if (hMaterial)
			{
				if (hMaterial.accessor()->idx() < hCudaWorld.materials.count())
				{
					materials[i] =
						hCudaWorld.materials.storageAddress() +
						hMaterial.accessor()->idx();
				}
				else materials[i] = hCudaWorld.default_material;
			}
			else materials[i] = hCudaWorld.default_material;
		}


		hInstance->stateRegister().MakeUnmodified();
	}
}