#include "cuda_world.cuh"

namespace RayZath
{
	namespace CudaEngine
	{
		HostPinnedMemory CudaWorld::m_hpm(0xFFFF);

		void CudaWorld::Reconstruct(
			World& hWorld,
			hipStream_t& mirror_stream)
		{
			if (!hWorld.GetStateRegister().IsModified()) return;

			textures.Reconstruct(*this, hWorld.GetTextures(), m_hpm, mirror_stream);
			materials.Reconstruct(*this, hWorld.GetMaterials(), m_hpm, mirror_stream);
			mesh_structures.Reconstruct(*this, hWorld.GetMeshStructures(), m_hpm, mirror_stream);

			cameras.Reconstruct(*this, hWorld.GetCameras(), m_hpm, mirror_stream);

			pointLights.Reconstruct(*this, hWorld.GetPointLights(), m_hpm, mirror_stream);
			spotLights.Reconstruct(*this, hWorld.GetSpotLights(), m_hpm, mirror_stream);
			directLights.Reconstruct(*this, hWorld.GetDirectLights(), m_hpm, mirror_stream);

			meshes.Reconstruct(*this, hWorld.GetMeshes(), m_hpm, mirror_stream);
			spheres.Reconstruct(*this, hWorld.GetSpheres(), m_hpm, mirror_stream);

			material = hWorld.GetMaterial();

			hWorld.GetStateRegister().MakeUnmodified();
		}
	}
}