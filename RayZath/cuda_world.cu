#include "cuda_world.cuh"

namespace RayZath
{
	namespace CudaEngine
	{
		HostPinnedMemory CudaWorld::m_hpm(0x10000);

		CudaWorld::CudaWorld() 
		{
			// default material
			CudaMaterial* hCudaMaterial = (CudaMaterial*)m_hpm.GetPointerToMemory();
			new (hCudaMaterial) CudaMaterial(
				CudaColor<float>(1.0f, 1.0f, 1.0f, 1.0f),
				0.0f, 0.0f, 0.0f, 1.0f, 0.0f, 0.0f);

			CudaErrorCheck(hipMalloc(&default_material, sizeof(*default_material)));
			CudaErrorCheck(hipMemcpy(
				default_material, hCudaMaterial, 
				sizeof(*default_material), 
				hipMemcpyKind::hipMemcpyHostToDevice));
		}
		CudaWorld::~CudaWorld()
		{
			// default material
			if (default_material) CudaErrorCheck(hipFree(default_material));
			default_material = nullptr;
		}

		void CudaWorld::Reconstruct(
			World& hWorld,
			hipStream_t& mirror_stream)
		{
			if (!hWorld.GetStateRegister().IsModified()) return;

			textures.Reconstruct(*this, hWorld.GetTextures(), m_hpm, mirror_stream);
			materials.Reconstruct(*this, hWorld.GetMaterials(), m_hpm, mirror_stream);
			mesh_structures.Reconstruct(*this, hWorld.GetMeshStructures(), m_hpm, mirror_stream);

			cameras.Reconstruct(*this, hWorld.GetCameras(), m_hpm, mirror_stream);

			pointLights.Reconstruct(*this, hWorld.GetPointLights(), m_hpm, mirror_stream);
			spotLights.Reconstruct(*this, hWorld.GetSpotLights(), m_hpm, mirror_stream);
			directLights.Reconstruct(*this, hWorld.GetDirectLights(), m_hpm, mirror_stream);

			meshes.Reconstruct(*this, hWorld.GetMeshes(), m_hpm, mirror_stream);
			spheres.Reconstruct(*this, hWorld.GetSpheres(), m_hpm, mirror_stream);
			planes.Reconstruct(*this, hWorld.GetPlanes(), m_hpm, mirror_stream);

			ReconstructMaterial(*this, hWorld.GetMaterial(), mirror_stream);
			ReconstructDefaultMaterial(*this, hWorld.GetDefaultMaterial(), mirror_stream);

			hWorld.GetStateRegister().MakeUnmodified();
		}
		__host__ void CudaWorld::ReconstructMaterial(
			const CudaWorld& hCudaWorld,
			const Material& hMaterial,
			hipStream_t& mirror_stream)
		{
			material = hMaterial;

			// texture
			auto& hTexture = hMaterial.GetTexture();
			if (hTexture)
			{
				if (hTexture.GetResource()->GetId() < hCudaWorld.textures.GetCount())
				{
					material.SetTexture(hCudaWorld.textures.GetStorageAddress() +
						hTexture.GetResource()->GetId());
				}
				else material.SetTexture(nullptr);
			}
			else material.SetTexture(nullptr);
		}
		void CudaWorld::ReconstructDefaultMaterial(
			const CudaWorld& hCudaWorld,
			const Material& hMaterial,
			hipStream_t& mirror_stream)
		{
			RZAssert(bool(default_material), L"default material was nullptr");

			CudaMaterial* hCudaMaterial = (CudaMaterial*)m_hpm.GetPointerToMemory();
			CudaErrorCheck(hipMemcpyAsync(
				hCudaMaterial, default_material,
				sizeof(*default_material),
				hipMemcpyKind::hipMemcpyDeviceToHost, mirror_stream));
			CudaErrorCheck(hipStreamSynchronize(mirror_stream));

			*hCudaMaterial = hMaterial;

			CudaErrorCheck(hipMemcpyAsync(
				default_material, hCudaMaterial,
				sizeof(*default_material),
				hipMemcpyKind::hipMemcpyHostToDevice, mirror_stream));
			CudaErrorCheck(hipStreamSynchronize(mirror_stream));
		}
	}
}