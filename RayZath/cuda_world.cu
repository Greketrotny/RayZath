#include "cuda_world.cuh"

namespace RayZath::Cuda
{
	HostPinnedMemory World::m_hpm(0x10000);

	World::World()
	{
		// default material
		Material* hCudaMaterial = (Material*)m_hpm.GetPointerToMemory();
		new (hCudaMaterial) Material(
			Color<float>(1.0f, 1.0f, 1.0f, 1.0f),
			0.0f, 0.0f, 1.0f, 0.0f, 0.0f);

		CudaErrorCheck(hipMalloc(&default_material, sizeof(*default_material)));
		CudaErrorCheck(hipMemcpy(
			default_material, hCudaMaterial,
			sizeof(*default_material),
			hipMemcpyKind::hipMemcpyHostToDevice));
	}
	World::~World()
	{
		// default material
		if (default_material) CudaErrorCheck(hipFree(default_material));
		default_material = nullptr;
	}

	__host__ void World::ReconstructResources(
		RayZath::Engine::World& hWorld,
		hipStream_t& update_stream)
	{
		ReconstructMaterial(hWorld.GetMaterial(), update_stream);
		ReconstructDefaultMaterial(hWorld.GetDefaultMaterial(), update_stream);

		textures.Reconstruct(*this, hWorld.Container<RayZath::Engine::World::ContainerType::Texture>(), m_hpm, update_stream);
		normal_maps.Reconstruct(*this, hWorld.Container<RayZath::Engine::World::ContainerType::NormalMap>(), m_hpm, update_stream);
		metalness_maps.Reconstruct(*this, hWorld.Container<RayZath::Engine::World::ContainerType::MetalnessMap>(), m_hpm, update_stream);
		roughness_maps.Reconstruct(*this, hWorld.Container<RayZath::Engine::World::ContainerType::RoughnessMap>(), m_hpm, update_stream);
		emission_maps.Reconstruct(*this, hWorld.Container<RayZath::Engine::World::ContainerType::EmissionMap>(), m_hpm, update_stream);

		materials.Reconstruct(*this, hWorld.Container<RayZath::Engine::World::ContainerType::Material>(), m_hpm, update_stream);
		mesh_structures.Reconstruct(*this, hWorld.Container<RayZath::Engine::World::ContainerType::MeshStructure>(), m_hpm, update_stream);
	}
	__host__ void World::ReconstructObjects(
		RayZath::Engine::World& hWorld,
		hipStream_t& update_stream)
	{
		point_lights.Reconstruct(*this, hWorld.Container<RayZath::Engine::World::ContainerType::PointLight>(), m_hpm, update_stream);
		spot_lights.Reconstruct(*this, hWorld.Container<RayZath::Engine::World::ContainerType::SpotLight>(), m_hpm, update_stream);
		direct_lights.Reconstruct(*this, hWorld.Container<RayZath::Engine::World::ContainerType::DirectLight>(), m_hpm, update_stream);

		meshes.Reconstruct(*this, hWorld.Container<RayZath::Engine::World::ContainerType::Mesh>(), m_hpm, update_stream);
		spheres.Reconstruct(*this, hWorld.Container<RayZath::Engine::World::ContainerType::Sphere>(), m_hpm, update_stream);
		planes.Reconstruct(*this, hWorld.Container<RayZath::Engine::World::ContainerType::Plane>(), m_hpm, update_stream);
	}
	__host__ void World::ReconstructCameras(
		RayZath::Engine::World& hWorld,
		hipStream_t& update_stream)
	{
		cameras.Reconstruct(*this, hWorld.Container<RayZath::Engine::World::ContainerType::Camera>(), m_hpm, update_stream);
	}
	void World::ReconstructAll(
		RayZath::Engine::World& hWorld,
		hipStream_t& update_stream)
	{
		if (!hWorld.GetStateRegister().IsModified()) return;

		ReconstructResources(hWorld, update_stream);
		ReconstructObjects(hWorld, update_stream);
		ReconstructCameras(hWorld, update_stream);

		hWorld.GetStateRegister().MakeUnmodified();
	}
	__host__ void World::ReconstructMaterial(
		const RayZath::Engine::Material& hMaterial,
		hipStream_t& mirror_stream)
	{
		material = hMaterial;

		// texture
		auto& hTexture = hMaterial.GetTexture();
		if (hTexture)
		{
			if (hTexture.GetAccessor()->GetIdx() < textures.GetCount())
			{
				material.SetTexture(textures.GetStorageAddress() +
					hTexture.GetAccessor()->GetIdx());
			}
			else material.SetTexture(nullptr);
		}
		else material.SetTexture(nullptr);
	}
	void World::ReconstructDefaultMaterial(
		const RayZath::Engine::Material& hMaterial,
		hipStream_t& mirror_stream)
	{
		RZAssert(bool(default_material), "default material was nullptr");

		Material* hCudaMaterial = (Material*)m_hpm.GetPointerToMemory();
		CudaErrorCheck(hipMemcpyAsync(
			hCudaMaterial, default_material,
			sizeof(*default_material),
			hipMemcpyKind::hipMemcpyDeviceToHost, mirror_stream));
		CudaErrorCheck(hipStreamSynchronize(mirror_stream));

		*hCudaMaterial = hMaterial;

		CudaErrorCheck(hipMemcpyAsync(
			default_material, hCudaMaterial,
			sizeof(*default_material),
			hipMemcpyKind::hipMemcpyHostToDevice, mirror_stream));
		CudaErrorCheck(hipStreamSynchronize(mirror_stream));
	}
}