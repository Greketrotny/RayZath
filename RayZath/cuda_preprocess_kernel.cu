#include "hip/hip_runtime.h"
#include "cuda_preprocess_kernel.cuh"

namespace RayZath::Cuda::Kernel
{
	__global__ void SwapBuffers(
		World* const world,
		const uint8_t camera_idx)
	{
		Camera& camera = world->cameras[camera_idx];
		camera.SwapImageBuffers();
		camera.GetPassesCount() = 1u;
	}
	__global__ void UpdatePassesCount(
		World* const world,
		const uint8_t camera_idx)
	{
		Camera& camera = world->cameras[camera_idx];
		camera.GetPassesCount() += 1u;
	}

	__global__ void GenerateCameraRay(
		GlobalKernel* const global_kernel,
		World* const world,
		const uint32_t camera_idx)
	{
		const GridThread thread;

		Camera& camera = world->cameras[camera_idx];
		if (thread.grid_pos.x >= camera.GetWidth() ||
			thread.grid_pos.y >= camera.GetHeight()) return;

		GlobalKernel& gkernel = *global_kernel;
		ConstantKernel& ckernel = const_kernel[gkernel.GetRenderIdx()];

		// create RNG
		RNG rng(
			vec2f(
				thread.grid_pos.x / float(camera.GetWidth()),
				thread.grid_pos.y / float(camera.GetHeight())),
			ckernel.GetSeeds().GetSeed(thread.grid_idx));

		// generate camera ray
		SceneRay camera_ray;
		camera.GenerateSimpleRay(
			camera_ray,
			thread,
			rng);
		camera_ray.material = &world->material;
		camera.GetTracingStates().SetRay(thread.grid_pos, camera_ray);
	}
}