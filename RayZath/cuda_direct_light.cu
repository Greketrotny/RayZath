#include "hip/hip_runtime.h"
#include "cuda_direct_light.cuh"

namespace RayZath
{
	__host__ CudaDirectLight::CudaDirectLight()
	{
	}
	__host__ CudaDirectLight::~CudaDirectLight()
	{
	}

	__host__ void CudaDirectLight::Reconstruct(
		DirectLight& hDirectLight, 
		hipStream_t& mirror_stream)
	{
		if (!hDirectLight.GetStateRegister().IsModified()) return;

		direction = hDirectLight.GetDirection();
		color = hDirectLight.GetColor();
		emission = hDirectLight.GetEmission();
		angular_size = hDirectLight.GetAngularSize();

		cos_angular_size = cosf(angular_size);

		hDirectLight.GetStateRegister().MakeUnmodified();
	}
}