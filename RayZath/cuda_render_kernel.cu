#include "hip/hip_runtime.h"
#include "cuda_render_kernel.cuh"

#include "hip/hip_math_constants.h"

namespace RayZath::Cuda::Kernel
{
	__global__ void RenderFirstPass(
		GlobalKernel* const global_kernel,
		World* const world,
		const uint32_t camera_idx)
	{
		FullThread thread;

		// get camera and clamp working threads
		Camera& camera = world->cameras[camera_idx];
		if (thread.in_grid.x >= camera.GetWidth() ||
			thread.in_grid.y >= camera.GetHeight()) return;

		// get kernels
		GlobalKernel& gkernel = *global_kernel;
		ConstantKernel& ckernel = const_kernel[gkernel.GetRenderIdx()];

		// create RNG
		RNG rng(
			vec2f(
				thread.in_grid.x / float(camera.GetWidth()),
				thread.in_grid.y / float(camera.GetHeight())),
			ckernel.GetSeeds().GetSeed(thread.in_grid_idx));

		// create intersection object
		RayIntersection intersection;
		intersection.ray = camera.GetTracingStates().GetRay(thread.in_grid);
		intersection.ray.near_far = camera.GetNearFar();

		// trace ray through scene
		TracingState tracing_state(ColorF(0.0f), 0u);
		const vec3f sample_direction = TraceRay(ckernel, *world, tracing_state, intersection, rng);

		// set depth
		camera.CurrentDepthBuffer().SetValue(
			thread.in_grid,
			intersection.ray.near_far.y);

		// set intersection point
		camera.SpaceBuffer().SetValue(
			thread.in_grid,
			intersection.ray.origin + intersection.ray.direction * intersection.ray.near_far.y);

		// set path depth
		camera.GetTracingStates().SetPathDepth(
			thread.in_grid,
			tracing_state.path_depth);

		// set color value
		tracing_state.final_color.alpha = 1.0f;
		camera.CurrentImageBuffer().SetValue(
			thread.in_grid,
			tracing_state.final_color);

		if (tracing_state.path_depth < ckernel.GetRenderConfig().GetTracing().GetMaxDepth())
		{
			intersection.ray.color.Blend(
				intersection.ray.color * intersection.color,
				intersection.next_ray_metalness);

			intersection.RepositionRay(sample_direction);
			camera.GetTracingStates().SetRay(thread.in_grid, intersection.ray);

			camera.AppendPathPos(thread.in_grid);
		}
	}
	__global__ void RenderRegeneratedPass(
		GlobalKernel* const global_kernel,
		World* const world,
		const uint32_t camera_idx)
	{
		FullThread thread;

		Camera& camera = world->cameras[camera_idx];
		if (thread.in_grid.x >= camera.GetWidth() ||
			thread.in_grid.y >= camera.GetHeight()) return;

		// get kernels
		GlobalKernel& gkernel = *global_kernel;
		ConstantKernel& ckernel = const_kernel[gkernel.GetRenderIdx()];

		TracingState tracing_state(
			ColorF(0.0f),
			camera.GetTracingStates().GetPathDepth(thread.in_grid));

		RNG rng(
			vec2f(
				thread.in_grid.x / float(camera.GetWidth()),
				thread.in_grid.y / float(camera.GetHeight())),
			ckernel.GetSeeds().GetSeed(thread.in_grid_idx + tracing_state.path_depth));

		// create intersection object
		RayIntersection intersection;
		intersection.ray = camera.GetTracingStates().GetRay(thread.in_grid);

		// trace ray through scene
		const vec3f sample_direction = TraceRay(ckernel, *world, tracing_state, intersection, rng);
		const bool path_continues = tracing_state.path_depth < ckernel.GetRenderConfig().GetTracing().GetMaxDepth();

		// update path depth
		camera.GetTracingStates().SetPathDepth(
			thread.in_grid,
			tracing_state.path_depth);

		// append additional light contribution passing along traced ray
		ColorF sample = camera.CurrentImageBuffer().GetValue(thread.in_grid);
		sample.red += tracing_state.final_color.red;
		sample.green += tracing_state.final_color.green;
		sample.blue += tracing_state.final_color.blue;
		sample.alpha += float(!path_continues);
		camera.CurrentImageBuffer().SetValue(thread.in_grid, sample);

		if (path_continues)
		{
			intersection.ray.color.Blend(
				intersection.ray.color * intersection.color,
				intersection.next_ray_metalness);

			intersection.RepositionRay(sample_direction);
			camera.GetTracingStates().SetRay(thread.in_grid, intersection.ray);

			camera.AppendPathPos(thread.in_grid);
		}
	}
	__global__ void RenderCumulativePass(
		GlobalKernel* const global_kernel,
		World* const world,
		const uint32_t camera_idx)
	{
		FullThread thread;

		Camera& camera = world->cameras[camera_idx];
		if (thread.in_grid.x >= camera.GetWidth() ||
			thread.in_grid.y >= camera.GetHeight()) return;
		if (thread.in_grid.y * camera.GetWidth() + thread.in_grid.x >= camera.GetTracingStates().GetPathCount())
			return;

		// get kernels
		GlobalKernel& gkernel = *global_kernel;
		ConstantKernel& ckernel = const_kernel[gkernel.GetRenderIdx()];

		const vec2ui32 path_pixel(vec2ui32(camera.GetTracingStates().GetPathPos(thread.in_grid)));
		TracingState tracing_state(
			ColorF(0.0f),
			camera.GetTracingStates().GetPathDepth(path_pixel));

		RNG rng(
			vec2f(
				thread.in_grid.x / float(camera.GetWidth()),
				thread.in_grid.y / float(camera.GetHeight())),
			ckernel.GetSeeds().GetSeed(thread.in_grid_idx + tracing_state.path_depth));

		// create intersection object
		RayIntersection intersection;
		intersection.ray = camera.GetTracingStates().GetRay(path_pixel);

		// trace ray through scene
		const vec3f sample_direction = TraceRay(ckernel, *world, tracing_state, intersection, rng);
		const bool path_continues = tracing_state.path_depth < ckernel.GetRenderConfig().GetTracing().GetMaxDepth();

		// update path depth
		camera.GetTracingStates().SetPathDepth(
			path_pixel,
			tracing_state.path_depth);

		// append additional light contribution passing along traced ray
		ColorF sample = camera.CurrentImageBuffer().GetValue(path_pixel);
		sample.red += tracing_state.final_color.red;
		sample.green += tracing_state.final_color.green;
		sample.blue += tracing_state.final_color.blue;
		sample.alpha += float(!path_continues);
		camera.CurrentImageBuffer().SetValue(path_pixel, sample);

		if (path_continues)
		{
			intersection.ray.color.Blend(
				intersection.ray.color * intersection.color,
				intersection.next_ray_metalness);

			intersection.RepositionRay(sample_direction);
			camera.GetTracingStates().SetRay(path_pixel, intersection.ray);

			camera.AppendPathPos(path_pixel);
		}
	}
	__global__ void RegenerateTerminatedRay(
		GlobalKernel* const global_kernel,
		World* const world,
		const uint32_t camera_idx)
	{
		Camera& camera = world->cameras[camera_idx];
		GridThread thread;
		if (thread.in_grid.x >= camera.GetWidth() ||
			thread.in_grid.y >= camera.GetHeight()) return;

		GlobalKernel& gkernel = *global_kernel;
		ConstantKernel& ckernel = const_kernel[gkernel.GetRenderIdx()];

		const uint8_t path_depth = camera.GetTracingStates().GetPathDepth(thread.in_grid);

		// create RNG
		RNG rng(
			vec2f(
				thread.in_grid.x / float(camera.GetWidth()),
				thread.in_grid.y / float(camera.GetHeight())),
			ckernel.GetSeeds().GetSeed(thread.in_grid_idx + path_depth));

		if (path_depth >= ckernel.GetRenderConfig().GetTracing().GetMaxDepth())
		{
			// generate camera ray
			SceneRay camera_ray;
			camera.GenerateRay(
				camera_ray,
				thread.in_grid,
				rng);
			camera_ray.material = &world->material;

			camera.GetTracingStates().SetRay(thread.in_grid, camera_ray);
			camera.GetTracingStates().SetPathDepth(thread.in_grid, 0u);
		}
	}
	__global__ void SwapPathIndexing(
		World* const world,
		const uint32_t camera_idx)
	{
		Camera& camera = world->cameras[camera_idx];
		camera.SwapPathIndexing();
	}


	__device__ vec3f TraceRay(
		ConstantKernel& ckernel,
		const World& world,
		TracingState& tracing_state,
		RayIntersection& intersection,
		RNG& rng)
	{
		// find closest intersection with the world
		const bool any_hit = world.ClosestIntersection(intersection, rng);

		// fetch color and emission at given point on material surface
		intersection.color =
			intersection.surface_material->GetOpacityColor(intersection.texcrd);
		intersection.emission =
			intersection.surface_material->GetEmission(intersection.texcrd);

		if (intersection.emission > 0.0f)
		{	// intersection with emitting object

			tracing_state.final_color +=
				intersection.ray.color *
				intersection.color *
				intersection.emission;
		}

		if (!any_hit)
		{	// nothing has been hit - terminate path

			tracing_state.EndPath();
			return vec3f(1.0f);
		}
		++tracing_state.path_depth;


		// [>] Apply Beer's law

		// P0 - light energy in front of an object
		// P - light energy after going through an object
		// A - absorbance

		// e - material absorbance (constant)
		// b - distance traveled in an object
		// c - molar concentration (constant)

		// A = 10 ^ -(e * b * c)
		// P = P0 * A

		intersection.ray.color *=
			intersection.ray.material->GetOpacityColor() *
			cui_powf(intersection.ray.material->GetOpacityColor().alpha, intersection.ray.near_far.y);


		// Fetch metalness  and roughness from surface material
		// (needed for BRDF and next even estimation)
		intersection.metalness =
			intersection.surface_material->GetMetalness(intersection.texcrd);
		intersection.roughness =
			intersection.surface_material->GetRoughness(intersection.texcrd);

		// calculate fresnel and reflectance ratio
		// (for BRDF and next ray generation)
		intersection.fresnel = FresnelSpecularRatio(
			intersection.mapped_normal,
			intersection.ray.direction,
			intersection.ray.material->GetIOR(),
			intersection.behind_material->GetIOR());
		intersection.reflectance = Lerp(intersection.fresnel, 1.0f, intersection.metalness);

		// find intersection point 
		// (needed for direct sampling and next ray generation)
		intersection.point =
			intersection.ray.origin +
			intersection.ray.direction *
			intersection.ray.near_far.y;

		// sample direction (importance sampling)
		// (for next ray generation and direct light sampling (MIS))
		const vec3f sample_direction = intersection.surface_material->SampleDirection(intersection, rng);

		// Direct sampling
		if (intersection.surface_material->SampleDirect(intersection) &&
			world.SampleDirect(ckernel))
		{
			// sample direct light
			const ColorF direct_illumination = DirectIllumination(ckernel, world, intersection, sample_direction, rng);

			// add direct light
			tracing_state.final_color +=
				direct_illumination * // incoming radiance from lights
				intersection.ray.color * // ray color mask
				Lerp(ColorF(1.0f), intersection.color, intersection.metalness); // metalic factor
		}

		return sample_direction;
	}

	__device__ ColorF SpotLightSampling(
		ConstantKernel& ckernel,
		const World& world,
		const RayIntersection& intersection,
		const vec3f& vS,
		const float vS_pdf,
		RNG& rng)
	{
		const uint32_t light_count = world.spot_lights.GetCount();
		const uint32_t sample_count = ckernel.GetRenderConfig().GetLightSampling().GetSpotLight();
		if (light_count == 0u || sample_count == 0u)
			return ColorF(0.0f);


		ColorF total_light(0.0f);
		for (uint32_t i = 0u; i < sample_count; ++i)
		{
			const SpotLight& light = world.spot_lights[uint32_t(rng.UnsignedUniform() * light_count)];

			// sample light
			float Se = 0.0f;
			const vec3f vPL = light.SampleDirection(
				intersection.point,
				vS, Se,
				rng);
			const float dPL = vPL.Length();

			const float brdf = intersection.surface_material->BRDF(intersection, vPL / dPL);
			if (brdf < 1.0e-4f) continue;
			const ColorF brdf_color = intersection.surface_material->BRDFColor(intersection);
			const float solid_angle = light.SolidAngle(dPL);
			const float sctr_factor = cui_expf(-dPL * intersection.ray.material->GetScattering());

			// beam illumination
			const float beamIllum = light.BeamIllumination(vPL);
			if (beamIllum < 1.0e-4f)
				continue;

			// calculate radiance at P
			const float L_pdf = 1.0f / solid_angle;
			const float vSw = vS_pdf / (vS_pdf + L_pdf);
			const float Lw = 1.0f - vSw;
			const float Le = light.GetEmission() * solid_angle * brdf;
			const float radiance = (Le * Lw + Se * vSw) * sctr_factor * beamIllum;
			if (radiance < 1.0e-4f) continue;	// unimportant light contribution

			// cast shadow ray and calculate color contribution
			const RangedRay shadowRay(intersection.point + intersection.surface_normal * 0.001f, vPL, vec2f(0.0f, dPL));
			const ColorF V_PL = world.AnyIntersection(shadowRay);
			total_light +=
				light.GetColor() *
				brdf_color *
				radiance *
				V_PL * V_PL.alpha;
		}

		const float pdf = sample_count / float(light_count);
		return total_light / pdf;
	}
	__device__ ColorF DirectLightSampling(
		ConstantKernel& ckernel,
		const World& world,
		const RayIntersection& intersection,
		const vec3f& vS,
		const float vS_pdf,
		RNG& rng)
	{
		const uint32_t light_count = world.direct_lights.GetCount();
		const uint32_t sample_count = ckernel.GetRenderConfig().GetLightSampling().GetDirectLight();
		if (light_count == 0u || sample_count == 0u)
			return ColorF(0.0f);

		ColorF total_light(0.0f);
		for (uint32_t i = 0u; i < sample_count; ++i)
		{
			const auto& light = world.direct_lights[uint32_t(rng.UnsignedUniform() * light_count)];

			// sample light
			float Se = 0.0f;
			const vec3f vPL = light.SampleDirection(
				vS, Se,
				rng);

			const float brdf = intersection.surface_material->BRDF(intersection, vPL.Normalized());
			const ColorF brdf_color = intersection.surface_material->BRDFColor(intersection);
			const float solid_angle = light.SolidAngle();

			// calculate radiance at P
			const float L_pdf = 1.0f / solid_angle;
			const float vSw = vS_pdf / (vS_pdf + L_pdf);
			const float Lw = 1.0f - vSw;
			const float Le = light.GetEmission() * solid_angle * brdf;
			const float radiance = (Le * Lw + Se * vSw);
			if (radiance < 1.0e-4f) continue;	// unimportant light contribution

			// cast shadow ray and calculate color contribution
			const RangedRay shadowRay(intersection.point + intersection.surface_normal * 0.0001f, vPL);
			const ColorF V_PL = world.AnyIntersection(shadowRay);
			total_light +=
				light.GetColor() *
				brdf_color *
				radiance *
				V_PL * V_PL.alpha;
		}

		const float pdf = sample_count / float(light_count);
		return total_light / pdf;
	}
	__device__ ColorF DirectIllumination(
		ConstantKernel& ckernel,
		const World& world,
		const RayIntersection& intersection,
		const vec3f& vS,
		RNG& rng)
	{
		const float vS_pdf = intersection.surface_material->BRDF(intersection, vS);

		return
			SpotLightSampling(ckernel, world, intersection, vS, vS_pdf, rng) +
			DirectLightSampling(ckernel, world, intersection, vS, vS_pdf, rng);
	}



	// kernels in shared memory:
	/*// create local thread structure
		ThreadData thread(global_kernel->randomNumbers.GetSeed(threadIdx.y * blockDim.x + threadIdx.x));

		//CudaKernelData* const kernel = global_kernel;

		// [>] Copy kernel to shared memory
		extern __shared__ CudaKernelData shared_kernel[];
		CudaKernelData* kernel = shared_kernel;

		// copy render index
		if (thread.thread_in_kernel == 0u)
			kernel->renderIndex = global_kernel->renderIndex;

		// copy unsigned random floats
		const uint32_t linear_block_size = blockDim.x * blockDim.y;
		for (uint32_t i = thread.thread_in_block; i < RNG::s_count; i += linear_block_size)
		{
			kernel->randomNumbers.m_unsigned_uniform[i] =
				global_kernel->randomNumbers.m_unsigned_uniform[i];
		}

		__syncthreads();*/
}