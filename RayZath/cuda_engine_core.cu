#include "hip/hip_runtime.h"
#include "cuda_engine_core.cuh"

#include "cuda_kernel_data.cuh"

#include "point.h"

namespace RayZath::Cuda
{
	EngineCore::EngineCore()
		: mp_dCudaWorld(nullptr)
		, mp_hCudaWorld(nullptr)
		, m_hpm_CudaWorld(sizeof(World))
		, m_hpm_CudaKernel(std::max(sizeof(Kernel::GlobalKernel), sizeof(Kernel::ConstantKernel)))
		, m_renderer(this)
		, m_fence_track(false)
	{
		hipSetDevice(0);

		CreateStreams();
		CreateGlobalKernels();
		CreateCudaWorld();
	}
	EngineCore::~EngineCore()
	{
		DestroyCudaWorld();
		DestroyGlobalKernels();
		DestroyStreams();

		//GetHardware().Reset();
	}

	void EngineCore::RenderWorld(
		RayZath::Engine::World& hWorld,
		const RayZath::Engine::RenderConfig& render_config,
		const bool block,
		const bool sync)
	{
		std::lock_guard<std::mutex> lg(m_mtx);

		// check reported exceptions and throw if any
		m_renderer.ThrowIfException();
		m_renderer.LaunchThread();

		m_core_time_table.ResetTable();
		m_core_time_table.ResetTime();


		// [>] Async reconstruction
		SetState(State::Work);
		SetStage(Stage::AsyncReconstruction);

		// update host world
		m_update_flag = hWorld.GetStateRegister().RequiresUpdate();
		mp_hWorld = &hWorld;
		hWorld.Update();
		m_core_time_table.AppendStage("hWorld update");

		// create launch configurations
		m_configs[m_indexer.UpdateIdx()].Construct(m_hardware, hWorld, m_update_flag);
		m_core_time_table.AppendStage("configs construct");

		// reconstruct cuda kernels
		m_render_config = render_config;
		ReconstructKernels();
		m_core_time_table.AppendStage("kernels reconstruct");

		m_fence_track.OpenGate(size_t(EngineCore::Stage::AsyncReconstruction));
		SetState(State::Wait);
		m_renderer.GetFenceTrack().WaitForEndOfAndClose(size_t(Renderer::Stage::MainRender));
		m_core_time_table.AppendStage("wait for main render");


		// [>] dCudaWorld async reconstruction
		SetState(State::Work);
		SetStage(Stage::WorldReconstruction);

		if (mp_hWorld->GetStateRegister().IsModified())
		{
			// reconstruct resources and objects
			CopyCudaWorldDeviceToHost();
			mp_hCudaWorld->ReconstructResources(hWorld, m_update_stream);
			mp_hCudaWorld->ReconstructObjects(hWorld, m_update_stream);
		}
		m_core_time_table.AppendStage("objects reconstruct");

		// wait for postprocess to end
		m_fence_track.OpenGate(size_t(EngineCore::Stage::WorldReconstruction));
		SetState(State::Wait);
		m_renderer.GetFenceTrack().WaitForEndOfAndClose(size_t(Renderer::Stage::Postprocess));
		m_core_time_table.AppendStage("wait for postprocess");


		// [>] dCudaWorld sync reconstruction (Camera reconstructions)
		SetState(State::Work);
		SetStage(Stage::CameraReconstruction);

		if (mp_hWorld->GetStateRegister().IsModified())
		{
			// reconstruct cameras
			mp_hCudaWorld->ReconstructCameras(hWorld, m_update_stream);
			CopyCudaWorldHostToDevice();
			mp_hWorld->GetStateRegister().MakeUnmodified();
		}
		m_core_time_table.AppendStage("cameras reconstruct");

		m_fence_track.OpenGate(size_t(EngineCore::Stage::CameraReconstruction));
		SetState(State::Wait);
		m_renderer.GetFenceTrack().WaitForEndOfAndClose(size_t(Renderer::Stage::Idle));


		// [>] Synchronize with renderer
		SetState(State::Work);
		SetStage(Stage::Synchronization);

		// swap indices
		m_indexer.Swap();
		m_render_time_table = m_renderer.GetTimeTable();

		m_fence_track.OpenGate(size_t(EngineCore::Stage::Synchronization));

		if (sync)
		{
			m_fence_track.OpenGate(size_t(EngineCore::Stage::ResultTransfer));
			SetState(State::Wait);
			m_renderer.GetFenceTrack().WaitForEndOf(size_t(Renderer::Stage::Postprocess));
			m_core_time_table.AppendStage("sync wait");
		}


		// [>] Transfer results to host side
		SetState(State::Work);
		SetStage(Stage::ResultTransfer);

		CopyRenderToHost();
		m_core_time_table.AppendStage("result tranfer");
		m_core_time_table.AppendFullCycle("full host cycle");

		SetState(State::None);
		SetStage(Stage::None);
		m_fence_track.OpenGate(size_t(EngineCore::Stage::ResultTransfer));
	}
	void EngineCore::CopyRenderToHost()
	{
		if (World::m_hpm.GetSize() < sizeof(Camera))
			ThrowException("insufficient host pinned memory for Camera");

		// [>] Get World from device
		World* hCudaWorld = (World*)m_hpm_CudaWorld.GetPointerToMemory();
		CudaErrorCheck(hipMemcpyAsync(
			hCudaWorld, mp_dCudaWorld,
			sizeof(World),
			hipMemcpyKind::hipMemcpyDeviceToHost, m_update_stream));
		CudaErrorCheck(hipStreamSynchronize(m_update_stream));

		if (hCudaWorld->cameras.GetCount() == 0u) return;	// hCudaWorld has no cameras


		const uint32_t count = std::min(
			hCudaWorld->cameras.GetCount(),
			mp_hWorld->Container<RayZath::Engine::World::ContainerType::Camera>().GetCount());
		for (uint32_t i = 0u; i < count; ++i)
		{
			// check if hostCamera is enabled
			const auto& hCamera = mp_hWorld->Container<RayZath::Engine::World::ContainerType::Camera>()[i];
			if (!hCamera) continue;	// no camera at this address
			if (!hCamera->Enabled()) continue;	// camera is disabled

			// [>] Get Camera class from hCudaWorld
			Camera* hCudaCamera = (Camera*)World::m_hpm.GetPointerToMemory();
			CudaErrorCheck(hipMemcpyAsync(
				hCudaCamera, &hCudaWorld->cameras[i],
				sizeof(Camera),
				hipMemcpyKind::hipMemcpyDeviceToHost, m_update_stream));
			CudaErrorCheck(hipStreamSynchronize(m_update_stream));
			

			// [>] Asynchronous copying
			hCamera->m_ray_count = hCudaCamera->GetResultRayCount();

			static_assert(
				sizeof(*hCamera->GetImageBuffer().GetMapAddress()) ==
				sizeof(Color<unsigned char>),
				"sizeof(Graphics::Color) != sizeof(Color<unsigned char>)");

			// check cameras resolution
			if (hCamera->GetWidth() != hCudaCamera->GetWidth() ||
				hCamera->GetHeight() != hCudaCamera->GetHeight()) continue;

			uint32_t chunkSize = uint32_t(
				hCudaCamera->hostPinnedMemory.GetSize() /
				(sizeof(Color<unsigned char>)));
			if (chunkSize < 1024u) ThrowException("Not enough host pinned memory for async image copy");

			uint32_t nPixels = hCamera->GetWidth() * hCamera->GetHeight();
			for (uint32_t startIndex = 0; startIndex < nPixels; startIndex += chunkSize)
			{
				// find start index
				if (startIndex + chunkSize > nPixels) chunkSize = nPixels - startIndex;

				// find offset point
				Graphics::Point<uint32_t> offset_point(
					startIndex % hCamera->GetWidth(),
					startIndex / hCamera->GetWidth());

				// copy final image data from hCudaCamera to hCudaPixels on pinned memory
				Color<unsigned char>* hCudaPixels =
					(Color<unsigned char>*)Camera::hostPinnedMemory.GetPointerToMemory();
				CudaErrorCheck(cudaMemcpyFromArrayAsync(
					hCudaPixels, hCudaCamera->FinalImageBuffer().GetCudaArray(),
					offset_point.x * sizeof(*hCudaPixels), offset_point.y,
					chunkSize * sizeof(*hCudaPixels),
					hipMemcpyKind::hipMemcpyDeviceToHost, m_update_stream));
				CudaErrorCheck(hipStreamSynchronize(m_update_stream));

				// copy final image data from hostCudaPixels on pinned memory to hostCamera
				hCamera->m_image_buffer.CopyFromMemory(
					hCudaPixels,
					chunkSize * sizeof(*hCudaPixels),
					offset_point.x, offset_point.y);


				// [>] Copy depth buffer
				float* hCudaDepthData =
					(float*)Camera::hostPinnedMemory.GetPointerToMemory();
				CudaErrorCheck(cudaMemcpyFromArrayAsync(
					hCudaDepthData, hCudaCamera->FinalDepthBuffer().GetCudaArray(),
					offset_point.x * sizeof(*hCudaDepthData), offset_point.y,
					chunkSize * sizeof(*hCudaDepthData),
					hipMemcpyKind::hipMemcpyDeviceToHost, m_update_stream));
				CudaErrorCheck(hipStreamSynchronize(m_update_stream));

				// copy final image data from hostCudaPixels on pinned memory to hostCamera
				hCamera->m_depth_buffer.CopyFromMemory(
					hCudaDepthData,
					chunkSize * sizeof(*hCudaDepthData),
					offset_point.x, offset_point.y);
			}
		}
	}


	void EngineCore::CreateStreams()
	{
		CudaErrorCheck(hipStreamCreate(&m_update_stream));
		CudaErrorCheck(hipStreamCreate(&m_render_stream));
	}
	void EngineCore::DestroyStreams()
	{
		CudaErrorCheck(hipStreamDestroy(m_update_stream));
		CudaErrorCheck(hipStreamDestroy(m_render_stream));
	}
	void EngineCore::CreateGlobalKernels()
	{
		Kernel::GlobalKernel* hCudaGlobalKernel =
			(Kernel::GlobalKernel*)m_hpm_CudaKernel.GetPointerToMemory();
		for (uint32_t i = 0u; i < 2u; ++i)
		{
			new (hCudaGlobalKernel) Kernel::GlobalKernel();

			CudaErrorCheck(hipMalloc(
				(void**)&mp_global_kernel[i], sizeof(Kernel::GlobalKernel)));
			CudaErrorCheck(hipMemcpy(mp_global_kernel[i], hCudaGlobalKernel,
				sizeof(Kernel::GlobalKernel), hipMemcpyKind::hipMemcpyHostToDevice));
		}
	}
	void EngineCore::DestroyGlobalKernels()
	{
		Kernel::GlobalKernel* hCudaKernelData =
			(Kernel::GlobalKernel*)m_hpm_CudaKernel.GetPointerToMemory();
		for (uint32_t i = 0u; i < 2u; ++i)
		{
			CudaErrorCheck(hipMemcpy(
				hCudaKernelData, mp_global_kernel[i],
				sizeof(Kernel::GlobalKernel),
				hipMemcpyKind::hipMemcpyDeviceToHost));

			hCudaKernelData->~GlobalKernel();

			CudaErrorCheck(hipFree(mp_global_kernel[i]));
			mp_global_kernel[i] = nullptr;
		}
	}
	void EngineCore::ReconstructKernels()
	{
		// [>] GlobalKernel
		// get hpm memory
		Kernel::GlobalKernel* hCudaGlobalKernel =
			(Kernel::GlobalKernel*)m_hpm_CudaKernel.GetPointerToMemory();

		// copy dCudaKernelData to host
		CudaErrorCheck(hipMemcpyAsync(
			hCudaGlobalKernel,
			mp_global_kernel[m_indexer.UpdateIdx()],
			sizeof(Kernel::GlobalKernel),
			hipMemcpyKind::hipMemcpyDeviceToHost,
			m_update_stream));
		CudaErrorCheck(hipStreamSynchronize(m_update_stream));

		// reconstruct hCudaGlobalKernel
		hCudaGlobalKernel->Reconstruct(
			m_indexer.UpdateIdx(),
			m_update_stream);

		// copy hCudaGlobalKernel to device
		CudaErrorCheck(hipMemcpyAsync(
			mp_global_kernel[m_indexer.UpdateIdx()],
			hCudaGlobalKernel,
			sizeof(Kernel::GlobalKernel),
			hipMemcpyKind::hipMemcpyHostToDevice,
			m_update_stream));
		CudaErrorCheck(hipStreamSynchronize(m_update_stream));


		// [>] ConstantKernel
		// get hpm memory
		Kernel::ConstantKernel* hCudaConstantKernel =
			(Kernel::ConstantKernel*)m_hpm_CudaKernel.GetPointerToMemory();

		// reconstruct hCudaConstantKernel
		hCudaConstantKernel->Reconstruct(
			m_render_config);

		// copy hCudaConstantKernel to device __constant__ memory
		Kernel::CopyConstantKernel(
			hCudaConstantKernel,
			m_indexer.UpdateIdx(), m_update_stream);
	}
	void EngineCore::CreateCudaWorld()
	{
		mp_hCudaWorld = (World*)m_hpm_CudaWorld.GetPointerToMemory();
		new (mp_hCudaWorld) World();
		CudaErrorCheck(hipMalloc(&mp_dCudaWorld, sizeof(World)));
		CopyCudaWorldHostToDevice();
	}
	void EngineCore::DestroyCudaWorld()
	{
		if (mp_dCudaWorld)
		{
			CopyCudaWorldDeviceToHost();
			mp_hCudaWorld->~World();
			CudaErrorCheck(hipFree(mp_dCudaWorld));
			mp_dCudaWorld = nullptr;
		}
	}
	void EngineCore::CopyCudaWorldDeviceToHost()
	{
		mp_hCudaWorld = (World*)m_hpm_CudaWorld.GetPointerToMemory();
		CudaErrorCheck(hipMemcpyAsync(
			mp_hCudaWorld, mp_dCudaWorld,
			sizeof(World),
			hipMemcpyKind::hipMemcpyDeviceToHost, m_update_stream));
		CudaErrorCheck(hipStreamSynchronize(m_update_stream));
	}
	void EngineCore::CopyCudaWorldHostToDevice()
	{
		if (mp_dCudaWorld && mp_hCudaWorld)
		{
			CudaErrorCheck(hipMemcpyAsync(
				mp_dCudaWorld, mp_hCudaWorld,
				sizeof(World),
				hipMemcpyKind::hipMemcpyHostToDevice, m_update_stream));
			CudaErrorCheck(hipStreamSynchronize(m_update_stream));
		}
	}


	Hardware& EngineCore::GetHardware()
	{
		return m_hardware;
	}
	Indexer& EngineCore::GetIndexer()
	{
		return m_indexer;
	}

	Renderer& EngineCore::GetRenderer()
	{
		return m_renderer;
	}
	LaunchConfigurations& EngineCore::GetLaunchConfigs(const bool idx)
	{
		return m_configs[idx];
	}
	Kernel::GlobalKernel* EngineCore::GetGlobalKernel(const bool idx)
	{
		return mp_global_kernel[idx];
	}
	const RayZath::Engine::RenderConfig& EngineCore::GetRenderConfig() const
	{
		return m_render_config;
	}
	World* EngineCore::GetCudaWorld()
	{
		return mp_dCudaWorld;
	}
	EngineCore::FenceTrack_t& EngineCore::GetFenceTrack()
	{
		return m_fence_track;
	}
	const TimeTable& EngineCore::GetCoreTimeTable() const
	{
		return m_core_time_table;
	}
	const TimeTable& EngineCore::GetRenderTimeTable() const
	{
		return m_render_time_table;
	}

	hipStream_t& EngineCore::GetUpdateStream()
	{
		return m_update_stream;
	}
	hipStream_t& EngineCore::GetRenderStream()
	{
		return m_render_stream;
	}

	const EngineCore::State& EngineCore::GetState()
	{
		return m_state;
	}
	const EngineCore::Stage& EngineCore::GetStage()
	{
		return m_stage;
	}
	void EngineCore::SetState(const EngineCore::State& state)
	{
		m_state = state;
	}
	void EngineCore::SetStage(const EngineCore::Stage& stage)
	{
		m_stage = stage;
	}
}