#include "hip/hip_runtime.h"
#include "cuda_engine_core.cuh"

#include "cuda_preprocess_kernel.cuh"
#include "cuda_render_kernel.cuh"
#include "cuda_postprocess_kernel.cuh"

#include "point.h"

#include <algorithm>
#include <ios>
#include <sstream>

namespace RayZath::Cuda
{
	// ~~~~~~~~ [STRUCT] Indexer ~~~~~~~~
	Indexer::Indexer()
		: m_update_idx(0u)
		, m_render_idx(1u)
	{}

	const bool& Indexer::UpdateIdx() const
	{
		return m_update_idx;
	}
	const bool& Indexer::RenderIdx() const
	{
		return m_render_idx;
	}
	void Indexer::Swap()
	{
		std::swap(m_update_idx, m_render_idx);
	}
	// ~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~


	// ~~~~~~~~ [STRUCT] TimeTable ~~~~~~~~
	TimeTable::TimeTable()
	{
		m_timer.Start();
	}

	void TimeTable::AppendStage(const std::string& s)
	{
		m_stamps.push_back({ s, m_timer.GetTime() });
	}
	void TimeTable::AppendFullCycle(const std::string& s)
	{
		m_stamps.push_back({ s, m_cycle_timer.GetTime() });
	}
	void TimeTable::ResetTime()
	{
		m_timer.Start();
		m_cycle_timer.Start();
	}
	void TimeTable::ResetTable()
	{
		m_stamps.clear();
	}
	std::string TimeTable::ToString(const uint32_t width) const
	{
		std::stringstream ss;
		for (auto& stamp : m_stamps)
		{
			ss.fill(' ');
			ss.width(width);
			ss << stamp.first << ": ";

			std::ignore = ss.width();
			ss.precision(3);
			ss << std::fixed << stamp.second << "ms\n";
		}
		return ss.str();
	}
	// ~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~


	// ~~~~~~~~ [STRUCT] Renderer ~~~~~~~~
	Renderer::Renderer(EngineCore* const engine_core)
		: mp_engine_core(engine_core)
		, m_is_thread_alive(false)
		, m_terminate_thread(false)
		, m_state(State::None)
		, m_stage(Stage::None)
		, mp_blocking_gate(nullptr)
		, m_fence_track(true)
	{}
	Renderer::~Renderer()
	{
		TerminateThread();
	}

	void Renderer::LaunchThread()
	{
		std::lock_guard<std::mutex> lg(m_mtx);
		if (!m_is_thread_alive)
		{
			m_terminate_thread = false;
			m_is_thread_alive = true;
			mp_blocking_gate = nullptr;
			ResetExceptions();

			if (mp_render_thread)
			{
				if (mp_render_thread->joinable())
					mp_render_thread->join();
			}
			mp_render_thread.reset(new std::thread(
				&Renderer::RenderFunctionWrapper,
				this));
		}
	}
	void Renderer::TerminateThread()
	{
		{
			std::lock_guard<std::mutex> lg(m_mtx);
			m_terminate_thread = true;
		}
		if (m_is_thread_alive)
		{
			mp_engine_core->GetFenceTrack().OpenAll();

			if (mp_render_thread->joinable())
				mp_render_thread->join();

			mp_render_thread.reset();
			m_is_thread_alive = false;
		}
	}

	FenceTrack<5>& Renderer::GetFenceTrack()
	{
		return m_fence_track;
	}
	const TimeTable& Renderer::GetTimeTable() const
	{
		return m_time_table;
	}
	const Renderer::State& Renderer::GetState() const
	{
		return m_state;
	}
	const Renderer::Stage& Renderer::GetStage() const
	{
		return m_stage;
	}

	void Renderer::SetState(const Renderer::State& state)
	{
		m_state = state;
	}
	void Renderer::SetStage(const Renderer::Stage& stage)
	{
		m_stage = stage;
	}

	void Renderer::RenderFunctionWrapper()
	{
		RenderFunction();
		std::lock_guard<std::mutex> lg(m_mtx);
		m_is_thread_alive = false;
		m_state = State::None;
		m_stage = Stage::None;
		m_fence_track.OpenAll();
	}
	void Renderer::RenderFunction() noexcept
	{
		m_time_table.ResetTime();
		try
		{
			while (!m_terminate_thread)
			{
				SetState(State::Idle);
				SetStage(Stage::Idle);

				// fine idling

				SetState(State::None);
				SetStage(Stage::None);
				m_fence_track.OpenGate(size_t(Renderer::Stage::Idle));
				mp_engine_core->GetFenceTrack().WaitForEndOfAndClose(size_t(EngineCore::Stage::Synchronization));
				if (CheckTermination()) return;
				m_time_table.ResetTable();
				m_time_table.AppendStage("wait for host");


				// Preprocess
				SetState(State::Work);
				SetStage(Stage::Preprocess);
				const auto& configs = mp_engine_core->GetLaunchConfigs(
					mp_engine_core->GetIndexer().RenderIdx()).GetConfigs();
				for (const auto& config : configs)
				{
					hipSetDevice(config.GetDeviceId());

					if (config.GetUpdateFlag())
					{
						Kernel::DepthBufferReset
							<< <
							config.GetGrid(),
							config.GetThreadBlock(),
							0u,
							mp_engine_core->GetRenderStream()
							>> >
							(mp_engine_core->GetCudaWorld(), config.GetCameraId());
						CudaErrorCheck(hipStreamSynchronize(mp_engine_core->GetRenderStream()));
						CudaErrorCheck(hipGetLastError());
					}
					m_time_table.AppendStage("buffer reset");

					Kernel::CudaCameraUpdateSamplesNumber
						<< <
						1u, 1u, 0u, mp_engine_core->GetRenderStream()
						>> >
						(mp_engine_core->GetCudaWorld(),
							config.GetCameraId(),
							config.GetUpdateFlag());
					CudaErrorCheck(hipStreamSynchronize(mp_engine_core->GetRenderStream()));
					CudaErrorCheck(hipGetLastError());
					m_time_table.AppendStage("sample update");
				}

				SetState(State::Wait);
				SetStage(Stage::None);
				m_fence_track.OpenGate(size_t(Stage::Preprocess));
				if (CheckTermination()) return;
				//mp_engine_core->GetFenceTrack().WaitForEndOfAndClose(?)


				// Main render
				SetState(State::Work);
				SetStage(Stage::MainRender);
				for (const auto& config : configs)
				{
					if (config.GetUpdateFlag())
					{
						Kernel::LaunchFirstPass
							<< <
							config.GetGrid(),
							config.GetThreadBlock(),
							config.GetSharedMemorySize(),
							mp_engine_core->GetRenderStream()
							>> >
							(mp_engine_core->GetGlobalKernel(mp_engine_core->GetIndexer().RenderIdx()),
								mp_engine_core->GetCudaWorld(),
								config.GetCameraId());
						CudaErrorCheck(hipStreamSynchronize(mp_engine_core->GetRenderStream()));
						CudaErrorCheck(hipGetLastError());
						m_time_table.AppendStage("main render");


						Kernel::SpacialReprojection
							<< <
							config.GetGrid(),
							config.GetThreadBlock(),
							0u,
							mp_engine_core->GetRenderStream()
							>> >
							(mp_engine_core->GetCudaWorld(), config.GetCameraId());
						CudaErrorCheck(hipStreamSynchronize(mp_engine_core->GetRenderStream()));
						CudaErrorCheck(hipGetLastError());
						m_time_table.AppendStage("reprojection");
					}
					else
					{
						Kernel::LaunchCumulativePass
							<< <
							config.GetGrid(),
							config.GetThreadBlock(),
							config.GetSharedMemorySize(),
							mp_engine_core->GetRenderStream()
							>> >
							(mp_engine_core->GetGlobalKernel(mp_engine_core->GetIndexer().RenderIdx()),
								mp_engine_core->GetCudaWorld(),
								config.GetCameraId());
						CudaErrorCheck(hipStreamSynchronize(mp_engine_core->GetRenderStream()));
						CudaErrorCheck(hipGetLastError());
						m_time_table.AppendStage("main render");
						m_time_table.AppendStage("reprojection");
					}
				}

				SetState(State::Wait);
				SetStage(Stage::None);
				m_fence_track.OpenGate(size_t(Stage::MainRender));
				mp_engine_core->GetFenceTrack().WaitForEndOfAndClose(size_t(EngineCore::Stage::ResultTransfer));
				if (CheckTermination()) return;
				m_time_table.AppendStage("wait for result transfer");

				// Postprocess
				SetState(State::Work);
				SetStage(Stage::Postprocess);
				for (const auto& config : configs)
				{
					Kernel::ToneMap
						<< <
						config.GetGrid(),
						config.GetThreadBlock(),
						0u,
						mp_engine_core->GetRenderStream()
						>> >
						(mp_engine_core->GetGlobalKernel(mp_engine_core->GetIndexer().RenderIdx()),
							mp_engine_core->GetCudaWorld(),
							config.GetCameraId());
					CudaErrorCheck(hipStreamSynchronize(mp_engine_core->GetRenderStream()));
					CudaErrorCheck(hipGetLastError());

					m_time_table.AppendStage("tone mapping");
				}

				m_time_table.AppendFullCycle("full render cycle");

				SetState(State::Idle);
				SetStage(Stage::Idle);
				m_fence_track.OpenGate(size_t(Stage::Postprocess));
			}
		}
		catch (const CudaException& e)
		{
			ReportCudaException(e);
		}
		catch (const Exception& e)
		{
			ReportException(e);
		}
		catch (...)
		{
			ReportException(Exception(
				"Rendering function unknown exception.",
				__FILE__, __LINE__));
		}
	}
	bool Renderer::CheckTermination()
	{
		return m_terminate_thread;
	}

	void Renderer::ReportException(const Exception& e)
	{
		if (!m_exception)
		{
			m_exception.reset(new Exception(e));
		}
	}
	void Renderer::ReportCudaException(const CudaException& e)
	{
		if (!m_cuda_exception)
		{
			m_cuda_exception.reset(new CudaException(e));
		}
	}
	void Renderer::ResetExceptions()
	{
		m_exception = nullptr;
		m_cuda_exception = nullptr;
	}
	void Renderer::ThrowIfException()
	{
		if (m_exception)
		{
			const Exception e = *m_exception;
			m_exception.reset();
			m_cuda_exception.reset();
			throw e;
		}

		if (m_cuda_exception)
		{
			const CudaException e = *m_cuda_exception;
			m_exception.reset();
			m_cuda_exception.reset();
			throw e;
		}
	}
	// ~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~


	// ~~~~~~~~ [STRUCT] EngineCore ~~~~~~~~
	EngineCore::EngineCore()
		: mp_dCudaWorld(nullptr)
		, mp_hCudaWorld(nullptr)
		, m_hpm_CudaWorld(sizeof(World))
		, m_hpm_CudaKernel(std::max(sizeof(GlobalKernel), sizeof(ConstantKernel)))
		, m_renderer(this)
		, m_fence_track(false)
	{
		hipSetDevice(0);

		CreateStreams();
		CreateGlobalKernels();
		CreateCudaWorld();
	}
	EngineCore::~EngineCore()
	{
		DestroyCudaWorld();
		DestroyGlobalKernels();
		DestroyStreams();

		//GetHardware().Reset();
	}


	void EngineCore::RenderWorld(
		RayZath::Engine::World& hWorld,
		const RayZath::Engine::RenderConfig& render_config,
		const bool block,
		const bool sync)
	{
		std::lock_guard<std::mutex> lg(m_mtx);

		// check reported exceptions and throw if any
		m_renderer.ThrowIfException();
		m_renderer.LaunchThread();

		m_core_time_table.ResetTable();
		m_core_time_table.ResetTime();


		// [>] Async reconstruction
		SetState(State::Work);
		SetStage(Stage::AsyncReconstruction);

		// update host world
		m_update_flag = hWorld.GetStateRegister().RequiresUpdate();
		mp_hWorld = &hWorld;
		hWorld.Update();
		m_core_time_table.AppendStage("hWorld update");

		// create launch configurations
		m_configs[m_indexer.UpdateIdx()].Construct(m_hardware, hWorld, m_update_flag);
		m_core_time_table.AppendStage("configs construct");

		// reconstruct cuda kernels
		m_render_config = render_config;
		ReconstructKernels();
		m_core_time_table.AppendStage("kernels reconstruct");

		m_fence_track.OpenGate(size_t(EngineCore::Stage::AsyncReconstruction));
		SetState(State::Wait);
		m_renderer.GetFenceTrack().WaitForEndOfAndClose(size_t(Renderer::Stage::MainRender));
		m_core_time_table.AppendStage("wait for main render");


		// [>] dCudaWorld async reconstruction
		SetState(State::Work);
		SetStage(Stage::WorldReconstruction);

		if (mp_hWorld->GetStateRegister().IsModified())
		{
			// reconstruct resources and objects
			CopyCudaWorldDeviceToHost();
			mp_hCudaWorld->ReconstructResources(hWorld, m_update_stream);
			mp_hCudaWorld->ReconstructObjects(hWorld, m_update_stream);
		}
		m_core_time_table.AppendStage("objects reconstruct");

		// wait for postprocess to end
		m_fence_track.OpenGate(size_t(EngineCore::Stage::WorldReconstruction));
		SetState(State::Wait);
		m_renderer.GetFenceTrack().WaitForEndOfAndClose(size_t(Renderer::Stage::Postprocess));
		m_core_time_table.AppendStage("wait for postprocess");


		// [>] dCudaWorld sync reconstruction (Camera reconstructions)
		SetState(State::Work);
		SetStage(Stage::CameraReconstruction);

		if (mp_hWorld->GetStateRegister().IsModified())
		{
			// reconstruct cameras
			mp_hCudaWorld->ReconstructCameras(hWorld, m_update_stream);
			CopyCudaWorldHostToDevice();
			mp_hWorld->GetStateRegister().MakeUnmodified();
		}
		m_core_time_table.AppendStage("cameras reconstruct");

		m_fence_track.OpenGate(size_t(EngineCore::Stage::CameraReconstruction));
		SetState(State::Wait);
		m_renderer.GetFenceTrack().WaitForEndOfAndClose(size_t(Renderer::Stage::Idle));


		// [>] Synchronize with renderer
		SetState(State::Work);
		SetStage(Stage::Synchronization);

		// swap indices
		m_indexer.Swap();
		m_render_time_table = m_renderer.GetTimeTable();

		m_fence_track.OpenGate(size_t(EngineCore::Stage::Synchronization));

		if (sync)
		{
			m_fence_track.OpenGate(size_t(EngineCore::Stage::ResultTransfer));
			SetState(State::Wait);
			m_renderer.GetFenceTrack().WaitForEndOf(size_t(Renderer::Stage::Postprocess));
			m_core_time_table.AppendStage("sync wait");
		}


		// [>] Transfer results to host side
		SetState(State::Work);
		SetStage(Stage::ResultTransfer);

		TransferResults();
		m_core_time_table.AppendStage("result tranfer");
		m_core_time_table.AppendFullCycle("full host cycle");

		SetState(State::None);
		SetStage(Stage::None);
		m_fence_track.OpenGate(size_t(EngineCore::Stage::ResultTransfer));
	}
	void EngineCore::TransferResults()
	{
		if (World::m_hpm.GetSize() < sizeof(Camera))
			ThrowException("insufficient host pinned memory for Camera");

		// [>] Get World from device
		World* hCudaWorld = (World*)m_hpm_CudaWorld.GetPointerToMemory();
		CudaErrorCheck(hipMemcpyAsync(
			hCudaWorld, mp_dCudaWorld,
			sizeof(World),
			hipMemcpyKind::hipMemcpyDeviceToHost, m_update_stream));
		CudaErrorCheck(hipStreamSynchronize(m_update_stream));

		if (hCudaWorld->cameras.GetCount() == 0u) return;	// hCudaWorld has no cameras


		const uint32_t count = std::min(
			hCudaWorld->cameras.GetCount(),
			mp_hWorld->Container<RayZath::Engine::World::ContainerType::Camera>().GetCount());
		for (uint32_t i = 0u; i < count; ++i)
		{
			// check if hostCamera is enabled
			const auto& hCamera = mp_hWorld->Container<RayZath::Engine::World::ContainerType::Camera>()[i];
			if (!hCamera) continue;	// no camera at this address
			if (!hCamera->Enabled()) continue;	// camera is disabled

			// [>] Get Camera class from hCudaWorld
			Camera* hCudaCamera = (Camera*)World::m_hpm.GetPointerToMemory();
			CudaErrorCheck(hipMemcpyAsync(
				hCudaCamera, &hCudaWorld->cameras[i],
				sizeof(Camera),
				hipMemcpyKind::hipMemcpyDeviceToHost, m_update_stream));
			CudaErrorCheck(hipStreamSynchronize(m_update_stream));


			// [>] Asynchronous copying
			hCamera->m_samples_count = hCudaCamera->GetPassesCount();

			static_assert(
				sizeof(*hCamera->GetImageBuffer().GetMapAddress()) ==
				sizeof(Color<unsigned char>),
				"sizeof(Graphics::Color) != sizeof(Color<unsigned char>)");

			// check cameras resolution
			if (hCamera->GetWidth() != hCudaCamera->GetWidth() ||
				hCamera->GetHeight() != hCudaCamera->GetHeight()) continue;

			uint32_t chunkSize = uint32_t(
				hCudaCamera->hostPinnedMemory.GetSize() /
				(sizeof(Color<unsigned char>)));
			if (chunkSize < 1024u) ThrowException("Not enough host pinned memory for async image copy");

			uint32_t nPixels = hCamera->GetWidth() * hCamera->GetHeight();
			for (uint32_t startIndex = 0; startIndex < nPixels; startIndex += chunkSize)
			{
				// find start index
				if (startIndex + chunkSize > nPixels) chunkSize = nPixels - startIndex;

				// find offset point
				Graphics::Point<uint32_t> offset_point(
					startIndex % hCamera->GetWidth(),
					startIndex / hCamera->GetWidth());

				// copy final image data from hCudaCamera to hCudaPixels on pinned memory
				Color<unsigned char>* hCudaPixels =
					(Color<unsigned char>*)Camera::hostPinnedMemory.GetPointerToMemory();
				CudaErrorCheck(cudaMemcpyFromArrayAsync(
					hCudaPixels, hCudaCamera->FinalImageBuffer().GetCudaArray(),
					offset_point.x * sizeof(*hCudaPixels), offset_point.y,
					chunkSize * sizeof(*hCudaPixels),
					hipMemcpyKind::hipMemcpyDeviceToHost, m_update_stream));
				CudaErrorCheck(hipStreamSynchronize(m_update_stream));

				// copy final image data from hostCudaPixels on pinned memory to hostCamera
				hCamera->m_image_buffer.CopyFromMemory(
					hCudaPixels,
					chunkSize * sizeof(*hCudaPixels),
					offset_point.x, offset_point.y);


				// [>] Copy depth buffer
				float* hCudaDepthData =
					(float*)Camera::hostPinnedMemory.GetPointerToMemory();
				CudaErrorCheck(cudaMemcpyFromArrayAsync(
					hCudaDepthData, hCudaCamera->FinalDepthBuffer().GetCudaArray(),
					offset_point.x * sizeof(*hCudaDepthData), offset_point.y,
					chunkSize * sizeof(*hCudaDepthData),
					hipMemcpyKind::hipMemcpyDeviceToHost, m_update_stream));
				CudaErrorCheck(hipStreamSynchronize(m_update_stream));

				// copy final image data from hostCudaPixels on pinned memory to hostCamera
				hCamera->m_depth_buffer.CopyFromMemory(
					hCudaDepthData,
					chunkSize * sizeof(*hCudaDepthData),
					offset_point.x, offset_point.y);
			}
		}
	}


	void EngineCore::CreateStreams()
	{
		CudaErrorCheck(hipStreamCreate(&m_update_stream));
		CudaErrorCheck(hipStreamCreate(&m_render_stream));
	}
	void EngineCore::DestroyStreams()
	{
		CudaErrorCheck(hipStreamDestroy(m_update_stream));
		CudaErrorCheck(hipStreamDestroy(m_render_stream));
	}
	void EngineCore::CreateGlobalKernels()
	{
		GlobalKernel* hCudaGlobalKernel =
			(GlobalKernel*)m_hpm_CudaKernel.GetPointerToMemory();
		for (uint32_t i = 0u; i < 2u; ++i)
		{
			new (hCudaGlobalKernel) GlobalKernel();

			CudaErrorCheck(hipMalloc(
				(void**)&mp_global_kernel[i], sizeof(GlobalKernel)));
			CudaErrorCheck(hipMemcpy(mp_global_kernel[i], hCudaGlobalKernel,
				sizeof(GlobalKernel), hipMemcpyKind::hipMemcpyHostToDevice));
		}
	}
	void EngineCore::DestroyGlobalKernels()
	{
		GlobalKernel* hCudaKernelData =
			(GlobalKernel*)m_hpm_CudaKernel.GetPointerToMemory();
		for (uint32_t i = 0u; i < 2u; ++i)
		{
			CudaErrorCheck(hipMemcpy(
				hCudaKernelData, mp_global_kernel[i],
				sizeof(GlobalKernel),
				hipMemcpyKind::hipMemcpyDeviceToHost));

			hCudaKernelData->~GlobalKernel();

			CudaErrorCheck(hipFree(mp_global_kernel[i]));
			mp_global_kernel[i] = nullptr;
		}
	}
	void EngineCore::ReconstructKernels()
	{
		// [>] GlobalKernel
		// get hpm memory
		GlobalKernel* hCudaGlobalKernel =
			(GlobalKernel*)m_hpm_CudaKernel.GetPointerToMemory();

		// copy dCudaKernelData to host
		CudaErrorCheck(hipMemcpyAsync(
			hCudaGlobalKernel,
			mp_global_kernel[m_indexer.UpdateIdx()],
			sizeof(GlobalKernel),
			hipMemcpyKind::hipMemcpyDeviceToHost,
			m_update_stream));
		CudaErrorCheck(hipStreamSynchronize(m_update_stream));

		// reconstruct hCudaGlobalKernel
		hCudaGlobalKernel->Reconstruct(
			m_indexer.UpdateIdx(),
			m_update_stream);

		// copy hCudaGlobalKernel to device
		CudaErrorCheck(hipMemcpyAsync(
			mp_global_kernel[m_indexer.UpdateIdx()],
			hCudaGlobalKernel,
			sizeof(GlobalKernel),
			hipMemcpyKind::hipMemcpyHostToDevice,
			m_update_stream));
		CudaErrorCheck(hipStreamSynchronize(m_update_stream));


		// [>] ConstantKernel
		// get hpm memory
		ConstantKernel* hCudaConstantKernel =
			(ConstantKernel*)m_hpm_CudaKernel.GetPointerToMemory();

		// reconstruct hCudaConstantKernel
		hCudaConstantKernel->Reconstruct(
			m_render_config);

		// copy hCudaConstantKernel to device __constant__ memory
		Kernel::CopyToConstantMemory(
			hCudaConstantKernel,
			m_indexer.UpdateIdx(), m_update_stream);
	}
	void EngineCore::CreateCudaWorld()
	{
		mp_hCudaWorld = (World*)m_hpm_CudaWorld.GetPointerToMemory();
		new (mp_hCudaWorld) World();
		CudaErrorCheck(hipMalloc(&mp_dCudaWorld, sizeof(World)));
		CopyCudaWorldHostToDevice();
	}
	void EngineCore::DestroyCudaWorld()
	{
		if (mp_dCudaWorld)
		{
			CopyCudaWorldDeviceToHost();
			mp_hCudaWorld->~World();
			CudaErrorCheck(hipFree(mp_dCudaWorld));
			mp_dCudaWorld = nullptr;
		}
	}
	void EngineCore::CopyCudaWorldDeviceToHost()
	{
		mp_hCudaWorld = (World*)m_hpm_CudaWorld.GetPointerToMemory();
		CudaErrorCheck(hipMemcpyAsync(
			mp_hCudaWorld, mp_dCudaWorld,
			sizeof(World),
			hipMemcpyKind::hipMemcpyDeviceToHost, m_update_stream));
		CudaErrorCheck(hipStreamSynchronize(m_update_stream));
	}
	void EngineCore::CopyCudaWorldHostToDevice()
	{
		if (mp_dCudaWorld && mp_hCudaWorld)
		{
			CudaErrorCheck(hipMemcpyAsync(
				mp_dCudaWorld, mp_hCudaWorld,
				sizeof(World),
				hipMemcpyKind::hipMemcpyHostToDevice, m_update_stream));
			CudaErrorCheck(hipStreamSynchronize(m_update_stream));
		}
	}


	Hardware& EngineCore::GetHardware()
	{
		return m_hardware;
	}
	Indexer& EngineCore::GetIndexer()
	{
		return m_indexer;
	}

	Renderer& EngineCore::GetRenderer()
	{
		return m_renderer;
	}
	LaunchConfigurations& EngineCore::GetLaunchConfigs(const bool idx)
	{
		return m_configs[idx];
	}
	GlobalKernel* EngineCore::GetGlobalKernel(const bool idx)
	{
		return mp_global_kernel[idx];
	}
	World* EngineCore::GetCudaWorld()
	{
		return mp_dCudaWorld;
	}
	EngineCore::FenceTrack_t& EngineCore::GetFenceTrack()
	{
		return m_fence_track;
	}
	const TimeTable& EngineCore::GetCoreTimeTable() const
	{
		return m_core_time_table;
	}
	const TimeTable& EngineCore::GetRenderTimeTable() const
	{
		return m_render_time_table;
	}

	hipStream_t& EngineCore::GetUpdateStream()
	{
		return m_update_stream;
	}
	hipStream_t& EngineCore::GetRenderStream()
	{
		return m_render_stream;
	}

	const EngineCore::State& EngineCore::GetState()
	{
		return m_state;
	}
	const EngineCore::Stage& EngineCore::GetStage()
	{
		return m_stage;
	}
	void EngineCore::SetState(const EngineCore::State& state)
	{
		m_state = state;
	}
	void EngineCore::SetStage(const EngineCore::Stage& stage)
	{
		m_stage = stage;
	}
}