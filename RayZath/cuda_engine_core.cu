#include "hip/hip_runtime.h"
#include "cuda_engine_core.cuh"

#include "cuda_preprocess_kernel.cuh"
#include "cuda_render_kernel.cuh"
#include "cuda_postprocess_kernel.cuh"

#include "point.h"

#include <algorithm>

namespace RayZath
{
	namespace CudaEngine
	{
		// ~~~~~~~~ [STRUCT] CudaIndexer ~~~~~~~~
		CudaIndexer::CudaIndexer()
			: m_update_idx(0u)
			, m_render_idx(1u)
		{}

		const bool& CudaIndexer::UpdateIdx() const
		{
			return m_update_idx;
		}
		const bool& CudaIndexer::RenderIdx() const
		{
			return m_render_idx;
		}
		void CudaIndexer::Swap()
		{
			std::swap(m_update_idx, m_render_idx);
		}
		// ~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~


		// ~~~~~~~~ [STRUCT] TimeTable ~~~~~~~~
		TimeTable::TimeTable()
		{
			m_timer.Start();
		}

		void TimeTable::AppendStage(const std::string& s)
		{
			m_stamps.push_back({ s, m_timer.GetTime() });
		}
		void TimeTable::AppendFullCycle(const std::string& s)
		{
			m_stamps.push_back({ s, m_cycle_timer.GetTime() });
		}
		void TimeTable::ResetTime()
		{
			m_timer.Start();
			m_cycle_timer.Start();
		}
		void TimeTable::ResetTable()
		{
			m_stamps.clear();
		}
		std::string TimeTable::ToString(const uint32_t width) const
		{
			std::stringstream ss;
			for (auto& stamp : m_stamps)
			{
				ss.fill(' ');
				ss.width(width);
				ss << stamp.first << ": ";

				ss.width();
				ss.precision(3);
				ss << std::fixed << stamp.second << "ms\n";
			}
			return ss.str();
		}
		// ~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~


		// ~~~~~~~~ [STRUCT] CudaRenderer ~~~~~~~~
		CudaRenderer::CudaRenderer(CudaEngineCore* const engine_core)
			: mp_engine_core(engine_core)
			, m_is_thread_alive(false)
			, m_terminate_thread(false)
			, m_state(State::None)
			, m_stage(Stage::None)
			, mp_blocking_gate(nullptr)
			, m_fence_track(true)
		{}
		CudaRenderer::~CudaRenderer()
		{
			TerminateThread();
		}

		void CudaRenderer::LaunchThread()
		{
			std::lock_guard<std::mutex> lg(m_mtx);
			if (!m_is_thread_alive)
			{
				m_terminate_thread = false;
				m_is_thread_alive = true;
				mp_blocking_gate = nullptr;
				ResetExceptions();
				mp_render_thread = std::make_unique<std::thread>(
					&CudaRenderer::RenderFunctionWrapper, 
					this);
			}			
		}
		void CudaRenderer::TerminateThread()
		{
			{
				std::lock_guard<std::mutex> lg(m_mtx);
				m_terminate_thread = true;
			}
			if (m_is_thread_alive)
			{
				mp_engine_core->GetFenceTrack().OpenAll();
				//if (mp_blocking_gate)
				//	mp_blocking_gate->Open();

				if (mp_render_thread->joinable())
					mp_render_thread->join();

				m_is_thread_alive = false;
				mp_render_thread = nullptr;
			}
		}

		FenceTrack<5>& CudaRenderer::GetFenceTrack()
		{
			return m_fence_track;
		}
		const TimeTable& CudaRenderer::GetTimeTable() const
		{
			return m_time_table;
		}
		const CudaRenderer::State& CudaRenderer::GetState() const
		{
			return m_state;
		}
		const CudaRenderer::Stage& CudaRenderer::GetStage() const
		{
			return m_stage;
		}

		void CudaRenderer::SetState(const CudaRenderer::State& state)
		{
			m_state = state;
		}
		void CudaRenderer::SetStage(const CudaRenderer::Stage& stage)
		{
			m_stage = stage;
		}

		void CudaRenderer::RenderFunctionWrapper()
		{
			RenderFunction();
			std::lock_guard<std::mutex> lg(m_mtx);
			m_is_thread_alive = false;
			m_state = State::None;
			m_stage = Stage::None;
			m_fence_track.OpenAll();
		}
		void CudaRenderer::RenderFunction() noexcept
		{
			m_time_table.ResetTime();
			try
			{
				while (!m_terminate_thread)
				{
					SetState(State::Idle);
					SetStage(Stage::Idle);

					// fine idling

					SetState(State::None);
					SetStage(Stage::None);
					m_fence_track.OpenGate(size_t(CudaRenderer::Stage::Idle));
					mp_engine_core->GetFenceTrack().WaitForEndOf(size_t(CudaEngineCore::Stage::CameraReconstruction));
					if (CheckTermination()) return;
					m_time_table.ResetTable();
					m_time_table.AppendStage("wait for host");


					// Preprocess
					SetState(State::Work);
					SetStage(Stage::Preprocess);
					const auto& configs = mp_engine_core->GetLaunchConfigs(
						mp_engine_core->GetIndexer().RenderIdx()).GetConfigs();
					for (const auto& config : configs)
					{
						hipSetDevice(config.GetDeviceId());

						if (config.GetUpdateFlag())
						{
							CudaKernel::DepthBufferReset
								<< <
								config.GetGrid(),
								config.GetThreadBlock(),
								0u,
								mp_engine_core->GetRenderStream()
								>> >
								(mp_engine_core->GetCudaWorld(), config.GetCameraId());
							CudaErrorCheck(hipStreamSynchronize(mp_engine_core->GetRenderStream()));
							CudaErrorCheck(hipGetLastError());
						}
						m_time_table.AppendStage("buffer reset");

						CudaKernel::CudaCameraUpdateSamplesNumber
							<< <
							1u, 1u, 0u, mp_engine_core->GetRenderStream()
							>> >
							(mp_engine_core->GetCudaWorld(),
								config.GetCameraId(),
								config.GetUpdateFlag());
						CudaErrorCheck(hipStreamSynchronize(mp_engine_core->GetRenderStream()));
						CudaErrorCheck(hipGetLastError());
						m_time_table.AppendStage("sample update");
					}

					SetState(State::Wait);
					SetStage(Stage::None);
					m_fence_track.OpenGate(size_t(Stage::Preprocess)); 
					if (CheckTermination()) return;
					//mp_engine_core->GetFenceTrack().WaitForEndOf(?)
						

					// Main render
					SetState(State::Work);
					SetStage(Stage::MainRender);
					for (const auto& config : configs)
					{
						if (config.GetUpdateFlag())
						{
							CudaKernel::LaunchFirstPass
								<< <
								config.GetGrid(),
								config.GetThreadBlock(),
								config.GetSharedMemorySize(),
								mp_engine_core->GetRenderStream()
								>> >
								(mp_engine_core->GetGlobalKernel(mp_engine_core->GetIndexer().RenderIdx()),
									mp_engine_core->GetCudaWorld(),
									config.GetCameraId());
							CudaErrorCheck(hipStreamSynchronize(mp_engine_core->GetRenderStream()));
							CudaErrorCheck(hipGetLastError());
							m_time_table.AppendStage("main render");


							CudaKernel::SpacialReprojection
								<< <
								config.GetGrid(),
								config.GetThreadBlock(),
								0u,
								mp_engine_core->GetRenderStream()
								>> >
								(mp_engine_core->GetCudaWorld(), config.GetCameraId());
							CudaErrorCheck(hipStreamSynchronize(mp_engine_core->GetRenderStream()));
							CudaErrorCheck(hipGetLastError());
							m_time_table.AppendStage("reprojection");
						}
						else
						{
							CudaKernel::LaunchCumulativePass
								<< <
								config.GetGrid(),
								config.GetThreadBlock(),
								config.GetSharedMemorySize(),
								mp_engine_core->GetRenderStream()
								>> >
								(mp_engine_core->GetGlobalKernel(mp_engine_core->GetIndexer().RenderIdx()),
									mp_engine_core->GetCudaWorld(),
									config.GetCameraId());
							CudaErrorCheck(hipStreamSynchronize(mp_engine_core->GetRenderStream()));
							CudaErrorCheck(hipGetLastError());
							m_time_table.AppendStage("main render");
							m_time_table.AppendStage("reprojection");
						}
					}

					SetState(State::Wait);
					SetStage(Stage::None);
					m_fence_track.OpenGate(size_t(Stage::MainRender));
					mp_engine_core->GetFenceTrack().WaitForEndOf(size_t(CudaEngineCore::Stage::ResultTransfer));
					if (CheckTermination()) return;
					m_time_table.AppendStage("wait for postprocess");

					// Postprocess
					SetState(State::Work);
					SetStage(Stage::Postprocess);
					for (const auto& config : configs)
					{
						CudaKernel::ToneMap
							<< <
							config.GetGrid(),
							config.GetThreadBlock(),
							0u,
							mp_engine_core->GetRenderStream()
							>> >
							(mp_engine_core->GetGlobalKernel(mp_engine_core->GetIndexer().RenderIdx()),
								mp_engine_core->GetCudaWorld(), 
								config.GetCameraId());
						CudaErrorCheck(hipStreamSynchronize(mp_engine_core->GetRenderStream()));
						CudaErrorCheck(hipGetLastError());

						m_time_table.AppendStage("tone mapping");
					}

					m_time_table.AppendFullCycle("full render cycle");

					SetState(State::Idle);
					SetStage(Stage::Idle);
					m_fence_track.OpenGate(size_t(Stage::Postprocess));
				}
			}
			catch (const CudaException& e)
			{
				ReportCudaException(e);
			}
			catch (const Exception& e)
			{
				ReportException(e);
			}
			catch (...)
			{
				ReportException(
					Exception(__FILE__, __LINE__, L"Rendering function unknown fail."));
			}
		}
		bool CudaRenderer::CheckTermination()
		{
			return m_terminate_thread;
		}

		void CudaRenderer::ReportException(const Exception& e)
		{
			if (!m_exception)
			{
				m_exception.reset(new Exception(e));
			}
		}
		void CudaRenderer::ReportCudaException(const CudaException& e)
		{
			if (!m_cuda_exception)
			{
				m_cuda_exception.reset(new CudaException(e));
			}
		}
		void CudaRenderer::ResetExceptions()
		{
			m_exception = nullptr;
			m_cuda_exception = nullptr;
		}
		void CudaRenderer::ThrowIfException()
		{
			if (m_exception)
			{
				const Exception e = *m_exception;
				m_exception.reset();
				m_cuda_exception.reset();
				throw e;
			}

			if (m_cuda_exception)
			{
				const CudaException e = *m_cuda_exception;
				m_exception.reset();
				m_cuda_exception.reset();
				throw e;
			}
		}
		// ~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~


		// ~~~~~~~~ [STRUCT] CudaEngineCore ~~~~~~~~
		CudaEngineCore::CudaEngineCore()
			: mp_dCudaWorld(nullptr)
			, m_hpm_CudaWorld(sizeof(CudaWorld))
			, m_hpm_CudaKernel(std::max(sizeof(CudaGlobalKernel), sizeof(CudaConstantKernel)))
			, m_renderer(this)
			, m_fence_track(false)
		{
			hipSetDevice(0);

			CreateStreams();
			CreateGlobalKernels();
			CreateCudaWorld();
		}
		CudaEngineCore::~CudaEngineCore()
		{
			DestroyCudaWorld();
			DestroyGlobalKernels();
			DestroyStreams();

			GetHardware().Reset();
		}


		void CudaEngineCore::RenderWorld(
			World& hWorld,
			const bool block,
			const bool sync)
		{
			std::lock_guard<std::mutex> lg(m_mtx);

			// check reported exceptions and throw if any
			m_renderer.ThrowIfException();
			m_renderer.LaunchThread();

			m_core_time_table.ResetTable();
			m_core_time_table.ResetTime();


			// [>] Async reconstruction
			SetState(State::Work);
			SetStage(Stage::AsyncReconstruction);

			// update host world
			m_update_flag = hWorld.GetStateRegister().RequiresUpdate();
			mp_hWorld = &hWorld;
			hWorld.Update();
			m_core_time_table.AppendStage("hWorld update");

			// create launch configurations
			m_configs[m_indexer.UpdateIdx()].Construct(m_hardware, hWorld, m_update_flag);
			m_core_time_table.AppendStage("configs construct");

			// reconstruct cuda kernels
			ReconstructKernels();
			m_core_time_table.AppendStage("kernels reconstruct");

			SetState(State::Wait);
			SetStage(Stage::None);
			m_fence_track.OpenGate(size_t(CudaEngineCore::Stage::AsyncReconstruction));
			m_renderer.GetFenceTrack().WaitForEndOf(size_t(CudaRenderer::Stage::MainRender));
			m_core_time_table.AppendStage("wait for main render");


			// [>] dCudaWorld async reconstruction
			SetState(State::Work);
			SetStage(Stage::WorldReconstruction);

			// reconstruct dCudaWorld
			//ReconstructCudaWorld();	// make this reconstruct all except cameras

			SetState(State::Wait);
			SetStage(Stage::None);
			m_fence_track.OpenGate(size_t(CudaEngineCore::Stage::WorldReconstruction));
			m_renderer.GetFenceTrack().WaitForEndOf(size_t(CudaRenderer::Stage::Postprocess));
			m_core_time_table.AppendStage("wait for postprocess");


			// [>] dCudaWorld sync reconstruction (CudaCamera reconstructions)
			SetState(State::Work);
			SetStage(Stage::CameraReconstruction);

			// reconstruct 
			ReconstructCudaWorld();	// TODO: make this rconstruct only cameras
			m_core_time_table.AppendStage("dCudaWorld reconstruct");

			// swap indices
			m_indexer.Swap();
			m_render_time_table = m_renderer.GetTimeTable();

			SetState(State::Wait);
			SetStage(Stage::None);
			m_fence_track.OpenGate(size_t(CudaEngineCore::Stage::CameraReconstruction));

			//m_fence_track.OpenGate(size_t(CudaEngineCore::Stage::ResultTransfer));
			//m_renderer.GetFenceTrack().WaitForEndOf(size_t(CudaRenderer::Stage::Postprocess));
			//m_time_table.AppendStage("sync wait");
			//m_renderer.GetFenceTrack().OpenGate(size_t(CudaRenderer::Stage::Postprocess));


			// [>] Transfer results to host side
			SetState(State::Work);
			SetStage(Stage::ResultTransfer);

			TransferResults();
			m_core_time_table.AppendStage("result tranfer");
			m_core_time_table.AppendFullCycle("full host cycle");

			SetState(State::None);
			SetStage(Stage::None);
			m_fence_track.OpenGate(size_t(CudaEngineCore::Stage::ResultTransfer));
		}
		void CudaEngineCore::TransferResults()
		{
			for (uint32_t i = 0u; i < mp_hWorld->Container<Camera>().GetCapacity(); ++i)
			{
				// check if hostCamera does exict
				const Handle<Camera>& hCamera = mp_hWorld->Container<Camera>()[i];
				if (!hCamera) continue;	// no camera at this address
				if (!hCamera->Enabled()) continue;	// camera is disabled


				// [>] Get CudaWorld from device
				CudaWorld* hCudaWorld = (CudaWorld*)m_hpm_CudaWorld.GetPointerToMemory();
				CudaErrorCheck(hipMemcpyAsync(
					hCudaWorld, mp_dCudaWorld,
					sizeof(CudaWorld),
					hipMemcpyKind::hipMemcpyDeviceToHost, m_update_stream));
				CudaErrorCheck(hipStreamSynchronize(m_update_stream));

				if (hCudaWorld->cameras.GetCount() == 0u) return;	// hCudaWorld has no cameras


				// [>] Get CudaCamera class from hCudaWorld
				CudaCamera* hCudaCamera = nullptr;
				if (CudaWorld::m_hpm.GetSize() < sizeof(*hCudaCamera))
					ThrowException(L"insufficient host pinned memory for CudaCamera");
				hCudaCamera = (CudaCamera*)CudaWorld::m_hpm.GetPointerToMemory();

				CudaErrorCheck(hipMemcpyAsync(
					hCudaCamera, &hCudaWorld->cameras[i],
					sizeof(CudaCamera),
					hipMemcpyKind::hipMemcpyDeviceToHost, m_update_stream));
				CudaErrorCheck(hipStreamSynchronize(m_update_stream));

				if (!hCudaCamera->Exist()) continue;


				// [>] Asynchronous copying
				hCamera->m_samples_count = hCudaCamera->GetPassesCount();

				static_assert(
					sizeof(*hCamera->GetImageBuffer().GetMapAddress()) ==
					sizeof(Color<unsigned char>),
					"sizeof(Graphics::Color) != sizeof(Color<unsigned char>)");

				// check cameras resolution
				if (hCamera->GetWidth() != hCudaCamera->GetWidth() ||
					hCamera->GetHeight() != hCudaCamera->GetHeight()) continue;

				uint32_t chunkSize =
					hCudaCamera->hostPinnedMemory.GetSize() /
					(sizeof(Color<unsigned char>));
				if (chunkSize < 16u) ThrowException(L"Not enough host pinned memory for async image copy");

				uint32_t nPixels = hCamera->GetWidth() * hCamera->GetHeight();
				for (uint32_t startIndex = 0; startIndex < nPixels; startIndex += chunkSize)
				{
					// find start index
					if (startIndex + chunkSize > nPixels) chunkSize = nPixels - startIndex;

					// find offset point
					Graphics::Point<uint32_t> offset_point(
						startIndex % hCamera->GetWidth(),
						startIndex / hCamera->GetWidth());

					// copy final image data from hCudaCamera to hCudaPixels on pinned memory
					Color<unsigned char>* hCudaPixels =
						(Color<unsigned char>*)CudaCamera::hostPinnedMemory.GetPointerToMemory();
					CudaErrorCheck(cudaMemcpyFromArrayAsync(
						hCudaPixels, hCudaCamera->FinalImageBuffer(m_indexer.UpdateIdx()).GetCudaArray(),
						offset_point.x * sizeof(*hCudaPixels), offset_point.y,
						chunkSize * sizeof(*hCudaPixels),
						hipMemcpyKind::hipMemcpyDeviceToHost, m_update_stream));
					CudaErrorCheck(hipStreamSynchronize(m_update_stream));

					// copy final image data from hostCudaPixels on pinned memory to hostCamera
					hCamera->m_image_buffer.CopyFromMemory(
						hCudaPixels,
						chunkSize * sizeof(*hCudaPixels),
						offset_point.x, offset_point.y);


					// [>] Copy depth buffer
					float* hCudaDepthData =
						(float*)CudaCamera::hostPinnedMemory.GetPointerToMemory();
					CudaErrorCheck(cudaMemcpyFromArrayAsync(
						hCudaDepthData, hCudaCamera->FinalDepthBuffer(m_indexer.UpdateIdx()).GetCudaArray(),
						offset_point.x * sizeof(*hCudaDepthData), offset_point.y,
						chunkSize * sizeof(*hCudaDepthData),
						hipMemcpyKind::hipMemcpyDeviceToHost, m_update_stream));
					CudaErrorCheck(hipStreamSynchronize(m_update_stream));

					// copy final image data from hostCudaPixels on pinned memory to hostCamera
					hCamera->m_depth_buffer.CopyFromMemory(
						hCudaDepthData,
						chunkSize * sizeof(*hCudaDepthData),
						offset_point.x, offset_point.y);
				}
			}
		}

		

		void CudaEngineCore::CreateStreams()
		{
			CudaErrorCheck(hipStreamCreate(&m_update_stream));
			CudaErrorCheck(hipStreamCreate(&m_render_stream));			
		}
		void CudaEngineCore::DestroyStreams()
		{
			CudaErrorCheck(hipStreamDestroy(m_update_stream));
			CudaErrorCheck(hipStreamDestroy(m_render_stream));
		}
		void CudaEngineCore::CreateGlobalKernels()
		{
			CudaGlobalKernel* hCudaGlobalKernel =
				(CudaGlobalKernel*)m_hpm_CudaKernel.GetPointerToMemory();
			for (uint32_t i = 0u; i < 2u; ++i)
			{
				new (hCudaGlobalKernel) CudaGlobalKernel();

				CudaErrorCheck(hipMalloc(
					(void**)&mp_global_kernel[i], sizeof(CudaGlobalKernel)));
				CudaErrorCheck(hipMemcpy(mp_global_kernel[i], hCudaGlobalKernel,
					sizeof(CudaGlobalKernel), hipMemcpyKind::hipMemcpyHostToDevice));
			}
		}
		void CudaEngineCore::DestroyGlobalKernels()
		{
			CudaGlobalKernel* hCudaKernelData =
				(CudaGlobalKernel*)m_hpm_CudaKernel.GetPointerToMemory();
			for (uint32_t i = 0u; i < 2u; ++i)
			{
				CudaErrorCheck(hipMemcpy(
					hCudaKernelData, mp_global_kernel[i],
					sizeof(CudaGlobalKernel),
					hipMemcpyKind::hipMemcpyDeviceToHost));

				hCudaKernelData->~CudaGlobalKernel();

				CudaErrorCheck(hipFree(mp_global_kernel[i]));
				mp_global_kernel[i] = nullptr;
			}
		}
		void CudaEngineCore::ReconstructKernels()
		{
			// [>] CudaGlobalKernel
			// get hpm memory
			CudaGlobalKernel* hCudaGlobalKernel =
				(CudaGlobalKernel*)m_hpm_CudaKernel.GetPointerToMemory();

			// copy dCudaKernelData to host
			CudaErrorCheck(hipMemcpyAsync(
				hCudaGlobalKernel,
				mp_global_kernel[m_indexer.UpdateIdx()],
				sizeof(CudaGlobalKernel),
				hipMemcpyKind::hipMemcpyDeviceToHost,
				m_update_stream));
			CudaErrorCheck(hipStreamSynchronize(m_update_stream));

			// reconstruct hCudaGlobalKernel
			hCudaGlobalKernel->Reconstruct(
				m_indexer.UpdateIdx(),
				m_update_stream);

			// copy hCudaGlobalKernel to device
			CudaErrorCheck(hipMemcpyAsync(
				mp_global_kernel[m_indexer.UpdateIdx()],
				hCudaGlobalKernel,
				sizeof(CudaGlobalKernel),
				hipMemcpyKind::hipMemcpyHostToDevice,
				m_update_stream));
			CudaErrorCheck(hipStreamSynchronize(m_update_stream));


			// [>] CudaConstantKernel
			// get hpm memory
			CudaConstantKernel* hCudaConstantKernel =
				(CudaConstantKernel*)m_hpm_CudaKernel.GetPointerToMemory();

			// reconstruct hCudaConstantKernel
			hCudaConstantKernel->Reconstruct();

			// copy hCudaConstantKernel to device __constant__ memory
			CudaKernel::CopyToConstantMemory(
				hCudaConstantKernel,
				m_indexer.UpdateIdx(), m_update_stream);
		}
		void CudaEngineCore::CreateCudaWorld()
		{
			CudaWorld* hCudaWorld = (CudaWorld*)m_hpm_CudaWorld.GetPointerToMemory();
			new (hCudaWorld) CudaWorld();
			CudaErrorCheck(hipMalloc(&mp_dCudaWorld, sizeof(CudaWorld)));
			CudaErrorCheck(hipMemcpy(
				mp_dCudaWorld, hCudaWorld,
				sizeof(CudaWorld),
				hipMemcpyKind::hipMemcpyHostToDevice));
		}
		void CudaEngineCore::DestroyCudaWorld()
		{
			if (mp_dCudaWorld)
			{
				CudaWorld* hCudaWorld = (CudaWorld*)m_hpm_CudaWorld.GetPointerToMemory();
				CudaErrorCheck(hipMemcpy(
					hCudaWorld, mp_dCudaWorld,
					sizeof(CudaWorld),
					hipMemcpyKind::hipMemcpyDeviceToHost));

				hCudaWorld->~CudaWorld();

				CudaErrorCheck(hipFree(mp_dCudaWorld));
				mp_dCudaWorld = nullptr;
			}
		}
		void CudaEngineCore::ReconstructCudaWorld()
		{
			if (!mp_hWorld->GetStateRegister().IsModified()) return;

			// copy CudaWorld to host
			CudaWorld* hCudaWorld = (CudaWorld*)m_hpm_CudaWorld.GetPointerToMemory();
			CudaErrorCheck(hipMemcpyAsync(
				hCudaWorld, mp_dCudaWorld,
				sizeof(CudaWorld),
				hipMemcpyKind::hipMemcpyDeviceToHost, m_update_stream));
			CudaErrorCheck(hipStreamSynchronize(m_update_stream));

			// reconstruct CudaWorld on host
			hCudaWorld->Reconstruct(*mp_hWorld, m_update_stream);

			// copy CudaWorld back to device
			CudaErrorCheck(hipMemcpyAsync(
				mp_dCudaWorld, hCudaWorld,
				sizeof(CudaWorld),
				hipMemcpyKind::hipMemcpyHostToDevice, m_update_stream));
			CudaErrorCheck(hipStreamSynchronize(m_update_stream));

			mp_hWorld->GetStateRegister().MakeUnmodified();
		}


		CudaHardware& CudaEngineCore::GetHardware()
		{
			return m_hardware;
		}
		CudaIndexer& CudaEngineCore::GetIndexer()
		{
			return m_indexer;
		}

		CudaRenderer& CudaEngineCore::GetRenderer()
		{
			return m_renderer;
		}
		LaunchConfigurations& CudaEngineCore::GetLaunchConfigs(const bool idx)
		{
			return m_configs[idx];
		}
		CudaGlobalKernel* CudaEngineCore::GetGlobalKernel(const bool idx)
		{
			return mp_global_kernel[idx];
		}
		CudaWorld* CudaEngineCore::GetCudaWorld()
		{
			return mp_dCudaWorld;
		}
		FenceTrack<5>& CudaEngineCore::GetFenceTrack()
		{
			return m_fence_track;
		}
		const TimeTable& CudaEngineCore::GetCoreTimeTable() const
		{
			return m_core_time_table;
		}
		const TimeTable& CudaEngineCore::GetRenderTimeTable() const
		{
			return m_render_time_table;
		}

		hipStream_t& CudaEngineCore::GetUpdateStream()
		{
			return m_update_stream;
		}
		hipStream_t& CudaEngineCore::GetRenderStream()
		{
			return m_render_stream;
		}

		const CudaEngineCore::State& CudaEngineCore::GetState()
		{
			return m_state;
		}
		const CudaEngineCore::Stage& CudaEngineCore::GetStage()
		{
			return m_stage;
		}
		void CudaEngineCore::SetState(const CudaEngineCore::State& state)
		{
			m_state = state;
		}
		void CudaEngineCore::SetStage(const CudaEngineCore::Stage& stage)
		{
			m_stage = stage;
		}
		// ~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~
	}
}