#include "cuda_material.cuh"
#include "cuda_world.cuh"

namespace RayZath
{
	namespace CudaEngine
	{
		// ~~~~~~~~ [STRUCT] CudaMaterial ~~~~~~~~
		CudaMaterial& CudaMaterial::operator=(const Material& hMaterial)
		{
			color = hMaterial.GetColor();
			reflectance = hMaterial.GetReflectance();
			glossiness = hMaterial.GetReflectance();
			transmittance = hMaterial.GetTransmittance();
			ior = hMaterial.GetIndexOfRefraction();
			emittance = hMaterial.GetEmittance();
			scattering = hMaterial.GetScattering();

			texture = nullptr;
			emittance_map = nullptr;

			return *this;
		}
		void CudaMaterial::Reconstruct(
			const CudaWorld& hCudaWorld,
			const Handle<Material>& hMaterial,
			hipStream_t& mirror_stream)
		{
			if (!hMaterial->GetStateRegister().IsModified()) return;

			// material properties
			color = hMaterial->GetColor();
			reflectance = hMaterial->GetReflectance();
			glossiness = hMaterial->GetGlossiness();
			transmittance = hMaterial->GetTransmittance();
			ior = hMaterial->GetIndexOfRefraction();
			emittance = hMaterial->GetEmittance();
			scattering = hMaterial->GetScattering();

			// texture
			auto& hTexture = hMaterial->GetTexture();
			if (hTexture)
			{
				if (hTexture.GetResource()->GetId() < hCudaWorld.textures.GetCount())
				{
					texture = hCudaWorld.textures.GetStorageAddress() +
						hTexture.GetResource()->GetId();
				}
				else texture = nullptr;
			}
			else texture = nullptr;

			// emittance map
			auto& hEmittanceMap = hMaterial->GetEmittanceMap();
			if (hEmittanceMap)
			{
				if (hEmittanceMap.GetResource()->GetId() < hCudaWorld.emittance_maps.GetCount())
				{
					emittance_map = hCudaWorld.emittance_maps.GetStorageAddress() +
						hEmittanceMap.GetResource()->GetId();
				}
				else emittance_map = nullptr;
			}
			else emittance_map = nullptr;

			hMaterial->GetStateRegister().MakeUnmodified();
		}
		// ~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~
	}
}