#include "cuda_plane.cuh"
#include "cuda_world.cuh"

namespace RayZath
{
	namespace CudaEngine
	{
		__host__ CudaPlane::CudaPlane()
			: material(nullptr)
		{}

		__host__ void CudaPlane::Reconstruct(
			const CudaWorld& hCudaWorld,
			const Handle<Plane>& hPlane,
			hipStream_t& mirror_stream)
		{
			if (!hPlane->GetStateRegister().IsModified()) return;

			position = hPlane->GetPosition();
			rotation = hPlane->GetRotation();
			center = hPlane->GetCenter();
			scale = hPlane->GetScale();

			// material
			auto& hMaterial = hPlane->GetMaterial();
			if (hMaterial)
			{
				if (hMaterial.GetResource()->GetId() < hCudaWorld.materials.GetCount())
				{
					this->material =
						hCudaWorld.materials.GetStorageAddress() +
						hMaterial.GetResource()->GetId();
				}
				else material = hCudaWorld.default_material;
			}
			else material = hCudaWorld.default_material;

			hPlane->GetStateRegister().MakeUnmodified();
		}
	}
}