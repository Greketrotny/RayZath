#include "cuda_sphere.cuh"
#include "cuda_world.cuh"

namespace RayZath
{
	namespace CudaEngine
	{
		__host__ CudaSphere::CudaSphere()
			: radius(1.0f)
			, material(nullptr)
		{}

		__host__ void CudaSphere::Reconstruct(
			const CudaWorld& hCudaWorld, 
			const Handle<Sphere>& hSphere, 
			hipStream_t& mirror_stream)
		{
			if (!hSphere->GetStateRegister().IsModified()) return;

			radius = hSphere->GetRadius();
			transformation = hSphere->GetTransformation();
			bounding_box = hSphere->GetBoundingBox();

			// material
			auto& hMaterial = hSphere->GetMaterial();
			if (hMaterial)
			{
				if (hMaterial.GetAccessor()->GetIdx() < hCudaWorld.materials.GetCount())
				{
					this->material =
						hCudaWorld.materials.GetStorageAddress() +
						hMaterial.GetAccessor()->GetIdx();
				}
				else material = hCudaWorld.default_material;
			}
			else material = hCudaWorld.default_material;


			hSphere->GetStateRegister().MakeUnmodified();
		}
	}
}