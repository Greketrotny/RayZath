#include "cuda_sphere.cuh"
#include "cuda_world.cuh"

namespace RayZath
{
	namespace CudaEngine
	{
		__host__ CudaSphere::CudaSphere()
			: radius(1.0f)
			, material(nullptr)
		{}

		__host__ void CudaSphere::Reconstruct(
			const CudaWorld& hCudaWorld, 
			const Handle<Sphere>& hSphere, 
			hipStream_t& mirror_stream)
		{
			if (!hSphere->GetStateRegister().IsModified()) return;

			// transposition
			transformation.position = hSphere->GetPosition();
			transformation.rotation = hSphere->GetRotation();
			transformation.center = hSphere->GetCenter();
			transformation.scale = hSphere->GetScale();
			this->transformation.g2l.ApplyRotationB(-hSphere->GetRotation());
			this->transformation.l2g.ApplyRotation(hSphere->GetRotation());
			this->radius = hSphere->GetRadius();

			// bounding box
			this->bounding_box = hSphere->GetBoundingBox();

			// material
			auto& hMaterial = hSphere->GetMaterial();
			if (hMaterial)
			{
				if (hMaterial.GetResource()->GetId() < hCudaWorld.materials.GetCount())
				{
					this->material =
						hCudaWorld.materials.GetStorageAddress() +
						hMaterial.GetResource()->GetId();
				}
				else material = hCudaWorld.default_material;
			}
			else material = hCudaWorld.default_material;


			hSphere->GetStateRegister().MakeUnmodified();
		}
	}
}