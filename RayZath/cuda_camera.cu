#include "hip/hip_runtime.h"
#include "cuda_camera.cuh"

namespace RayZath::Cuda
{
	__host__ TracingStates::TracingStates(const vec2ui32 resolution)
	{
		Resize(resolution);
	}
	__host__ void TracingStates::Resize(const vec2ui32 resolution)
	{
		m_path_depth.Reset(resolution);
		m_ray_origin.Reset(resolution);
		m_ray_direction.Reset(resolution);
		m_ray_material.Reset(resolution);
		m_ray_color.Reset(resolution);
	}

	__host__ FrameBuffers::FrameBuffers(const vec2ui32 resolution)
	{
		Resize(resolution);
	}
	__host__ void FrameBuffers::Resize(const vec2ui32 resolution)
	{
		for (size_t i = 0u; i < 2u; ++i)
		{
			m_image_buffer[i].Reset(resolution);
			m_depth_buffer[i].Reset(resolution);
		}
		m_space_buffer.Reset(resolution);
		m_final_image_buffer.Reset(resolution);
		m_final_depth_buffer.Reset(resolution);
	}


	HostPinnedMemory Camera::hostPinnedMemory(0x10000u);

	__host__ Camera::Camera(const vec2ui32 resolution)
		: resolution(resolution)
		, m_frame_buffers(resolution)
		, m_tracing_states(resolution)
		, m_mesh_idx{std::numeric_limits<uint32_t>::max()}
		, m_mesh_material_idx{std::numeric_limits<uint32_t>::max()}
	{}

	__host__ void Camera::Reconstruct(
		[[maybe_unused]] const World& hCudaWorld,
		const RayZath::Engine::Handle<RayZath::Engine::Camera>& hCamera,
		[[maybe_unused]] hipStream_t& mirror_stream)
	{
		if (!hCamera->GetStateRegister().IsModified()) return;

		SwapHistoryIdx();

		CurrentPosition() = hCamera->GetPosition();
		CurrentCoordSystem() = hCamera->GetCoordSystem();

		aspect_ratio = hCamera->GetAspectRatio();
		CurrentFov() = hCamera->GetFov().value();
		near_far = hCamera->GetNearFar();
		focal_distance = hCamera->GetFocalDistance();
		aperture = hCamera->GetAperture();
		exposure_time = hCamera->GetExposureTime();
		temporal_blend = hCamera->GetTemporalBlend();
		m_ray_cast_pixel = hCamera->GetRayCastPixel();

		if (resolution != hCamera->GetResolution())
		{// resize buffers to match size of hostCamera resolution

			resolution = hCamera->GetResolution();
			m_frame_buffers.Resize(resolution);
			m_tracing_states.Resize(resolution);

			// resize hostPinnedMemory for mirroring
			this->hostPinnedMemory.SetMemorySize(
				std::min(
					resolution.x * resolution.y * uint32_t(sizeof(Color<unsigned char>)),
					0x100000u)); // max 1MiB
		}

		hCamera->GetStateRegister().MakeUnmodified();
	}
}