#include "hip/hip_runtime.h"
#include "cuda_camera.cuh"

namespace RayZath
{
	namespace CudaEngine
	{
		HostPinnedMemory CudaCamera::hostPinnedMemory(0x10000u);

		__host__ CudaCamera::CudaCamera()
			: width(0), height(0)
			, aspect_ratio(1.0f)
			, enabled(true)
			, fov(2.0f)
			, focal_distance(10.0f)
			, aperture(0.01f)
			, exposure_time(1.0f / 60.0f)
			, passes_count(0u)
			, inv_passes_count(1.0f)
			, mp_sample_image_array(nullptr)
			, m_so_sample(0u)
			, mp_final_image_array{ 0u, 0u }
			, m_so_final{ 0u, 0u }
			, mp_tracing_paths(nullptr)
		{}
		__host__ CudaCamera::~CudaCamera()
		{
			// destroy sample image surface
			if (m_so_sample) CudaErrorCheck(hipDestroySurfaceObject(m_so_sample));
			m_so_sample = 0u;
			if (mp_sample_image_array) CudaErrorCheck(hipFreeArray(mp_sample_image_array));
			this->mp_sample_image_array = nullptr;

			// destroy final image surfaces
			for (uint32_t i = 0u; i < 2u; i++)
			{
				if (m_so_final[i]) CudaErrorCheck(hipDestroySurfaceObject(m_so_final[i]));
				m_so_final[i] = 0;
				if (mp_final_image_array[i]) CudaErrorCheck(hipFreeArray(mp_final_image_array[i]));
				this->mp_final_image_array[i] = nullptr;
			}

			// destroy tracing paths
			if (mp_tracing_paths) CudaErrorCheck(hipFree(mp_tracing_paths));
			mp_tracing_paths = nullptr;
		}

		__host__ void CudaCamera::Reconstruct(
			const CudaWorld& hCudaWorld, 
			const Handle<Camera>& hCamera, 
			hipStream_t& mirror_stream)
		{
			if (!hCamera->GetStateRegister().IsModified()) return;

			position = hCamera->GetPosition();
			rotation = hCamera->GetRotation();

			coord_system = hCamera->GetCoordSystem();

			aspect_ratio = hCamera->GetAspectRatio();
			fov = hCamera->GetFov().value();
			focal_distance = hCamera->GetFocalDistance();
			aperture = hCamera->GetAperture();
			exposure_time = hCamera->GetExposureTime();
			enabled = hCamera->Enabled();

			if (width != hCamera->GetWidth() || height != hCamera->GetHeight())
			{// resize pixel map to match size with hostCamera resolution

				// [>] Release CudaCamera resources
				// destroy surface objects
				if (m_so_sample) CudaErrorCheck(hipDestroySurfaceObject(m_so_sample));
				if (m_so_final[0]) CudaErrorCheck(hipDestroySurfaceObject(m_so_final[0]));
				if (m_so_final[1]) CudaErrorCheck(hipDestroySurfaceObject(m_so_final[1]));

				// free sampling image, final image and tracing paths memory
				if (mp_sample_image_array)		CudaErrorCheck(hipFreeArray(mp_sample_image_array));
				if (mp_final_image_array[0])	CudaErrorCheck(hipFreeArray(mp_final_image_array[0]));
				if (mp_final_image_array[1])	CudaErrorCheck(hipFreeArray(mp_final_image_array[1]));
				if (mp_tracing_paths)			CudaErrorCheck(hipFree(mp_tracing_paths));


				// [>] Update CudaCamera resolution
				width = hCamera->GetWidth();
				height = hCamera->GetHeight();


				// [>] Reallocate resources
				// create sample image surface
				hipChannelFormatDesc cd_sample = hipCreateChannelDesc<float4>();
				CudaErrorCheck(hipMallocArray(
					&mp_sample_image_array, 
					&cd_sample, 
					width, height,
					hipArraySurfaceLoadStore));

				hipResourceDesc rd_sample;
				memset(&rd_sample, 0, sizeof(rd_sample));
				rd_sample.resType = hipResourceTypeArray;

				rd_sample.res.array.array = mp_sample_image_array;
				m_so_sample = 0;
				CudaErrorCheck(hipCreateSurfaceObject(&m_so_sample, &rd_sample));

				// create final image surfaces
				hipChannelFormatDesc cd_final = hipCreateChannelDesc<uchar4>();
				CudaErrorCheck(hipMallocArray(
					&mp_final_image_array[0], 
					&cd_final, 
					width, height, 
					hipArraySurfaceLoadStore));
				CudaErrorCheck(hipMallocArray(
					&mp_final_image_array[1], 
					&cd_final, 
					width, height, 
					hipArraySurfaceLoadStore));

				hipResourceDesc rd_final;
				memset(&rd_final, 0, sizeof(rd_final));
				rd_final.resType = hipResourceTypeArray;

				rd_final.res.array.array = mp_final_image_array[0];
				m_so_final[0] = 0;
				CudaErrorCheck(hipCreateSurfaceObject(&m_so_final[0], &rd_final));
				rd_final.res.array.array = mp_final_image_array[1];
				m_so_final[1] = 0;
				CudaErrorCheck(hipCreateSurfaceObject(&m_so_final[1], &rd_final));

				// allocate memory for tracing paths
				CudaErrorCheck(hipMalloc(
					(void**)&mp_tracing_paths, 
					size_t(width) * size_t(height) * size_t(sizeof(*mp_tracing_paths))));


				// [>] Resize hostPinnedMemory for mirroring
				this->hostPinnedMemory.SetMemorySize(
					std::min(
						width * height * uint32_t(sizeof(Color<unsigned char>)),
						0x100000u)); // max 1MiB
				passes_count = 0u;
			}

			hCamera->GetStateRegister().MakeUnmodified();
		}
		// ~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~
	}
}