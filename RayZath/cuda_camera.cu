#include "hip/hip_runtime.h"
#include "cuda_camera.cuh"

namespace RayZath
{
	HostPinnedMemory CudaCamera::hostPinnedMemory(0xFFFF);

	__host__ CudaCamera::CudaCamera()
		: width(0), height(0)
		, max_width(0), max_height(0)
		, aspect_ratio(1.0f)
		, enabled(true)
		, fov(2.0f)
		, focal_distance(10.0f)
		, aperture(0.01f)
		, sampling_image(nullptr)
		, samples_count(0)
		//, tracingPaths(nullptr)
	{
		final_image[0] = nullptr;
		final_image[1] = nullptr;
	}
	__host__ CudaCamera::~CudaCamera()
	{
		if (sampling_image) CudaErrorCheck(hipFree(sampling_image));
		this->sampling_image = nullptr;

		if (final_image[0]) CudaErrorCheck(hipFree(final_image[0]));
		final_image[0] = nullptr;
		if (final_image[1]) CudaErrorCheck(hipFree(final_image[1]));
		final_image[1] = nullptr;

		//if (this->tracingPaths) CudaErrorCheck(hipFree(this->tracingPaths));
		//this->tracingPaths = nullptr;

		max_width = 0u;
		max_height = 0u;
	}

	__host__ void CudaCamera::Reconstruct(const Camera& hCamera, hipStream_t* mirror_stream)
	{
		position = hCamera.GetPosition();
		rotation = hCamera.GetRotation();

		width = hCamera.GetWidth();
		height = hCamera.GetHeight();

		if (max_width * max_height != hCamera.GetMaxWidth() * hCamera.GetMaxHeight())
		{// resize pixel map to match size with hostCamera resolution

			// free sampling image and final image memory
			if (sampling_image)		CudaErrorCheck(hipFree(sampling_image));
			if (final_image[0])		CudaErrorCheck(hipFree(final_image[0]));
			if (final_image[1])		CudaErrorCheck(hipFree(final_image[1]));
			//if (this->tracingPaths)		CudaErrorCheck(hipFree(this->tracingPaths));

			// update max width and max height
			max_width = hCamera.GetMaxWidth();
			max_height = hCamera.GetMaxHeight();

			// allocate device memory for sampingImage and final_image
			CudaErrorCheck(hipMalloc((void**)&sampling_image, max_width * max_height * sizeof(*sampling_image)));
			CudaErrorCheck(hipMalloc((void**)&final_image[0], max_width * max_height * sizeof(*(final_image[0]))));
			CudaErrorCheck(hipMalloc((void**)&final_image[1], max_width * max_height * sizeof(*(final_image[1]))));
			//CudaErrorCheck(hipMalloc((void**)&this->tracingPaths, max_width * max_height * sizeof(*tracingPaths)));

			// resize hostPinnedMemory for mirroring
			this->hostPinnedMemory.SetMemorySize(std::min(max_width * max_height * sizeof(*sampling_image), uint64_t(0xFFFFFFllu)));
			samples_count = 0;
		}
	}
	/*__host__ CudaColor<unsigned char>* CudaCamera::FinalImageGetAddress(unsigned int bufferIndex)
	{
		return final_image[bufferIndex];
	}*/
	// ~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~
}