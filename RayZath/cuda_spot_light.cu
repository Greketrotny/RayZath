#include "hip/hip_runtime.h"
#include "cuda_spot_light.cuh"

namespace RayZath
{
	namespace CudaEngine
	{
		__host__ CudaSpotLight::CudaSpotLight()
			: size(1.0f)
			, angle(1.0f)
			, cos_angle(0.5f)
			, sharpness(1.0f)
		{}
		__host__ CudaSpotLight::~CudaSpotLight()
		{}


		__host__ void CudaSpotLight::Reconstruct(SpotLight& hSpotLight, hipStream_t& mirror_stream)
		{
			if (!hSpotLight.GetStateRegister().IsModified()) return;

			position = hSpotLight.GetPosition();
			direction = hSpotLight.GetDirection();
			size = hSpotLight.GetSize();
			angle = hSpotLight.GetBeamAngle();
			sharpness = hSpotLight.GetSharpness();

			material.color = hSpotLight.GetColor();
			material.emittance = hSpotLight.GetEmission();

			cos_angle = cos(angle);

			hSpotLight.GetStateRegister().MakeUnmodified();
		}
	}
}