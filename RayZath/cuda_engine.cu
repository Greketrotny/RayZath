#include "cuda_engine.cuh"

namespace RayZath
{
	CudaEngine::CudaEngine()
		: mp_dCudaWorld(nullptr)
		, m_hpm_CudaWorld(sizeof(CudaWorld))
	{
		hipSetDevice(0);

		// create streams
		CudaErrorCheck(hipStreamCreate(&m_mirror_stream));

		// create empty dCudaWorld
		CudaWorld* hCudaWorld = (CudaWorld*)m_hpm_CudaWorld.GetPointerToMemory();
		new (hCudaWorld) CudaWorld();
		CudaErrorCheck(hipMalloc(&mp_dCudaWorld, sizeof(CudaWorld)));
		CudaErrorCheck(hipMemcpy(
			mp_dCudaWorld, hCudaWorld,
			sizeof(CudaWorld),
			hipMemcpyKind::hipMemcpyHostToDevice));
	}
	CudaEngine::~CudaEngine()
	{
		// destroy mp_dCudaWorld
		if (mp_dCudaWorld)
		{
			CudaWorld* hCudaWorld = (CudaWorld*)m_hpm_CudaWorld.GetPointerToMemory();
			CudaErrorCheck(hipMemcpy(
				hCudaWorld, mp_dCudaWorld,
				sizeof(CudaWorld),
				hipMemcpyKind::hipMemcpyDeviceToHost));

			hCudaWorld->~CudaWorld();

			CudaErrorCheck(hipFree(mp_dCudaWorld));
			mp_dCudaWorld = nullptr;
		}

		// destroy streams
		CudaErrorCheck(hipStreamDestroy(m_mirror_stream));
	}


	void CudaEngine::RenderWorld(World& hWorld)
	{
		ReconstructCudaWorld(mp_dCudaWorld, hWorld, &m_mirror_stream);

		CudaKernel::Kernel<<<500u, 256u>>>(mp_dCudaWorld);

		CudaErrorCheck(hipDeviceSynchronize());
		CudaErrorCheck(hipGetLastError());

		TransferResultsToHost(mp_dCudaWorld, hWorld, &m_mirror_stream);
		/*CudaWorld* hCudaWorld = (CudaWorld*)malloc(sizeof(CudaWorld));
		CudaErrorCheck(hipMemcpy(
			hCudaWorld, mp_dCudaWorld,
			sizeof(CudaWorld),
			hipMemcpyKind::hipMemcpyDeviceToHost));

		CudaCamera* hCudaCamera = (CudaCamera*)malloc(sizeof(CudaCamera));
		CudaErrorCheck(hipMemcpy(
			hCudaCamera, &hCudaWorld->cameras[0],
			sizeof(CudaCamera),
			hipMemcpyKind::hipMemcpyDeviceToHost));

		float x = hCudaCamera->position.x;


		free(hCudaCamera);
		free(hCudaWorld);*/
		//CudaKernel::CallKernel();
	}
	void CudaEngine::ReconstructCudaWorld(
		CudaWorld* dCudaWorld,
		World& hWorld,
		hipStream_t* mirror_stream)
	{
		// copy CudaWorld to host
		CudaWorld* hCudaWorld = (CudaWorld*)m_hpm_CudaWorld.GetPointerToMemory();
		CudaErrorCheck(hipMemcpyAsync(
			hCudaWorld, dCudaWorld,
			sizeof(CudaWorld),
			hipMemcpyKind::hipMemcpyDeviceToHost, *mirror_stream));
		CudaErrorCheck(hipStreamSynchronize(*mirror_stream));

		// reconstruct CudaWorld on host
		hCudaWorld->Reconstruct(hWorld, mirror_stream);

		// copy CudaWorld back to device
		CudaErrorCheck(hipMemcpyAsync(
			dCudaWorld, hCudaWorld,
			sizeof(CudaWorld),
			hipMemcpyKind::hipMemcpyHostToDevice, *mirror_stream));
		CudaErrorCheck(hipStreamSynchronize(*mirror_stream));

		hWorld.Updated();
	}
	void CudaEngine::TransferResultsToHost(
		CudaWorld* dCudaWorld,
		World& hWorld,
		hipStream_t* mirror_stream)
	{
		for (size_t i = 0; i < hWorld.GetCameras().GetCapacity(); ++i)
		{
			// check if hostCamera does exict
			Camera* hCamera = hWorld.GetCameras()[i];
			if (hCamera == nullptr) continue;	// no camera at this address
			if (!hCamera->Enabled()) continue;	// camera is disabled


			// [>] Get CudaWorld from device
			CudaWorld* hCudaWorld = (CudaWorld*)m_hpm_CudaWorld.GetPointerToMemory();
			CudaErrorCheck(hipMemcpyAsync(
				hCudaWorld, dCudaWorld, 
				sizeof(CudaWorld), 
				hipMemcpyKind::hipMemcpyDeviceToHost, *mirror_stream));
			CudaErrorCheck(hipStreamSynchronize(*mirror_stream));

			if (hCudaWorld->cameras.GetCount() == 0) return;	// hCudaWorld has no cameras

			// [>] Get CudaCamera class from hostCudaWorld
			CudaCamera* hCudaCamera = (CudaCamera*)CudaWorld::m_hpm.GetPointerToMemory();
			if (CudaWorld::m_hpm.GetSize() < sizeof(*hCudaCamera))
				throw Exception(__FILE__, __LINE__, L"insufficient host pinned memory for CudaCamera");
			CudaErrorCheck(hipMemcpyAsync(
				hCudaCamera, &hCudaWorld->cameras[i], 
				sizeof(CudaCamera), 
				hipMemcpyKind::hipMemcpyDeviceToHost, *mirror_stream));
			CudaErrorCheck(hipStreamSynchronize(*mirror_stream));

			if (!hCudaCamera->Exist()) continue;

			hCamera->m_samples_count = hCudaCamera->samples_count;

			// [>] Asynchronous copying
			static_assert(
				sizeof(*hCamera->GetBitmap().GetMapAddress()) == 
/* change index */				sizeof(*hCudaCamera->final_image[0]), 
				"sizeof(Graphics::Color) != sizeof(CudaColor<unsigned char>)");

			// check cameras resolution
			if (hCamera->GetWidth() != hCudaCamera->width || 
				hCamera->GetHeight() != hCudaCamera->height) continue;
			if (hCamera->GetMaxWidth() != hCudaCamera->max_width || 
				hCamera->GetMaxHeight() != hCudaCamera->max_height) continue;

/* change index */			size_t chunkSize = hCudaCamera->hostPinnedMemory.GetSize() / (sizeof(*hCudaCamera->final_image[0]));
			if (chunkSize == 16u)
				throw Exception(__FILE__, __LINE__, 
					L"Not enough host pinned memory for async image copy");

			size_t nPixels = hCamera->GetWidth() * hCamera->GetHeight();
			for (size_t startIndex = 0; startIndex < nPixels; startIndex += chunkSize)
			{
				if (startIndex + chunkSize > nPixels) chunkSize = nPixels - startIndex;

				// copy final image data from hostCudaCamera to hostCudaPixels on pinned memory
				CudaColor<unsigned char>* hCudaPixels = (CudaColor<unsigned char>*)CudaCamera::hostPinnedMemory.GetPointerToMemory();
/* change index */				CudaErrorCheck(hipMemcpyAsync(hCudaPixels, hCudaCamera->final_image[0] + startIndex, 
					chunkSize * sizeof(*hCudaPixels), 
					hipMemcpyKind::hipMemcpyDeviceToHost, *mirror_stream));
				CudaErrorCheck(hipStreamSynchronize(*mirror_stream));

				// copy final image data from hostCudaPixels on pinned memory to hostCamera
				memcpy(hCamera->GetBitmap().GetMapAddress() + startIndex, hCudaPixels, 
					chunkSize * sizeof(*hCudaPixels));
			}
		}
	}
}