#include "cuda_engine.cuh"
#include "point.h"

namespace RayZath
{
	namespace CudaEngine
	{
		Engine::Engine()
			: mp_dCudaWorld(nullptr)
			, m_hpm_CudaWorld(sizeof(CudaWorld))
			, m_hpm_CudaKernelData(sizeof(CudaKernelData))
			, m_update_flag(true)
		{
			hipSetDevice(0);

			// create streams
			CudaErrorCheck(hipStreamCreate(&m_mirror_stream));
			CudaErrorCheck(hipStreamCreate(&m_render_stream));

			// create empty CudaKernelData
			CudaKernelData* hCudaKernelData =
				(CudaKernelData*)m_hpm_CudaKernelData.GetPointerToMemory();
			for (uint32_t i = 0; i < sizeof(mp_kernel_data) / sizeof(*mp_kernel_data); ++i)
			{
				//new (hCudaKernelData) CudaKernelData();
				hCudaKernelData->renderIndex = 0u;

				CudaErrorCheck(hipMalloc(
					(void**)&mp_kernel_data[i], sizeof(CudaKernelData)));
				CudaErrorCheck(hipMemcpy(mp_kernel_data[i], hCudaKernelData,
					sizeof(CudaKernelData), hipMemcpyKind::hipMemcpyHostToDevice));
			}


			// create empty dCudaWorld
			CudaWorld* hCudaWorld = (CudaWorld*)m_hpm_CudaWorld.GetPointerToMemory();
			new (hCudaWorld) CudaWorld();
			CudaErrorCheck(hipMalloc(&mp_dCudaWorld, sizeof(CudaWorld)));
			CudaErrorCheck(hipMemcpy(
				mp_dCudaWorld, hCudaWorld,
				sizeof(CudaWorld),
				hipMemcpyKind::hipMemcpyHostToDevice));

			// create and launch kernel launching thread
			mp_launch_thread = new std::thread(&Engine::LaunchFunction, this);
			mp_launch_thread->detach();
		}
		Engine::~Engine()
		{
			// terminate and delete kernel launching thread
			m_launch_thread_terminate = true;
			m_kernel_gate.Open();
			m_host_gate.WaitForOpen();
			delete mp_launch_thread;

			// destroy mp_dCudaWorld
			if (mp_dCudaWorld)
			{
				CudaWorld* hCudaWorld = (CudaWorld*)m_hpm_CudaWorld.GetPointerToMemory();
				CudaErrorCheck(hipMemcpy(
					hCudaWorld, mp_dCudaWorld,
					sizeof(CudaWorld),
					hipMemcpyKind::hipMemcpyDeviceToHost));

				hCudaWorld->~CudaWorld();

				CudaErrorCheck(hipFree(mp_dCudaWorld));
				mp_dCudaWorld = nullptr;
			}

			// destroy dCudaKernelData
			CudaKernelData* hCudaKernelData =
				(CudaKernelData*)m_hpm_CudaKernelData.GetPointerToMemory();
			for (uint32_t i = 0; i < sizeof(mp_kernel_data) / sizeof(*mp_kernel_data); ++i)
			{
				CudaErrorCheck(hipMemcpy(
					hCudaKernelData, mp_kernel_data[i],
					sizeof(CudaKernelData),
					hipMemcpyKind::hipMemcpyDeviceToHost));

				hCudaKernelData->~CudaKernelData();

				CudaErrorCheck(hipFree(mp_kernel_data[i]));
				mp_kernel_data[i] = nullptr;
			}

			// destroy streams
			CudaErrorCheck(hipStreamDestroy(m_mirror_stream));
			CudaErrorCheck(hipStreamDestroy(m_render_stream));
		}

		void Engine::RenderWorld(World& hWorld)
		{
			mainDebugInfo.Clear();

			m_update_flag = hWorld.GetStateRegister().IsModified();
			Timer function_timer, step_timer;
			std::wstring timing_string = L"Host side:\n";


			// [>] Create Launch configurations
			step_timer.Start();
			CreateLaunchConfigurations(hWorld);
			AppendTimeToString(timing_string, L"create launch configs: ", step_timer.GetTime());


			// [>] Reconstruct CudaKernelData
			step_timer.Start();
			ReconstructKernelData(m_mirror_stream);
			AppendTimeToString(timing_string, L"reconstruct kernel data: ", step_timer.GetTime());


			// [>] Synchronize with kernel function
			step_timer.Start();
			m_host_gate.WaitForOpen();	// wait for kernel to finish render
			m_host_gate.Close();		// close gate for itself
			AppendTimeToString(timing_string, L"wait for kernel: ", step_timer.GetTime());
			mainDebugInfo.AddDebugString(renderTimingString);


			// [>] Reconstruct dCudaWorld
			step_timer.Start();
			hWorld.Update();
			ReconstructCudaWorld(mp_dCudaWorld, hWorld, m_mirror_stream);
			AppendTimeToString(timing_string, L"reconstruct CudaWorld: ", step_timer.GetTime());

			CudaKernel::CopyToConstantMemory(mp_kernel_data[m_update_ix], m_mirror_stream);

			// [>] Swap indexes
			std::swap(m_update_ix, m_render_ix);


			// [>] Launch kernel
			m_kernel_gate.Open();	// open gate for kernel
			//m_host_gate.WaitForOpen(); // <- uncoment for sync rendering


			// [>] Transfer results to host
			step_timer.Start();
			TransferResultsToHost(mp_dCudaWorld, hWorld, m_mirror_stream);
			AppendTimeToString(timing_string, L"copy final render to host: ", step_timer.GetTime());


			// [>] Sum up timings and add debug string
			AppendTimeToString(timing_string, L"render function full time: ", function_timer.GetTime());
			mainDebugInfo.AddDebugString(timing_string);

			std::wstring camera_str = L"camera: ";
			camera_str += std::to_wstring(hWorld.GetCameras()[0]->GetSamplesCount()) + L"spp\n";
			mainDebugInfo.AddDebugString(camera_str);
		}
		void Engine::CreateLaunchConfigurations(const World& world)
		{
			m_launch_configs[m_update_ix].clear();
			for (uint32_t i = 0; i < world.GetCameras().GetCapacity(); ++i)
			{
				const Camera* camera = world.GetCameras()[i];
				if (camera == nullptr) continue;	// no camera at the index
				if (!camera->Enabled()) continue;	// camera is disabled

				m_launch_configs[m_update_ix].push_back(
					LaunchConfiguration(
						m_hardware, *camera, m_update_flag));
			}
		}
		void Engine::ReconstructKernelData(hipStream_t& mirror_stream)
		{
			CudaKernelData* hCudaKernelData =
				(CudaKernelData*)m_hpm_CudaKernelData.GetPointerToMemory();

			// copy dCudaKernelData to host
			CudaErrorCheck(hipMemcpyAsync(
				hCudaKernelData,
				mp_kernel_data[m_update_ix],
				sizeof(CudaKernelData),
				hipMemcpyKind::hipMemcpyDeviceToHost,
				mirror_stream));
			CudaErrorCheck(hipStreamSynchronize(mirror_stream));

			// reconstruct hCudaKernelData
			hCudaKernelData->Reconstruct(
				m_update_ix,
				mirror_stream);

			// copy hCudaKernelData to device
			CudaErrorCheck(hipMemcpyAsync(
				mp_kernel_data[m_update_ix],
				hCudaKernelData,
				sizeof(CudaKernelData),
				hipMemcpyKind::hipMemcpyHostToDevice,
				mirror_stream));
			CudaErrorCheck(hipStreamSynchronize(mirror_stream));
		}
		void Engine::ReconstructCudaWorld(
			CudaWorld* dCudaWorld,
			World& hWorld,
			hipStream_t& mirror_stream)
		{
			if (!hWorld.GetStateRegister().IsModified()) return;

			// copy CudaWorld to host
			CudaWorld* hCudaWorld = (CudaWorld*)m_hpm_CudaWorld.GetPointerToMemory();
			CudaErrorCheck(hipMemcpyAsync(
				hCudaWorld, dCudaWorld,
				sizeof(CudaWorld),
				hipMemcpyKind::hipMemcpyDeviceToHost, mirror_stream));
			CudaErrorCheck(hipStreamSynchronize(mirror_stream));

			// reconstruct CudaWorld on host
			hCudaWorld->Reconstruct(hWorld, mirror_stream);

			// copy CudaWorld back to device
			CudaErrorCheck(hipMemcpyAsync(
				dCudaWorld, hCudaWorld,
				sizeof(CudaWorld),
				hipMemcpyKind::hipMemcpyHostToDevice, mirror_stream));
			CudaErrorCheck(hipStreamSynchronize(mirror_stream));

			hWorld.GetStateRegister().MakeUnmodified();
		}
		void Engine::TransferResultsToHost(
			CudaWorld* dCudaWorld,
			World& hWorld,
			hipStream_t& mirror_stream)
		{
			for (uint32_t i = 0; i < hWorld.GetCameras().GetCapacity(); ++i)
			{
				// check if hostCamera does exict
				Camera* hCamera = hWorld.GetCameras()[i];
				if (hCamera == nullptr) continue;	// no camera at this address
				if (!hCamera->Enabled()) continue;	// camera is disabled


				// [>] Get CudaWorld from device
				CudaWorld* hCudaWorld = (CudaWorld*)m_hpm_CudaWorld.GetPointerToMemory();
				CudaErrorCheck(hipMemcpyAsync(
					hCudaWorld, dCudaWorld,
					sizeof(CudaWorld),
					hipMemcpyKind::hipMemcpyDeviceToHost, mirror_stream));
				CudaErrorCheck(hipStreamSynchronize(mirror_stream));

				if (hCudaWorld->cameras.GetCount() == 0) return;	// hCudaWorld has no cameras


				// [>] Get CudaCamera class from hCudaWorld
				CudaCamera* hCudaCamera = nullptr;
				if (CudaWorld::m_hpm.GetSize() < sizeof(*hCudaCamera))
					ThrowException(L"insufficient host pinned memory for CudaCamera");
				hCudaCamera = (CudaCamera*)CudaWorld::m_hpm.GetPointerToMemory();

				CudaErrorCheck(hipMemcpyAsync(
					hCudaCamera, &hCudaWorld->cameras[i],
					sizeof(CudaCamera),
					hipMemcpyKind::hipMemcpyDeviceToHost, mirror_stream));
				CudaErrorCheck(hipStreamSynchronize(mirror_stream));

				if (!hCudaCamera->Exist()) continue;


				// [>] Asynchronous copying
				hCamera->m_samples_count = hCudaCamera->samples_count;

				static_assert(
					sizeof(*hCamera->GetBitmap().GetMapAddress()) ==
					sizeof(CudaColor<unsigned char>),
					"sizeof(Graphics::Color) != sizeof(CudaColor<unsigned char>)");

				// check cameras resolution
				if (hCamera->GetWidth() != hCudaCamera->width ||
					hCamera->GetHeight() != hCudaCamera->height) continue;

				uint32_t chunkSize =
					hCudaCamera->hostPinnedMemory.GetSize() /
					(sizeof(CudaColor<unsigned char>));
				if (chunkSize < 16u) ThrowException(L"Not enough host pinned memory for async image copy");

				uint32_t nPixels = hCamera->GetWidth() * hCamera->GetHeight();
				for (uint32_t startIndex = 0; startIndex < nPixels; startIndex += chunkSize)
				{
					// find start index
					if (startIndex + chunkSize > nPixels) chunkSize = nPixels - startIndex;

					// find offset point
					Graphics::Point<uint32_t> offset_point(startIndex % hCamera->GetWidth(), startIndex / hCamera->GetWidth());

					// copy final image data from hCudaCamera to hCudaPixels on pinned memory
					CudaColor<unsigned char>* hCudaPixels =
						(CudaColor<unsigned char>*)CudaCamera::hostPinnedMemory.GetPointerToMemory();
					CudaErrorCheck(cudaMemcpyFromArrayAsync(
						hCudaPixels, hCudaCamera->mp_final_image_array[m_update_ix],
						offset_point.x * sizeof(*hCudaPixels), offset_point.y,
						chunkSize * sizeof(*hCudaPixels),
						hipMemcpyKind::hipMemcpyDeviceToHost, mirror_stream));
					CudaErrorCheck(hipStreamSynchronize(mirror_stream));

					// copy final image data from hostCudaPixels on pinned memory to hostCamera
					memcpy(hCamera->GetBitmap().GetMapAddress() + startIndex, hCudaPixels,
						chunkSize * sizeof(*hCudaPixels));
				}
			}
		}

		void Engine::LaunchFunction()
		{
			Timer function_timer, step_timer;

			while (true)
			{
				function_timer.Start();
				step_timer.Start();

				m_host_gate.Open();			// allow host to get things ready to render
				m_kernel_gate.WaitForOpen();// wait for host to prepare resources
				m_kernel_gate.Close();		// close gate for itself

				renderTimingString = L"Device side: \n";
				AppendTimeToString(renderTimingString, L"wait for host: ", step_timer.GetTime());

				if (m_launch_thread_terminate)
				{
					m_host_gate.Open();
					return;	// terminate launch function
				}


				// [>] Launch kernel for each camera
				for (uint32_t i = 0; i < m_launch_configs[m_render_ix].size(); ++i)
				{
					LaunchConfiguration& config = m_launch_configs[m_render_ix][i];
					hipSetDevice(config.GetDeviceId());

					// [>] Update CudaCamera samples
					// reset samples values if needed
					step_timer.Start();
					if (config.GetUpdateFlag())
					{
						CudaKernel::CudaCameraSampleReset
							<< <
							config.GetGrid(),
							config.GetThreadBlock(),
							0u,
							m_render_stream
							>> >
							(mp_dCudaWorld, config.GetCameraId());
						CudaErrorCheck(hipStreamSynchronize(m_render_stream));
						CudaErrorCheck(hipGetLastError());
					}

					// increment samples number
					CudaKernel::CudaCameraUpdateSamplesNumber
						<< <
						1u, 1u, 0u, m_render_stream
						>> >
						(mp_dCudaWorld, config.GetCameraId(), config.GetUpdateFlag());
					CudaErrorCheck(hipStreamSynchronize(m_render_stream));
					CudaErrorCheck(hipGetLastError());
					AppendTimeToString(renderTimingString, L"update samples: ", step_timer.GetTime());


					// [>] Main render function
					step_timer.Start();
					CudaKernel::GenerateCameraRay
						<< <
						config.GetGrid(),
						config.GetThreadBlock(),
						config.GetSharedMemorySize(),
						m_render_stream
						>> >
						(/*mp_kernel_data[m_render_ix],*/
							mp_dCudaWorld,
							m_launch_configs[m_render_ix][i].GetCameraId());

					CudaErrorCheck(hipStreamSynchronize(m_render_stream));
					CudaErrorCheck(hipGetLastError());
					AppendTimeToString(renderTimingString, L"main render: ", step_timer.GetTime());


					// [>] Tone mapping
					step_timer.Start();
					CudaKernel::ToneMap
						<< <
						config.GetGrid(),
						config.GetThreadBlock(),
						0u,
						m_render_stream
						>> >
						(mp_kernel_data[m_render_ix],
							mp_dCudaWorld, config.GetCameraId());

					CudaErrorCheck(hipStreamSynchronize(m_render_stream));
					CudaErrorCheck(hipGetLastError());
					AppendTimeToString(renderTimingString, L"tone mapping: ", step_timer.GetTime());
					AppendTimeToString(renderTimingString, L"render full time: ", function_timer.GetTime());
				}
			}
		}
	}
}