#include "cuda_engine.cuh"

namespace RayZath
{
	CudaEngine::CudaEngine()
		: mp_dCudaWorld(nullptr)
		, m_hpm_CudaWorld(sizeof(CudaWorld))
	{
		hipSetDevice(0);

		// create streams
		CudaErrorCheck(hipStreamCreate(&m_mirror_stream));

		// create empty dCudaWorld
		CudaWorld* hCudaWorld = (CudaWorld*)m_hpm_CudaWorld.GetPointerToMemory();
		new (hCudaWorld) CudaWorld();
		CudaErrorCheck(hipMalloc(&mp_dCudaWorld, sizeof(CudaWorld)));
		CudaErrorCheck(hipMemcpy(
			mp_dCudaWorld, hCudaWorld,
			sizeof(CudaWorld),
			hipMemcpyKind::hipMemcpyHostToDevice));
	}
	CudaEngine::~CudaEngine()
	{
		// destroy mp_dCudaWorld
		if (mp_dCudaWorld)
		{
			CudaWorld* hCudaWorld = (CudaWorld*)m_hpm_CudaWorld.GetPointerToMemory();
			CudaErrorCheck(hipMemcpy(
				hCudaWorld, mp_dCudaWorld,
				sizeof(CudaWorld),
				hipMemcpyKind::hipMemcpyDeviceToHost));

			hCudaWorld->~CudaWorld();

			CudaErrorCheck(hipFree(mp_dCudaWorld));
			mp_dCudaWorld = nullptr;
		}

		// destroy streams
		CudaErrorCheck(hipStreamDestroy(m_mirror_stream));
	}


	void CudaEngine::RenderWorld(World& hWorld)
	{
		ReconstructCudaWorld(mp_dCudaWorld, hWorld, &m_mirror_stream);

		CudaKernel::Kernel<<<1u, 1u>>>(mp_dCudaWorld);

		CudaErrorCheck(hipDeviceSynchronize());
		CudaErrorCheck(hipGetLastError());

		CudaWorld* hCudaWorld = (CudaWorld*)malloc(sizeof(CudaWorld));
		CudaErrorCheck(hipMemcpy(
			hCudaWorld, mp_dCudaWorld,
			sizeof(CudaWorld),
			hipMemcpyKind::hipMemcpyDeviceToHost));

		CudaCamera* hCudaCamera = (CudaCamera*)malloc(sizeof(CudaCamera));
		CudaErrorCheck(hipMemcpy(
			hCudaCamera, &hCudaWorld->cameras[0],
			sizeof(CudaCamera),
			hipMemcpyKind::hipMemcpyDeviceToHost));

		float x = hCudaCamera->position.x;


		free(hCudaCamera);
		free(hCudaWorld);
		//CudaKernel::CallKernel();
	}
	void CudaEngine::ReconstructCudaWorld(
		CudaWorld* dCudaWorld,
		World& hWorld,
		hipStream_t* mirror_stream)
	{
		// copy CudaWorld to host
		CudaWorld* hCudaWorld = (CudaWorld*)m_hpm_CudaWorld.GetPointerToMemory();
		CudaErrorCheck(hipMemcpyAsync(
			hCudaWorld, dCudaWorld,
			sizeof(CudaWorld),
			hipMemcpyKind::hipMemcpyDeviceToHost, *mirror_stream));
		CudaErrorCheck(hipStreamSynchronize(*mirror_stream));

		// reconstruct CudaWorld on host
		hCudaWorld->Reconstruct(hWorld, mirror_stream);

		// copy CudaWorld back to device
		CudaErrorCheck(hipMemcpyAsync(
			dCudaWorld, hCudaWorld,
			sizeof(CudaWorld),
			hipMemcpyKind::hipMemcpyHostToDevice, *mirror_stream));
		CudaErrorCheck(hipStreamSynchronize(*mirror_stream));
	}
}