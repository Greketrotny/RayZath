#include "hip/hip_runtime.h"
#include "cuda_engine_kernel.cuh"

namespace RayZath
{
	namespace CudaKernel
	{
		__global__ void GenerateCameraRay(
			CudaKernelData* const global_kernel,
			CudaWorld* world,
			const int camera_id)
		{
			// create local thread structure
			ThreadData thread(global_kernel->randomNumbers.GetSeed(threadIdx.y * blockDim.x + threadIdx.x));

			//CudaKernelData* const kernel = global_kernel;

			// [>] Copy kernel to shared memory
			extern __shared__ CudaKernelData shared_kernel[];
			CudaKernelData* kernel = shared_kernel;

			// copy render index
			if (thread.thread_in_kernel == 0u)
				kernel->renderIndex = global_kernel->renderIndex;

			// copy unsigned random floats
			const uint32_t linear_block_size = blockDim.x * blockDim.y;
			for (uint32_t i = thread.thread_in_block; i < RandomNumbers::s_count; i += linear_block_size)
			{
				kernel->randomNumbers.m_unsigned_uniform[i] =
					global_kernel->randomNumbers.m_unsigned_uniform[i];
			}

			__syncthreads();


			CudaCamera* const camera = &world->cameras[camera_id];
			if (thread.thread_x >= camera->width || thread.thread_y >= camera->height) return;


			// create intersection object
			RayIntersection intersection;
			intersection.ray.direction = cudaVec3<float>(0.0f, 0.0f, 1.0f);

			// ray to screen deflection
			const float x_shift = __tanf(camera->fov * 0.5f);
			const float y_shift = -x_shift / camera->aspect_ratio;
			intersection.ray.direction.x = ((thread.thread_x / (float)camera->width - 0.5f) * x_shift);
			intersection.ray.direction.y = ((thread.thread_y / (float)camera->height - 0.5f) * y_shift);

			// pixel position distortion (antialiasing)
			intersection.ray.direction.x +=
				((0.5f / (float)camera->width) * (kernel->randomNumbers.GetUnsignedUniform(thread) * 2.0f - 1.0f));
			intersection.ray.direction.y +=
				((0.5f / (float)camera->height) * (kernel->randomNumbers.GetUnsignedUniform(thread) * 2.0f - 1.0f));

			// focal point
			const cudaVec3<float> focalPoint = intersection.ray.direction * camera->focal_distance;

			// aperture distortion
			const float apertureAngle = kernel->randomNumbers.GetUnsignedUniform(thread) * 6.28318530f;
			const float apertureSample = kernel->randomNumbers.GetUnsignedUniform(thread) * camera->aperture;
			intersection.ray.origin += cudaVec3<float>(
				apertureSample * __sinf(apertureAngle),
				apertureSample * __cosf(apertureAngle),
				0.0f);

			// depth of field ray
			intersection.ray.direction = focalPoint - intersection.ray.origin;


			// ray direction rotation
			intersection.ray.direction.RotateZ(camera->rotation.z);
			intersection.ray.direction.RotateX(camera->rotation.x);
			intersection.ray.direction.RotateY(camera->rotation.y);
			intersection.ray.direction.Normalize();

			// ray origin rotation
			intersection.ray.origin.RotateZ(camera->rotation.z);
			intersection.ray.origin.RotateX(camera->rotation.x);
			intersection.ray.origin.RotateY(camera->rotation.y);

			// ray transposition
			intersection.ray.origin += camera->position;


			// trace ray from camera
			TracingPath* tracingPath = &camera->GetTracingPath(thread.thread_y * camera->width + thread.thread_x);
			tracingPath->ResetPath();

			/*camera->AppendSample(
				CudaColor<float>(
					kernel->randomNumbers.GetUnsignedUniform(thread),
					kernel->randomNumbers.GetUnsignedUniform(thread),
					kernel->randomNumbers.GetUnsignedUniform(thread)), 
				thread.thread_x, thread.thread_y);
			return;*/

			TraceRay(*kernel, thread, *world, *tracingPath, intersection);
			camera->AppendSample(tracingPath->CalculateFinalColor(), thread.thread_x, thread.thread_y);

			global_kernel->randomNumbers.SetSeed(thread.thread_in_block, thread.seed);
		}

		__device__ void TraceRay(
			CudaKernelData& kernel,
			ThreadData& thread,
			const CudaWorld& world,
			TracingPath& tracing_path,
			RayIntersection& intersection)
		{
			CudaColor<float> color_mask(1.0f, 1.0f, 1.0f);

			do
			{
				bool light_hit = LightsIntersection(world, intersection);
				bool object_hit = ClosestIntersection(world, intersection);

				//color_mask *= intersection.bvh_factor;

				if (!(light_hit || object_hit))
				{	// no hit, return background color

					tracing_path.finalColor += CudaColor<float>::BlendProduct(
						color_mask,
						CudaColor<float>(1.0f, 1.0f, 1.0f) * 0.0f);
					return;
				}

				if (intersection.material.emitance > 0.0f)
				{	// intersection with emitting object

					tracing_path.finalColor += CudaColor<float>::BlendProduct(
						color_mask,
						intersection.surface_color * intersection.material.emitance);
					return;
				}


				// [>] apply Beer's law

				// P0 - light energy in front of an object
				// P - light energy after going through an object
				// A - absorbance

				// e - material absorbance (constant)
				// b - distance traveled in an object
				// c - molar concentration (constant)

				// A = 10 ^ -(e * b * c)
				// P = P0 * A

				color_mask.BlendProduct(
					intersection.surface_color *
					__powf(intersection.ray.material.transmitance, intersection.ray.length));
				


				/*static constexpr float rcp256 = 1.0f / 256.0f;
				static constexpr float max_radiance = 1000.0f;
				static constexpr float min_contribution = rcp256 / max_radiance;
				if (color_mask.red < min_contribution && 
					color_mask.green < min_contribution && 
					color_mask.blue < min_contribution)
					return;*/



				if (!tracing_path.NextNodeAvailable()) return;

				// [>] Generate next ray
				if (intersection.material.transmitance > 0.0f)
				{	// ray fallen into material/object					

					GenerateTransmissiveRay(kernel, thread, intersection);
				}
				else
				{	// ray is reflected from sufrace

					if (kernel.randomNumbers.GetUnsignedUniform(thread) > intersection.material.reflectance)
					{	// diffuse reflection

						CudaColor<float> light_color = TraceLightRays(kernel, thread, world, intersection);
						tracing_path.finalColor += CudaColor<float>::BlendProduct(
							color_mask,
							CudaColor<float>::BlendProduct(intersection.surface_color, light_color));

						GenerateDiffuseRay(kernel, thread, intersection);
					}
					else
					{	// glossy reflection

						GenerateGlossyRay(kernel, thread, intersection);
					}
				}

			} while (tracing_path.FindNextNodeToTrace());
		}
		__device__ bool LightsIntersection(
			const CudaWorld& world,
			RayIntersection& intersection)
		{
			bool hit = false;

			// [>] PointLights
			for (uint32_t index = 0u, tested = 0u; 
				(index < world.pointLights.GetCapacity() && tested < world.pointLights.GetCount()); 
				++index)
			{
				const CudaPointLight* pointLight = &world.pointLights[index];
				if (!pointLight->Exist()) continue;
				++tested;

				const cudaVec3<float> vPL = pointLight->position - intersection.ray.origin;
				const float dPL = vPL.Length();

				// check if light is close enough
				if (dPL >= intersection.ray.length) continue;
				// check if light is in front of ray
				if (cudaVec3<float>::DotProduct(vPL, intersection.ray.direction) < 0.0f) continue;


				const float dist = RayToPointDistance(intersection.ray, pointLight->position);
				if (dist < pointLight->size)
				{	// ray intersects with the light
					intersection.ray.length = dPL;
					intersection.surface_color = pointLight->color;
					intersection.material.emitance = pointLight->emission;
					hit = true;
				}
			}


			// [>] SpotLights
			for (uint32_t index = 0u, tested = 0u; 
				(index < world.spotLights.GetCapacity() && tested < world.spotLights.GetCount()); 
				++index)
			{
				const CudaSpotLight* spotLight = &world.spotLights[index];
				if (!spotLight->Exist()) continue;
				++tested;

				const cudaVec3<float> vPL = spotLight->position - intersection.ray.origin;
				const float dPL = vPL.Length();

				if (dPL >= intersection.ray.length) continue;
				const float vPL_dot_vD = cudaVec3<float>::DotProduct(vPL, intersection.ray.direction);
				if (vPL_dot_vD < 0.0f) continue;

				const float dist = RayToPointDistance(intersection.ray, spotLight->position);
				if (dist < spotLight->size)
				{
					const float t_dist = sqrtf(
						(spotLight->size + spotLight->sharpness) *
						(spotLight->size + spotLight->sharpness) -
						dist * dist);

					const cudaVec3<float> test_point =
						intersection.ray.origin + intersection.ray.direction * vPL_dot_vD -
						intersection.ray.direction * t_dist;

					const float LP_dot_D = cudaVec3<float>::Similarity(
						test_point - spotLight->position, spotLight->direction);
					if (LP_dot_D > spotLight->cos_angle)
					{
						intersection.ray.length = dPL;
						intersection.surface_color = spotLight->color;
						intersection.material.emitance = spotLight->emission;
						hit = true;
					}
				}
			}


			// [>] DirectLights
			if (!(intersection.ray.length < 3.402823466e+38f))
			{
				for (uint32_t index = 0u, tested = 0u; 
					(index < world.directLights.GetCapacity() && tested < world.directLights.GetCount()); 
					++index)
				{
					const CudaDirectLight* directLight = &world.directLights[index];
					if (!directLight->Exist()) continue;
					++tested;

					const float dot = cudaVec3<float>::DotProduct(
						intersection.ray.direction, 
						-directLight->direction);
					if (dot > directLight->cos_angular_size)
					{
						intersection.surface_color = directLight->color;
						intersection.material.emitance = directLight->emission;
						hit = true;
					}
				}
			}

			return hit;
		}
		__device__ bool ClosestIntersection(
			const CudaWorld& World,
			RayIntersection& intersection)
		{
			const CudaRenderObject* closest_object = nullptr;

			// ~~~~ linear search ~~~~
			/*// [>] Check every single sphere
			for (uint32_t index = 0u, tested = 0u; 
				(index < World.spheres.GetContainer().GetCapacity() && 
					tested < World.spheres.GetContainer().GetCount());
				++index)
			{
				if (!World.spheres.GetContainer()[index].Exist()) continue;
				const CudaSphere* sphere = &World.spheres.GetContainer()[index];
				++tested;

				if (sphere->RayIntersect(currentIntersection))
				{
					closest_object = sphere;
				}
			}*/

			World.spheres.GetBVH().ClosestIntersection(
				intersection,
				closest_object);

			World.meshes.GetBVH().ClosestIntersection(
				intersection,
				closest_object);


			return closest_object != nullptr;
		}
		__device__ float AnyIntersection(
			const CudaWorld& world,
			const CudaRay& shadow_ray)
		{
			float total_shadow = 1.0f;

			/*// [>] Test intersection with every sphere
			for (uint32_t index = 0u, tested = 0u; 
				(index < world.spheres.GetContainer().GetCapacity() && 
					tested < world.spheres.GetContainer().GetCount());
				++index)
			{
				if (!world.spheres.GetContainer()[index].Exist()) continue;
				const CudaSphere* sphere = &world.spheres.GetContainer()[index];
				++tested;

				total_shadow *= sphere->ShadowRayIntersect(shadow_ray);
				if (total_shadow < 0.0001f) return total_shadow;
			}*/

			total_shadow *= world.spheres.GetBVH().AnyIntersection(shadow_ray);
			if (total_shadow < 0.0001f) return total_shadow;

			total_shadow *= world.meshes.GetBVH().AnyIntersection(shadow_ray);
			if (total_shadow < 0.0001f) return total_shadow;


			return total_shadow;
		}
		__device__ CudaColor<float> TraceLightRays(
			CudaKernelData& kernel,
			ThreadData& thread,
			const CudaWorld& world,
			RayIntersection& intersection)
		{
			// Legend:
			// L - position of current light
			// P - point of intersetion
			// vN - surface normal

			float distFactor = 1.0f;
			float vPL_dot_vN = 1.0f;
			float dPL = 0.0f;

			CudaColor<float> accLightColor(0.0f, 0.0f, 0.0f);

			// [>] PointLights
			for (uint32_t index = 0u, tested = 0u; 
				(index < world.pointLights.GetCapacity() && tested < world.pointLights.GetCount()); 
				++index)
			{
				const CudaPointLight* point_light = &world.pointLights[index];
				if (!point_light->Exist()) continue;
				++tested;


				// randomize point light position
				const cudaVec3<float> distLightPos = point_light->position + cudaVec3<float>(
					kernel.randomNumbers.GetUnsignedUniform(thread) * 2.0f - 1.0f,
					kernel.randomNumbers.GetUnsignedUniform(thread) * 2.0f - 1.0f,
					kernel.randomNumbers.GetUnsignedUniform(thread) * 2.0f - 1.0f) * point_light->size;

				// vector from point to light position
				const cudaVec3<float> vPL = distLightPos - intersection.point;

				// dot product with surface normal
				vPL_dot_vN = cudaVec3<float>::Similarity(vPL, intersection.mapped_normal);
				if (vPL_dot_vN <= 0.0f) continue;

				// calculate light energy P
				dPL = vPL.Length();
				distFactor = 1.0f / (dPL * dPL + 1.0f);
				float energyAtP = point_light->emission * distFactor * vPL_dot_vN;
				if (energyAtP < 0.0001f) continue;	// unimportant light contribution

				// cast shadow ray and calculate color contribution
				CudaRay shadowRay(intersection.point + intersection.surface_normal * 0.0001f, vPL, dPL);
				accLightColor += point_light->color * energyAtP * AnyIntersection(world, shadowRay);
			}


			// [>] SpotLights
			for (uint32_t index = 0u, tested = 0u; 
				(index < world.spotLights.GetCapacity() && tested < world.spotLights.GetCount()); 
				++index)
			{
				const CudaSpotLight* spotLight = &world.spotLights[index];
				if (!spotLight->Exist()) continue;
				++tested;

				// randomize spot light position
				const cudaVec3<float> distLightPos = spotLight->position + cudaVec3<float>(
					kernel.randomNumbers.GetUnsignedUniform(thread) * 2.0f - 1.0f,
					kernel.randomNumbers.GetUnsignedUniform(thread) * 2.0f - 1.0f,
					kernel.randomNumbers.GetUnsignedUniform(thread) * 2.0f - 1.0f) * spotLight->size;

				// vector from point to light position
				const cudaVec3<float> vPL = distLightPos - intersection.point;

				// dot product with surface normal
				vPL_dot_vN = cudaVec3<float>::Similarity(vPL, intersection.mapped_normal);
				if (vPL_dot_vN <= 0.0f) continue;

				// calculate light energy at P
				dPL = vPL.Length();
				distFactor = 1.0f / (dPL * dPL + 1.0f);

				float beamIllum = 1.0f;
				float LP_dot_D = cudaVec3<float>::Similarity(-vPL, spotLight->direction);
				if (LP_dot_D < spotLight->cos_angle) beamIllum = 0.0f;
				else beamIllum = 1.0f;

				const float energyAtP = spotLight->emission * distFactor * beamIllum * vPL_dot_vN;
				if (energyAtP < 0.0001f) continue;	// unimportant light contribution

				// cast shadow ray and calculate color contribution
				const CudaRay shadowRay(intersection.point + intersection.surface_normal * 0.001f, vPL, dPL);
				accLightColor += spotLight->color * energyAtP * AnyIntersection(world, shadowRay);
			}


			// [>] DirectLights
			for (uint32_t index = 0u, tested = 0u; 
				(index < world.directLights.GetCapacity() && tested < world.directLights.GetCount()); 
				++index)
			{
				const CudaDirectLight* directLight = &world.directLights[index];
				if (!directLight->Exist()) continue;
				++tested;

				// vector from point to direct light (reversed direction)
				cudaVec3<float> vPL = SampleSphere(
					kernel.randomNumbers.GetUnsignedUniform(thread),
					kernel.randomNumbers.GetUnsignedUniform(thread) * directLight->angular_size * 0.318309f,
					-directLight->direction);

				// dot product with sufrace normal
				vPL_dot_vN = cudaVec3<float>::Similarity(vPL, intersection.mapped_normal);
				if (vPL_dot_vN <= 0.0f) continue;

				// calculate light energy at P
				float energyAtP = directLight->emission * vPL_dot_vN;
				if (energyAtP < 0.0001f) continue;	// unimportant light contribution

				// cast shadow ray and calculate color contribution
				CudaRay shadowRay(intersection.point + intersection.surface_normal * 0.0001f, vPL);
				accLightColor += directLight->color * energyAtP * AnyIntersection(world, shadowRay);
			}

			return accLightColor;
		}

		__device__ void GenerateDiffuseRay(
			CudaKernelData& kernel,
			ThreadData& thread,
			RayIntersection& intersection)
		{
			cudaVec3<float> sample = CosineSampleHemisphere(
				kernel.randomNumbers.GetUnsignedUniform(thread),
				kernel.randomNumbers.GetUnsignedUniform(thread),
				intersection.mapped_normal);
			sample.Normalize();

			// flip sample above surface if needed
			const float vR_dot_vN = cudaVec3<float>::Similarity(sample, intersection.surface_normal);
			if (vR_dot_vN < 0.0f) sample += intersection.surface_normal * -2.0f * vR_dot_vN;

			new (&intersection.ray) CudaSceneRay(
				intersection.point + intersection.surface_normal * 0.0001f,
				sample,
				intersection.ray.material);
		}
		__device__ void GenerateSpecularRay(
			CudaKernelData& kernel,
			RayIntersection& intersection)
		{
			cudaVec3<float> reflect = ReflectVector(
				intersection.ray.direction,
				intersection.mapped_normal);

			// flip sample above surface if needed
			const float vR_dot_vN = cudaVec3<float>::Similarity(reflect, intersection.surface_normal);
			if (vR_dot_vN < 0.0f) reflect += intersection.surface_normal * -2.0f * vR_dot_vN;

			new (&intersection.ray) CudaSceneRay(
				intersection.point + intersection.surface_normal * 0.0001f,
				reflect, intersection.ray.material);
		}
		__device__ void GenerateGlossyRay(
			CudaKernelData& kernel,
			ThreadData& thread,
			RayIntersection& intersection)
		{
			if (intersection.material.glossiness > 0.0f)
			{
				const cudaVec3<float> vNd = SampleHemisphere(
					kernel.randomNumbers.GetUnsignedUniform(thread),
					1.0f - __powf(
						kernel.randomNumbers.GetUnsignedUniform(thread),
						intersection.material.glossiness),
					intersection.mapped_normal);

				// calculate reflection direction
				cudaVec3<float> vR = ReflectVector(
					intersection.ray.direction,
					vNd);

				// reflect sample above surface if needed
				const float vR_dot_vN = cudaVec3<float>::Similarity(vR, intersection.surface_normal);
				if (vR_dot_vN < 0.0f) vR += intersection.surface_normal * -2.0f * vR_dot_vN;

				// create next glossy CudaSceneRay
				new (&intersection.ray) CudaSceneRay(
					intersection.point + intersection.surface_normal * 0.0001f,
					vR,
					intersection.ray.material);
			}
			else
			{	// minimum/zero glossiness = perfect mirror

				GenerateSpecularRay(kernel, intersection);
			}

			/*
			* GlossySpecular::sample_f(const ShadeRec& sr,
				const Vector3D& wo,
				Vector3D& wi,
				float& pdf) const
			{
				float ndotwo = sr.normal * wo;
				Vector3D r = -wo + 2.0 * sr.normal * ndotwo; // direction of mirror reflection


				Vector3D w = r;
				Vector3D u = Vector3D(0.00424, 1, 0.00764) ^ w;
				u.normalize();
				Vector3D v = u ^ w;

				Point3D sp = sampler_ptr->sample_hemisphere();
				wi = sp.x * u + sp.y * v + sp.z * w; // reflected ray direction

				if (sr.normal * wi < 0.0) // reflected ray is below surface
				wi = -sp.x * u - sp.y * v + sp.z * w;

				float phong_lobe = pow(r * wi, exp);
				pdf = phong_lobe * (sr.normal * wi);

				return (ks * cs * phong_lobe);
			}
			*/
		}
		__device__ void GenerateTransmissiveRay(
			CudaKernelData& kernel,
			ThreadData& thread,
			RayIntersection& intersection)
		{
			if (intersection.material.ior != intersection.ray.material.ior)
			{	// refraction ray

				const float cosi = fabsf(cudaVec3<float>::DotProduct(
					intersection.ray.direction, intersection.mapped_normal));

				// calculate sin^2 theta from Snell's law
				const float n1 = intersection.ray.material.ior;
				const float n2 = intersection.material.ior;
				const float ratio = n1 / n2;
				const float sin2_t = ratio * ratio * (1.0f - cosi * cosi);

				if (sin2_t >= 1.0f)
				{	// TIR

					// calculate reflection vector
					cudaVec3<float> vR = ReflectVector(
						intersection.ray.direction,
						intersection.mapped_normal);

					// flip sample above surface if needed
					const float vR_dot_vN = cudaVec3<float>::DotProduct(vR, intersection.surface_normal);
					if (vR_dot_vN < 0.0f) vR += intersection.surface_normal * -2.0f * vR_dot_vN;

					// create new internal reflection CudaSceneRay
					new (&intersection.ray) CudaSceneRay(
						intersection.point + intersection.surface_normal * 0.0001f,
						vR,
						intersection.ray.material);
				}
				else
				{
					// calculate fresnel
					const float cost = sqrtf(1.0f - sin2_t);
					const float Rp = ((n1 * cosi) - (n2 * cost)) / ((n1 * cosi) + (n2 * cost));
					const float Rs = ((n2 * cosi) - (n1 * cost)) / ((n2 * cosi) + (n1 * cost));
					const float f = (Rs * Rs + Rp * Rp) / 2.0f;

					if (f < kernel.randomNumbers.GetUnsignedUniform(thread))
					{	// transmission/refraction

						// calculate refraction direction
						const cudaVec3<float> vR = intersection.ray.direction * ratio +
							intersection.mapped_normal * (ratio * cosi - cost);

						// create new refraction CudaSceneRay
						new (&intersection.ray) CudaSceneRay(
							intersection.point - intersection.surface_normal * 0.0001f,
							vR,
							intersection.material);
					}
					else
					{	// reflection

						// calculate reflection direction
						cudaVec3<float> vR = ReflectVector(
							intersection.ray.direction,
							intersection.mapped_normal);

						// flip sample above surface if needed
						const float vR_dot_vN = cudaVec3<float>::DotProduct(vR, intersection.surface_normal);
						if (vR_dot_vN < 0.0f) vR += intersection.surface_normal * -2.0f * vR_dot_vN;

						// create new reflection CudaSceneRay
						new (&intersection.ray) CudaSceneRay(
							intersection.point + intersection.surface_normal * 0.0001f,
							vR,
							intersection.ray.material);
					}
				}
			}
			else
			{	// transparent ray

				cudaVec3<float> vD;

				if (intersection.material.glossiness > 0.0f)
				{
					vD = SampleSphere(
						kernel.randomNumbers.GetUnsignedUniform(thread),
						1.0f - __powf(
							kernel.randomNumbers.GetUnsignedUniform(thread),
							intersection.material.glossiness),
						intersection.ray.direction);

					const float vS_dot_vN = cudaVec3<float>::DotProduct(vD, -intersection.surface_normal);
					if (vS_dot_vN < 0.0f) vD += -intersection.surface_normal * -2.0f * vS_dot_vN;
				}
				else
				{
					vD = intersection.ray.direction;
				}

				new (&intersection.ray) CudaSceneRay(
					intersection.point - intersection.surface_normal * 0.0001f,
					vD,
					intersection.material);
			}
		}



		// [>] Tone mapping
		__global__ void ToneMap(
			CudaKernelData* const kernel_data,
			CudaWorld* const world,
			const int camera_id)
		{
			CudaCamera* const camera = &world->cameras[camera_id];

			// calculate thread position
			const uint32_t thread_x = blockIdx.x * blockDim.x + threadIdx.x;
			const uint32_t thread_y = blockIdx.y * blockDim.y + threadIdx.y;
			if (thread_x >= camera->width || thread_y >= camera->height) return;

			// average sample color by dividing by number of samples
			CudaColor<float> samplingColor =
				camera->GetSample(thread_x, thread_y) / (float)camera->samples_count;

			// tone map sample color
			camera->SetFinalPixel(kernel_data->renderIndex,
				CudaColor<unsigned char>(
					(samplingColor.red / (samplingColor.red + 1.0f)) * 255.0f,
					(samplingColor.green / (samplingColor.green + 1.0f)) * 255.0f,
					(samplingColor.blue / (samplingColor.blue + 1.0f)) * 255.0f,
					255u),
				thread_x, thread_y);
		}


		// [>] CudaCamera progressive rendering management
		__global__ void CudaCameraSampleReset(
			CudaWorld* const world,
			const int camera_id)
		{
			CudaCamera* const camera = &world->cameras[camera_id];
			if (!camera->Exist()) return;

			// calculate thread position
			const uint32_t thread_x = blockIdx.x * blockDim.x + threadIdx.x;
			const uint32_t thread_y = blockIdx.y * blockDim.y + threadIdx.y;
			if (thread_x >= camera->width || thread_y >= camera->height) return;

			// reset sample buffer 
			camera->SetSample(CudaColor<float>(0.0f, 0.0f, 0.0f), thread_x, thread_y);

			// TODO: reset tracing paths
		}
		__global__ void CudaCameraUpdateSamplesNumber(
			CudaWorld* const world,
			const int camera_id,
			bool reset_flag)
		{
			CudaCamera* const camera = &world->cameras[camera_id];
			if (reset_flag)	camera->samples_count = 1u;
			else			camera->samples_count += 1u;
		}
	}
}